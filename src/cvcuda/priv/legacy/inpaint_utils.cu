#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "CvCudaUtils.cuh"
#include "reduce_kernel_utils.cuh"

using namespace nvcv::legacy::cuda_op;

__global__ void deviceReducePoints(const int *g_in, int *g_out, const int N)
{
    int tid = threadIdx.x;
    int sum = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        sum += g_in[i];
    }
    sum = blockReduceSum(sum);
    if (tid == 0)
    {
        g_out[tid] = sum;
    }
}
