#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

#include <nvcv/IImage.hpp>
#include <nvcv/ImageData.hpp>
#include <nvcv/TensorData.hpp>

#include <cstdio>

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;
using namespace nvcv::cuda::osd;

namespace nvcv::legacy::cuda_op {

template<typename _T>
static __forceinline__ __device__ _T limit(_T value, _T low, _T high)
{
    return value < low ? low : (value > high ? high : value);
}

template<class SrcWrapper, class DstWrapper>
static __global__ void render_p2p_kernel(SrcWrapper src, DstWrapper dst, int batch, int height, int width, int channel)
{
    int       ix        = blockDim.x * blockIdx.x + threadIdx.x;
    int       iy        = blockDim.y * blockIdx.y + threadIdx.y;
    const int batch_idx = get_batch_idx();

    if (ix >= width || iy >= height || batch_idx >= batch)
        return;

    if (channel == 3)
    {
        *(uchar3 *)(dst.ptr(batch_idx, iy, ix, 0)) = *(uchar3 *)(src.ptr(batch_idx, iy, ix, 0));
    }
    else
    {
        *(uchar4 *)(dst.ptr(batch_idx, iy, ix, 0)) = *(uchar4 *)(src.ptr(batch_idx, iy, ix, 0));
    }
}

template<class SrcWrapper, class DstWrapper>
static __global__ void render_blur_rgb_kernel(SrcWrapper src, DstWrapper dst, const BoxBlurCommand *commands,
                                              int num_command, int image_batch, int image_width, int image_height)
{
    if (blockIdx.x >= num_command)
        return;
    const BoxBlurCommand &box = commands[blockIdx.x];
    if (box.batch_index >= image_batch)
        return;

    __shared__ uchar3 crop[32][32];
    int               ix = threadIdx.x;
    int               iy = threadIdx.y;

    int boxwidth  = box.bounding_right - box.bounding_left;
    int boxheight = box.bounding_bottom - box.bounding_top;
    int sx        = limit((int)(ix / 32.0f * (float)boxwidth + 0.5f + box.bounding_left), 0, image_width);
    int sy        = limit((int)(iy / 32.0f * (float)boxheight + 0.5f + box.bounding_top), 0, image_height);

    crop[iy][ix] = *(uchar3 *)(src.ptr(box.batch_index, sy, sx, 0));
    __syncthreads();

    uint3 color = make_uint3(0, 0, 0);
    int   n     = 0;
    for (int i = -box.kernel_size / 2; i <= box.kernel_size / 2; ++i)
    {
        for (int j = -box.kernel_size / 2; j <= box.kernel_size / 2; ++j)
        {
            int u = i + iy;
            int v = j + ix;
            if (u >= 0 && u < 32 && v >= 0 && v < 32)
            {
                auto &c = crop[u][v];
                color.x += c.x;
                color.y += c.y;
                color.z += c.z;
                n++;
            }
        }
    }
    __syncthreads();
    crop[iy][ix] = make_uchar3(color.x / n, color.y / n, color.z / n);
    __syncthreads();

    int gap_width  = (boxwidth + 31) / 32;
    int gap_height = (boxheight + 31) / 32;
    for (int i = 0; i < gap_height; ++i)
    {
        for (int j = 0; j < gap_width; ++j)
        {
            int fx = ix * gap_width + j + box.bounding_left;
            int fy = iy * gap_height + i + box.bounding_top;
            if (fx >= 0 && fx < image_width && fy >= 0 && fy < image_height)
            {
                int sx = (ix * gap_width + j) / (float)boxwidth * 32;
                int sy = (iy * gap_height + i) / (float)boxheight * 32;
                if (sx < 32 && sy < 32)
                {
                    auto &pix                                        = crop[sy][sx];
                    *(uchar3 *)(dst.ptr(box.batch_index, fy, fx, 0)) = make_uchar3(pix.x, pix.y, pix.z);
                }
            }
        }
    }
}

template<class SrcWrapper, class DstWrapper>
static __global__ void render_blur_rgba_kernel(SrcWrapper src, DstWrapper dst, const BoxBlurCommand *commands,
                                               int num_command, int image_batch, int image_width, int image_height)
{
    if (blockIdx.x >= num_command)
        return;
    const BoxBlurCommand &box = commands[blockIdx.x];
    if (box.batch_index >= image_batch)
        return;

    __shared__ uchar3 crop[32][32];
    int               ix = threadIdx.x;
    int               iy = threadIdx.y;

    int boxwidth  = box.bounding_right - box.bounding_left;
    int boxheight = box.bounding_bottom - box.bounding_top;
    int sx        = limit((int)(ix / 32.0f * (float)boxwidth + 0.5f + box.bounding_left), 0, image_width);
    int sy        = limit((int)(iy / 32.0f * (float)boxheight + 0.5f + box.bounding_top), 0, image_height);

    crop[iy][ix] = *(uchar3 *)(src.ptr(box.batch_index, sy, sx, 0));
    __syncthreads();

    uint3 color = make_uint3(0, 0, 0);
    int   n     = 0;
    for (int i = -box.kernel_size / 2; i <= box.kernel_size / 2; ++i)
    {
        for (int j = -box.kernel_size / 2; j <= box.kernel_size / 2; ++j)
        {
            int u = i + iy;
            int v = j + ix;
            if (u >= 0 && u < 32 && v >= 0 && v < 32)
            {
                auto &c = crop[u][v];
                color.x += c.x;
                color.y += c.y;
                color.z += c.z;
                n++;
            }
        }
    }
    __syncthreads();
    crop[iy][ix] = make_uchar3(color.x / n, color.y / n, color.z / n);
    __syncthreads();

    int gap_width  = (boxwidth + 31) / 32;
    int gap_height = (boxheight + 31) / 32;
    for (int i = 0; i < gap_height; ++i)
    {
        for (int j = 0; j < gap_width; ++j)
        {
            int fx = ix * gap_width + j + box.bounding_left;
            int fy = iy * gap_height + i + box.bounding_top;
            if (fx >= 0 && fx < image_width && fy >= 0 && fy < image_height)
            {
                int sx = (ix * gap_width + j) / (float)boxwidth * 32;
                int sy = (iy * gap_height + i) / (float)boxheight * 32;
                if (sx < 32 && sy < 32)
                {
                    auto &pix                                        = crop[sy][sx];
                    *(uchar4 *)(dst.ptr(box.batch_index, fy, fx, 0)) = make_uchar4(pix.x, pix.y, pix.z, 255);
                }
            }
        }
    }
}

static void cuosd_apply(cuOSDContext_t context, hipStream_t stream)
{
    if (!context->blur_commands.empty())
    {
        if (context->gpu_blur_commands == nullptr)
        {
            context->gpu_blur_commands.reset(new Memory<BoxBlurCommand>());
        }

        context->gpu_blur_commands->alloc_or_resize_to(context->blur_commands.size());

        for (int i = 0; i < (int)context->blur_commands.size(); ++i)
        {
            auto &cmd = context->blur_commands[i];
            memcpy((void *)(context->gpu_blur_commands->host() + i), (void *)cmd.get(), sizeof(BoxBlurCommand));
        }

        context->gpu_blur_commands->copy_host_to_device(stream);
    }
}

inline ErrorCode ApplyBoxBlur_RGB(const nvcv::TensorDataStridedCuda &inData, const nvcv::TensorDataStridedCuda &outData,
                                  cuOSDContext_t context, hipStream_t stream)
{
    auto inAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);

    cuda_op::DataType  inDataType = helpers::GetLegacyDataType(inData.dtype());
    cuda_op::DataShape inputShape = helpers::GetLegacyDataShape(inAccess->infoShape());

    auto outAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    cuda_op::DataType  outDataType = helpers::GetLegacyDataType(outData.dtype());
    cuda_op::DataShape outputShape = helpers::GetLegacyDataShape(outAccess->infoShape());

    if (outDataType != inDataType)
    {
        LOG_ERROR("Unsupported input/output DataType " << inDataType << "/" << outDataType);
        return ErrorCode::INVALID_DATA_TYPE;
    }
    if (outputShape.H != inputShape.H || outputShape.W != inputShape.W || outputShape.N != inputShape.N
        || outputShape.C != inputShape.C || outputShape.C != 3)
    {
        LOG_ERROR("Invalid output shape " << outputShape);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    cuosd_apply(context, stream);

    auto src = nvcv::cuda::CreateTensorWrapNHWC<uint8_t>(inData);
    auto dst = nvcv::cuda::CreateTensorWrapNHWC<uint8_t>(outData);

    if (inData.basePtr() != outData.basePtr())
    {
        dim3 blockSize(32, 32);
        dim3 gridSize(divUp(int(inputShape.W + 1), (int)blockSize.x), divUp(int(inputShape.H + 1), (int)blockSize.y),
                      inputShape.N);

        render_p2p_kernel<<<gridSize, blockSize, 0, stream>>>(src, dst, inputShape.N, inputShape.H, inputShape.W,
                                                              inputShape.C);
        checkKernelErrors();
    }

    if (context->blur_commands.size() > 0)
    {
        dim3 blockSize(32, 32);
        dim3 gridSize(context->blur_commands.size(), 1);

        render_blur_rgb_kernel<<<gridSize, blockSize, 0, stream>>>(
            src, dst, context->gpu_blur_commands ? context->gpu_blur_commands->device() : nullptr,
            context->blur_commands.size(), inputShape.N, inputShape.W, inputShape.H);
        checkKernelErrors();
    }
    return ErrorCode::SUCCESS;
}

inline ErrorCode ApplyBoxBlur_RGBA(const nvcv::TensorDataStridedCuda &inData,
                                   const nvcv::TensorDataStridedCuda &outData, cuOSDContext_t context,
                                   hipStream_t stream)
{
    auto inAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);

    cuda_op::DataType  inDataType = helpers::GetLegacyDataType(inData.dtype());
    cuda_op::DataShape inputShape = helpers::GetLegacyDataShape(inAccess->infoShape());

    auto outAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    cuda_op::DataType  outDataType = helpers::GetLegacyDataType(outData.dtype());
    cuda_op::DataShape outputShape = helpers::GetLegacyDataShape(outAccess->infoShape());

    if (outDataType != inDataType)
    {
        LOG_ERROR("Unsupported input/output DataType " << inDataType << "/" << outDataType);
        return ErrorCode::INVALID_DATA_TYPE;
    }
    if (outputShape.H != inputShape.H || outputShape.W != inputShape.W || outputShape.N != inputShape.N
        || outputShape.C != inputShape.C)
    {
        LOG_ERROR("Invalid output shape " << outputShape);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    cuosd_apply(context, stream);

    auto src = nvcv::cuda::CreateTensorWrapNHWC<uint8_t>(inData);
    auto dst = nvcv::cuda::CreateTensorWrapNHWC<uint8_t>(outData);

    if (inData.basePtr() != outData.basePtr())
    {
        dim3 blockSize(32, 32);
        dim3 gridSize(divUp(int(inputShape.W + 1), (int)blockSize.x), divUp(int(inputShape.H + 1), (int)blockSize.y),
                      inputShape.N);

        render_p2p_kernel<<<gridSize, blockSize, 0, stream>>>(src, dst, inputShape.N, inputShape.H, inputShape.W,
                                                              inputShape.C);
        checkKernelErrors();
    }

    if (context->blur_commands.size() > 0)
    {
        dim3 blockSize(32, 32);
        dim3 gridSize(context->blur_commands.size(), 1);

        render_blur_rgba_kernel<<<gridSize, blockSize, 0, stream>>>(
            src, dst, context->gpu_blur_commands ? context->gpu_blur_commands->device() : nullptr,
            context->blur_commands.size(), inputShape.N, inputShape.W, inputShape.H);
        checkKernelErrors();
    }
    return ErrorCode::SUCCESS;
}

static ErrorCode cuosd_draw_boxblur(cuOSDContext_t context, int width, int height, NVCVBlurBoxesI bboxes)
{
    for (int n = 0; n < bboxes.batch; n++)
    {
        auto numBoxes = bboxes.numBoxes[n];

        for (int i = 0; i < numBoxes; i++)
        {
            auto bbox   = bboxes.boxes[i];
            int  left   = max(min(bbox.box.x, width - 1), 0);
            int  top    = max(min(bbox.box.y, height - 1), 0);
            int  right  = max(min(left + bbox.box.width - 1, width - 1), 0);
            int  bottom = max(min(top + bbox.box.height - 1, height - 1), 0);

            if (left == right || top == bottom)
            {
                LOG_DEBUG("Skipped box_blur(" << bbox.box.x << ", " << bbox.box.y << ", " << bbox.box.width << ", "
                                              << bbox.box.height << ") in image(" << width << ", " << height << ")");
                continue;
            }

            if (bbox.box.width < 3 || bbox.box.height < 3 || bbox.kernelSize < 1)
            {
                LOG_DEBUG(
                    "This operation will be ignored because the region of interest is too small, or the kernel is too "
                    "small at box_blur("
                    << bbox.box.x << ", " << bbox.box.y << bbox.box.width << ", " << bbox.box.height
                    << ") with kernelSize=" << bbox.kernelSize);
                continue;
            }

            auto cmd             = std::make_shared<BoxBlurCommand>();
            cmd->batch_index     = n;
            cmd->kernel_size     = bbox.kernelSize;
            cmd->bounding_left   = left;
            cmd->bounding_right  = right;
            cmd->bounding_top    = top;
            cmd->bounding_bottom = bottom;
            context->blur_commands.emplace_back(cmd);
        }

        bboxes.boxes = (NVCVBlurBoxI *)((uint8_t *)bboxes.boxes + numBoxes * sizeof(NVCVBlurBoxI));
    }
    return ErrorCode::SUCCESS;
}

BoxBlur::BoxBlur(DataShape max_input_shape, DataShape max_output_shape)
    : CudaBaseOp(max_input_shape, max_output_shape)
{
    m_context = new cuOSDContext();
    if (m_context->gpu_blur_commands == nullptr)
    {
        m_context->gpu_blur_commands.reset(new Memory<BoxBlurCommand>());
    }
    m_context->gpu_blur_commands->alloc_or_resize_to(PREALLOC_CMD_NUM * sizeof(BoxBlurCommand));
}

BoxBlur::~BoxBlur()
{
    if (m_context)
    {
        m_context->blur_commands.clear();
        cuOSDContext *p = (cuOSDContext *)m_context;
        delete p;
    }
}

size_t BoxBlur::calBufferSize(DataShape max_input_shape, DataShape max_output_shape, DataType max_data_type)
{
    return 0;
}

ErrorCode BoxBlur::infer(const nvcv::TensorDataStridedCuda &inData, const nvcv::TensorDataStridedCuda &outData,
                         NVCVBlurBoxesI bboxes, hipStream_t stream)
{
    cuda_op::DataFormat input_format  = GetLegacyDataFormat(inData.layout());
    cuda_op::DataFormat output_format = GetLegacyDataFormat(outData.layout());

    if (!(input_format == kNHWC || input_format == kHWC) || !(output_format == kNHWC || output_format == kHWC))
    {
        LOG_ERROR("Invliad DataFormat both Input and Output must be kNHWC or kHWC");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (inData.dtype() != outData.dtype())
    {
        LOG_ERROR("Input and Output formats must be same input format =" << inData.dtype()
                                                                         << " output format = " << outData.dtype());
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto inAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(inData);
    if (!inAccess)
    {
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    int batch    = inAccess->numSamples();
    int channels = inAccess->numChannels();
    int rows     = inAccess->numRows();
    int cols     = inAccess->numCols();

    if (channels > 4 || channels < 1)
    {
        LOG_ERROR("Invalid channel number ch = " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    if (bboxes.batch != batch)
    {
        LOG_ERROR("Invalid bboxes batch = " << bboxes.batch);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    auto outAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(outData);
    if (!outAccess)
    {
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto ret = cuosd_draw_boxblur(m_context, cols, rows, bboxes);
    if (ret != ErrorCode::SUCCESS)
    {
        return ret;
    }

    typedef ErrorCode (*func_t)(const nvcv::TensorDataStridedCuda &inData, const nvcv::TensorDataStridedCuda &outData,
                                cuOSDContext_t context, hipStream_t stream);

    static const func_t funcs[] = {
        ApplyBoxBlur_RGB,
        ApplyBoxBlur_RGBA,
    };

    int type_idx = channels - 3;
    funcs[type_idx](inData, outData, m_context, stream);
    m_context->blur_commands.clear(); // Clear the command buffer so next render does not contain previous boxes.
    m_context->rect_commands.clear();
    return ErrorCode::SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
