#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

#include <nvcv/Image.hpp>
#include <nvcv/ImageData.hpp>
#include <nvcv/TensorData.hpp>

#include <cstdio>

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;
using namespace nvcv::cuda::osd;

namespace nvcv::legacy::cuda_op {

template<typename _T>
static __host__ __device__ uint8_t u8cast(_T value)
{
    return value < 0 ? 0 : (value > 255 ? 255 : value);
}

// inbox_single_pixel:
// check if given coordinate is in box
//      a --- d
//      |     |
//      b --- c
static __device__ __forceinline__ bool inbox_single_pixel(float ix, float iy, float ax, float ay, float bx, float by,
                                                          float cx, float cy, float dx, float dy)
{
    return ((bx - ax) * (iy - ay) - (by - ay) * (ix - ax)) < 0 && ((cx - bx) * (iy - by) - (cy - by) * (ix - bx)) < 0
        && ((dx - cx) * (iy - cy) - (dy - cy) * (ix - cx)) < 0 && ((ax - dx) * (iy - dy) - (ay - dy) * (ix - dx)) < 0;
}

static __device__ void blend_single_color(uchar4 &color, uint8_t &c0, uint8_t &c1, uint8_t &c2, uint8_t a)
{
    int foreground_alpha = a;
    int background_alpha = color.w;
    int blend_alpha      = ((background_alpha * (255 - foreground_alpha)) >> 8) + foreground_alpha;
    color.x = u8cast((((color.x * background_alpha * (255 - foreground_alpha)) >> 8) + (c0 * foreground_alpha))
                     / blend_alpha);
    color.y = u8cast((((color.y * background_alpha * (255 - foreground_alpha)) >> 8) + (c1 * foreground_alpha))
                     / blend_alpha);
    color.z = u8cast((((color.z * background_alpha * (255 - foreground_alpha)) >> 8) + (c2 * foreground_alpha))
                     / blend_alpha);
    color.w = blend_alpha;
}

// render_rectangle_fill:
// render filled rectangle with border msaa4x interpolation off
static __device__ void render_rectangle_fill(int ix, int iy, RectangleCommand *p, uchar4 color[4])
{
    if (inbox_single_pixel(ix, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1))
    {
        blend_single_color(color[0], p->c0, p->c1, p->c2, p->c3);
    }
    if (inbox_single_pixel(ix + 1, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1))
    {
        blend_single_color(color[1], p->c0, p->c1, p->c2, p->c3);
    }
    if (inbox_single_pixel(ix, iy + 1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1))
    {
        blend_single_color(color[2], p->c0, p->c1, p->c2, p->c3);
    }
    if (inbox_single_pixel(ix + 1, iy + 1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1))
    {
        blend_single_color(color[3], p->c0, p->c1, p->c2, p->c3);
    }
}

// render_rectangle_border:
// render hollow rectangle with border msaa4x interpolation off
static __device__ void render_rectangle_border(int ix, int iy, RectangleCommand *p, uchar4 color[4])
{
    if (!inbox_single_pixel(ix, iy, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2)
        && inbox_single_pixel(ix, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1))
    {
        blend_single_color(color[0], p->c0, p->c1, p->c2, p->c3);
    }
    if (!inbox_single_pixel(ix + 1, iy, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2)
        && inbox_single_pixel(ix + 1, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1))
    {
        blend_single_color(color[1], p->c0, p->c1, p->c2, p->c3);
    }
    if (!inbox_single_pixel(ix, iy + 1, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2)
        && inbox_single_pixel(ix, iy + 1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1))
    {
        blend_single_color(color[2], p->c0, p->c1, p->c2, p->c3);
    }
    if (!inbox_single_pixel(ix + 1, iy + 1, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2)
        && inbox_single_pixel(ix + 1, iy + 1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1))
    {
        blend_single_color(color[3], p->c0, p->c1, p->c2, p->c3);
    }
}

static __device__ void do_rectangle_woMSAA(RectangleCommand *cmd, int ix, int iy, uchar4 context_color[4])
{
    if (cmd->thickness == -1)
    {
        render_rectangle_fill(ix, iy, cmd, context_color);
    }
    else
    {
        render_rectangle_border(ix, iy, cmd, context_color);
    }
}

template<class SrcWrapper, class DstWrapper, typename T = typename DstWrapper::ValueType>
static __device__ void blending_rgb_pixel(SrcWrapper src, DstWrapper dst, int x, int y, uchar4 plot_colors[4])
{
    const int batch_idx = get_batch_idx();

    for (int i = 0; i < 2; ++i)
    {
        T *in  = src.ptr(batch_idx, y + i, x, 0);
        T *out = dst.ptr(batch_idx, y + i, x, 0);
        for (int j = 0; j < 2; ++j, in += 3, out += 3)
        {
            uchar4 &rcolor           = plot_colors[i * 2 + j];
            int     foreground_alpha = rcolor.w;
            int     background_alpha = 255;
            int     blend_alpha      = ((background_alpha * (255 - foreground_alpha)) >> 8) + foreground_alpha;
            out[0]
                = u8cast((((in[0] * background_alpha * (255 - foreground_alpha)) >> 8) + (rcolor.x * foreground_alpha))
                         / blend_alpha);
            out[1]
                = u8cast((((in[1] * background_alpha * (255 - foreground_alpha)) >> 8) + (rcolor.y * foreground_alpha))
                         / blend_alpha);
            out[2]
                = u8cast((((in[2] * background_alpha * (255 - foreground_alpha)) >> 8) + (rcolor.z * foreground_alpha))
                         / blend_alpha);
        }
    }
}

template<class SrcWrapper, class DstWrapper, typename T = typename DstWrapper::ValueType>
static __device__ void blending_rgba_pixel(SrcWrapper src, DstWrapper dst, int x, int y, uchar4 plot_colors[4])
{
    const int batch_idx = get_batch_idx();

    for (int i = 0; i < 2; ++i)
    {
        T *in  = src.ptr(batch_idx, y + i, x, 0);
        T *out = dst.ptr(batch_idx, y + i, x, 0);
        for (int j = 0; j < 2; ++j, in += 4, out += 4)
        {
            uchar4 &rcolor           = plot_colors[i * 2 + j];
            int     foreground_alpha = rcolor.w;
            int     background_alpha = in[3];
            int     blend_alpha      = ((background_alpha * (255 - foreground_alpha)) >> 8) + foreground_alpha;
            out[0]
                = u8cast((((in[0] * background_alpha * (255 - foreground_alpha)) >> 8) + (rcolor.x * foreground_alpha))
                         / blend_alpha);
            out[1]
                = u8cast((((in[1] * background_alpha * (255 - foreground_alpha)) >> 8) + (rcolor.y * foreground_alpha))
                         / blend_alpha);
            out[2]
                = u8cast((((in[2] * background_alpha * (255 - foreground_alpha)) >> 8) + (rcolor.z * foreground_alpha))
                         / blend_alpha);
            out[3] = blend_alpha;
        }
    }
}

template<class SrcWrapper, class DstWrapper>
static __global__ void render_bndbox_rgb_womsaa_kernel(SrcWrapper src, DstWrapper dst, int bx, int by,
                                                       const RectangleCommand *commands, int num_command, int width,
                                                       int height, bool inplace)
{
    int ix = ((blockDim.x * blockIdx.x + threadIdx.x) << 1) + bx;
    int iy = ((blockDim.y * blockIdx.y + threadIdx.y) << 1) + by;
    if (ix < 0 || iy < 0 || ix >= width - 1 || iy >= height - 1)
        return;

    uchar4    context_color[4] = {0};
    const int batch_idx        = get_batch_idx();

    for (int i = 0; i < num_command; ++i)
    {
        RectangleCommand pcommand = commands[i];
        if (pcommand.batch_index != batch_idx)
            continue;
        do_rectangle_woMSAA(&pcommand, ix, iy, context_color);
    }

    if (context_color[0].w == 0 && context_color[1].w == 0 && context_color[2].w == 0 && context_color[3].w == 0)
    {
        if (inplace)
            return;
        *(uchar3 *)(dst.ptr(batch_idx, iy, ix, 0))         = *(uchar3 *)(src.ptr(batch_idx, iy, ix, 0));
        *(uchar3 *)(dst.ptr(batch_idx, iy, ix + 1, 0))     = *(uchar3 *)(src.ptr(batch_idx, iy, ix + 1, 0));
        *(uchar3 *)(dst.ptr(batch_idx, iy + 1, ix, 0))     = *(uchar3 *)(src.ptr(batch_idx, iy + 1, ix, 0));
        *(uchar3 *)(dst.ptr(batch_idx, iy + 1, ix + 1, 0)) = *(uchar3 *)(src.ptr(batch_idx, iy + 1, ix + 1, 0));
        return;
    }

    blending_rgb_pixel(src, dst, ix, iy, context_color);
}

template<class SrcWrapper, class DstWrapper>
static __global__ void render_bndbox_rgba_womsaa_kernel(SrcWrapper src, DstWrapper dst, int bx, int by,
                                                        const RectangleCommand *commands, int num_command, int width,
                                                        int height, bool inplace)
{
    int ix = ((blockDim.x * blockIdx.x + threadIdx.x) << 1) + bx;
    int iy = ((blockDim.y * blockIdx.y + threadIdx.y) << 1) + by;
    if (ix < 0 || iy < 0 || ix >= width - 1 || iy >= height - 1)
        return;

    uchar4    context_color[4] = {0};
    const int batch_idx        = get_batch_idx();

    for (int i = 0; i < num_command; ++i)
    {
        RectangleCommand pcommand = commands[i];
        if (pcommand.batch_index != batch_idx)
            continue;
        do_rectangle_woMSAA(&pcommand, ix, iy, context_color);
    }

    if (context_color[0].w == 0 && context_color[1].w == 0 && context_color[2].w == 0 && context_color[3].w == 0)
    {
        if (inplace)
            return;
        *(uchar4 *)(dst.ptr(batch_idx, iy, ix, 0))         = *(uchar4 *)(src.ptr(batch_idx, iy, ix, 0));
        *(uchar4 *)(dst.ptr(batch_idx, iy, ix + 1, 0))     = *(uchar4 *)(src.ptr(batch_idx, iy, ix + 1, 0));
        *(uchar4 *)(dst.ptr(batch_idx, iy + 1, ix, 0))     = *(uchar4 *)(src.ptr(batch_idx, iy + 1, ix, 0));
        *(uchar4 *)(dst.ptr(batch_idx, iy + 1, ix + 1, 0)) = *(uchar4 *)(src.ptr(batch_idx, iy + 1, ix + 1, 0));
        return;
    }

    blending_rgba_pixel(src, dst, ix, iy, context_color);
}

static ErrorCode cuosd_draw_rectangle(cuOSDContext_t context, int width, int height, NVCVBndBoxesI bboxes)
{
    for (int n = 0; n < bboxes.batch; n++)
    {
        auto numBoxes = bboxes.numBoxes[n];

        for (int i = 0; i < numBoxes; i++)
        {
            auto bbox   = bboxes.boxes[i];
            int  left   = max(min(bbox.box.x, width - 1), 0);
            int  top    = max(min(bbox.box.y, height - 1), 0);
            int  right  = max(min(left + bbox.box.width - 1, width - 1), 0);
            int  bottom = max(min(top + bbox.box.height - 1, height - 1), 0);

            if (left == right || top == bottom || bbox.box.width <= 0 || bbox.box.height <= 0)
            {
                LOG_DEBUG("Skipped bnd_box(" << bbox.box.x << ", " << bbox.box.y << ", " << bbox.box.width << ", "
                                             << bbox.box.height << ") in image(" << width << ", " << height << ")");
                continue;
            }

            if (bbox.borderColor.a == 0)
                continue;
            if (bbox.fillColor.a || bbox.thickness == -1)
            {
                if (bbox.thickness == -1)
                {
                    bbox.fillColor = bbox.borderColor;
                }

                auto cmd           = std::make_shared<RectangleCommand>();
                cmd->batch_index   = n;
                cmd->thickness     = -1;
                cmd->interpolation = false;
                cmd->c0            = bbox.fillColor.r;
                cmd->c1            = bbox.fillColor.g;
                cmd->c2            = bbox.fillColor.b;
                cmd->c3            = bbox.fillColor.a;

                // a   d
                // b   c
                cmd->ax1             = left;
                cmd->ay1             = top;
                cmd->dx1             = right;
                cmd->dy1             = top;
                cmd->cx1             = right;
                cmd->cy1             = bottom;
                cmd->bx1             = left;
                cmd->by1             = bottom;
                cmd->bounding_left   = left;
                cmd->bounding_right  = right;
                cmd->bounding_top    = top;
                cmd->bounding_bottom = bottom;
                context->rect_commands.emplace_back(cmd);
            }
            if (bbox.thickness == -1)
                continue;

            auto cmd           = std::make_shared<RectangleCommand>();
            cmd->batch_index   = n;
            cmd->thickness     = bbox.thickness;
            cmd->interpolation = false;
            cmd->c0            = bbox.borderColor.r;
            cmd->c1            = bbox.borderColor.g;
            cmd->c2            = bbox.borderColor.b;
            cmd->c3            = bbox.borderColor.a;

            float half_thickness = bbox.thickness / 2.0f;
            cmd->ax2             = left + half_thickness;
            cmd->ay2             = top + half_thickness;
            cmd->dx2             = right - half_thickness;
            cmd->dy2             = top + half_thickness;
            cmd->cx2             = right - half_thickness;
            cmd->cy2             = bottom - half_thickness;
            cmd->bx2             = left + half_thickness;
            cmd->by2             = bottom - half_thickness;

            // a   d
            // b   c
            cmd->ax1 = left - half_thickness;
            cmd->ay1 = top - half_thickness;
            cmd->dx1 = right + half_thickness;
            cmd->dy1 = top - half_thickness;
            cmd->cx1 = right + half_thickness;
            cmd->cy1 = bottom + half_thickness;
            cmd->bx1 = left - half_thickness;
            cmd->by1 = bottom + half_thickness;

            int int_half         = ceil(half_thickness);
            cmd->bounding_left   = left - int_half;
            cmd->bounding_right  = right + int_half;
            cmd->bounding_top    = top - int_half;
            cmd->bounding_bottom = bottom + int_half;
            context->rect_commands.emplace_back(cmd);
        }

        bboxes.boxes = (NVCVBndBoxI *)((uint8_t *)bboxes.boxes + numBoxes * sizeof(NVCVBndBoxI));
    }
    return ErrorCode::SUCCESS;
}

static void cuosd_apply(cuOSDContext_t context, int width, int height, hipStream_t stream)
{
    context->bounding_left   = width;
    context->bounding_top    = height;
    context->bounding_right  = 0;
    context->bounding_bottom = 0;

    for (int i = 0; i < (int)context->rect_commands.size(); ++i)
    {
        auto &cmd                = context->rect_commands[i];
        context->bounding_left   = min(context->bounding_left, cmd->bounding_left);
        context->bounding_top    = min(context->bounding_top, cmd->bounding_top);
        context->bounding_right  = max(context->bounding_right, cmd->bounding_right);
        context->bounding_bottom = max(context->bounding_bottom, cmd->bounding_bottom);
    }

    if (context->gpu_rect_commands == nullptr)
    {
        context->gpu_rect_commands.reset(new Memory<RectangleCommand>());
    }

    context->gpu_rect_commands->alloc_or_resize_to(context->rect_commands.size());

    for (int i = 0; i < (int)context->rect_commands.size(); ++i)
    {
        auto &cmd = context->rect_commands[i];
        memcpy((void *)(context->gpu_rect_commands->host() + i), cmd.get(), sizeof(RectangleCommand));
    }

    context->gpu_rect_commands->copy_host_to_device(stream);
}

inline ErrorCode ApplyBndBox_RGB(const nvcv::TensorDataStridedCuda &inData, const nvcv::TensorDataStridedCuda &outData,
                                 cuOSDContext_t context, hipStream_t stream)
{
    auto inAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);

    cuda_op::DataType  inDataType = helpers::GetLegacyDataType(inData.dtype());
    cuda_op::DataShape inputShape = helpers::GetLegacyDataShape(inAccess->infoShape());

    auto outAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    cuda_op::DataType  outDataType = helpers::GetLegacyDataType(outData.dtype());
    cuda_op::DataShape outputShape = helpers::GetLegacyDataShape(outAccess->infoShape());

    if (outDataType != inDataType)
    {
        LOG_ERROR("Unsupported input/output DataType " << inDataType << "/" << outDataType);
        return ErrorCode::INVALID_DATA_TYPE;
    }
    if (outputShape.H != inputShape.H || outputShape.W != inputShape.W || outputShape.N != inputShape.N
        || outputShape.C != inputShape.C || outputShape.C != 3)
    {
        LOG_ERROR("Invalid output shape " << outputShape);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    cuosd_apply(context, inputShape.W, inputShape.H, stream);

    dim3 blockSize(16, 8);
    dim3 gridSize(divUp(int((inputShape.W + 1) / 2), (int)blockSize.x),
                  divUp(int((inputShape.H + 1) / 2), (int)blockSize.y), inputShape.N);

    auto src = nvcv::cuda::CreateTensorWrapNHWC<uint8_t>(inData);
    auto dst = nvcv::cuda::CreateTensorWrapNHWC<uint8_t>(outData);

    render_bndbox_rgb_womsaa_kernel<<<gridSize, blockSize, 0, stream>>>(
        src, dst, 0, 0, context->gpu_rect_commands ? context->gpu_rect_commands->device() : nullptr,
        context->rect_commands.size(), inputShape.W, inputShape.H, inData.basePtr() == outData.basePtr());
    checkKernelErrors();

    return ErrorCode::SUCCESS;
}

inline ErrorCode ApplyBndBox_RGBA(const nvcv::TensorDataStridedCuda &inData, const nvcv::TensorDataStridedCuda &outData,
                                  cuOSDContext_t context, hipStream_t stream)
{
    auto inAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);

    cuda_op::DataType  inDataType = helpers::GetLegacyDataType(inData.dtype());
    cuda_op::DataShape inputShape = helpers::GetLegacyDataShape(inAccess->infoShape());

    auto outAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    cuda_op::DataType  outDataType = helpers::GetLegacyDataType(outData.dtype());
    cuda_op::DataShape outputShape = helpers::GetLegacyDataShape(outAccess->infoShape());

    if (outDataType != inDataType)
    {
        LOG_ERROR("Unsupported input/output DataType " << inDataType << "/" << outDataType);
        return ErrorCode::INVALID_DATA_TYPE;
    }
    if (outputShape.H != inputShape.H || outputShape.W != inputShape.W || outputShape.N != inputShape.N
        || outputShape.C != inputShape.C || outputShape.C != 4)
    {
        LOG_ERROR("Invalid output shape " << outputShape);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    cuosd_apply(context, inputShape.W, inputShape.H, stream);

    dim3 blockSize(16, 8);
    dim3 gridSize(divUp(int((inputShape.W + 1) / 2), (int)blockSize.x),
                  divUp(int((inputShape.H + 1) / 2), (int)blockSize.y), inputShape.N);

    auto src = nvcv::cuda::CreateTensorWrapNHWC<uint8_t>(inData);
    auto dst = nvcv::cuda::CreateTensorWrapNHWC<uint8_t>(outData);

    render_bndbox_rgba_womsaa_kernel<<<gridSize, blockSize, 0, stream>>>(
        src, dst, 0, 0, context->gpu_rect_commands ? context->gpu_rect_commands->device() : nullptr,
        context->rect_commands.size(), inputShape.W, inputShape.H, inData.basePtr() == outData.basePtr());
    checkKernelErrors();

    return ErrorCode::SUCCESS;
}

BndBox::BndBox(DataShape max_input_shape, DataShape max_output_shape)
    : CudaBaseOp(max_input_shape, max_output_shape)
{
    m_context = new cuOSDContext();
    if (m_context->gpu_rect_commands == nullptr)
    {
        m_context->gpu_rect_commands.reset(new Memory<RectangleCommand>());
    }
    m_context->gpu_rect_commands->alloc_or_resize_to(PREALLOC_CMD_NUM * sizeof(RectangleCommand));
}

BndBox::~BndBox()
{
    if (m_context)
    {
        m_context->rect_commands.clear();
        cuOSDContext *p = (cuOSDContext *)m_context;
        delete p;
    }
}

size_t BndBox::calBufferSize(DataShape max_input_shape, DataShape max_output_shape, DataType max_data_type)
{
    return 0;
}

ErrorCode BndBox::infer(const nvcv::TensorDataStridedCuda &inData, const nvcv::TensorDataStridedCuda &outData,
                        NVCVBndBoxesI bboxes, hipStream_t stream)
{
    cuda_op::DataFormat input_format  = GetLegacyDataFormat(inData.layout());
    cuda_op::DataFormat output_format = GetLegacyDataFormat(outData.layout());

    if (!(input_format == kNHWC || input_format == kHWC) || !(output_format == kNHWC || output_format == kHWC))
    {
        LOG_ERROR("Invliad DataFormat both Input and Output must be kNHWC or kHWC");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (inData.dtype() != outData.dtype())
    {
        LOG_ERROR("Input and Output formats must be same input format =" << inData.dtype()
                                                                         << " output format = " << outData.dtype());
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto inAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(inData);
    if (!inAccess)
    {
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    int batch    = inAccess->numSamples();
    int channels = inAccess->numChannels();
    int rows     = inAccess->numRows();
    int cols     = inAccess->numCols();

    if (channels > 4 || channels < 1)
    {
        LOG_ERROR("Invalid channel number ch = " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    if (bboxes.batch != batch)
    {
        LOG_ERROR("Invalid bboxes batch = " << bboxes.batch);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    auto outAccess = nvcv::TensorDataAccessStridedImagePlanar::Create(outData);
    if (!outAccess)
    {
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto ret = cuosd_draw_rectangle(m_context, cols, rows, bboxes);
    if (ret != ErrorCode::SUCCESS)
    {
        return ret;
    }

    typedef ErrorCode (*func_t)(const nvcv::TensorDataStridedCuda &inData, const nvcv::TensorDataStridedCuda &outData,
                                cuOSDContext_t context, hipStream_t stream);

    static const func_t funcs[] = {
        ApplyBndBox_RGB,
        ApplyBndBox_RGBA,
    };

    int type_idx = channels - 3;
    funcs[type_idx](inData, outData, m_context, stream);
    m_context->rect_commands.clear(); // Clear the command buffer so next render does not contain previous boxes.
    m_context->blur_commands.clear();
    return ErrorCode::SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
