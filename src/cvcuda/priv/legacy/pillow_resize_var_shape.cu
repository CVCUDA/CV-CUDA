#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

#include <nvcv/ImageBatch.hpp>

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;

#define BLOCK           32
#define SHARE_MEM_LIMIT 4096
#define work_type       float

namespace nvcv::legacy::cuda_op {

static constexpr float        bilinear_filter_support_var_shape = 1.;
static constexpr unsigned int precision_bits_var_shape          = 32 - 8 - 2;

namespace {

class BilinearFilterVarShape
{
public:
    __host__ __device__ BilinearFilterVarShape()
        : _support(bilinear_filter_support_var_shape){};

    __host__ __device__ work_type filter(work_type x)
    {
        if (x < 0.0)
        {
            x = -x;
        }
        if (x < 1.0)
        {
            return 1.0 - x;
        }
        return 0.0;
    }

    __host__ __device__ work_type support() const
    {
        return _support;
    };

private:
    work_type _support;
};

template<class Filter>
__global__ void _precomputeCoeffsVarShape(int *in_size_batch, int *in0_batch, work_type *scale_batch,
                                          work_type *filterscale_batch, work_type *support_batch, int *out_size_batch,
                                          int *k_size_batch, Filter filterp, int *bounds_out_batch,
                                          int *bound_out_offset, work_type *kk_out_batch, int *kk_out_offset,
                                          bool normalize_coeff, bool use_share_mem)
{
    const int xx       = blockIdx.x * blockDim.x + threadIdx.x;
    const int local_id = threadIdx.x;
    const int x_offset = blockIdx.x * blockDim.x;

    const int  batch_idx   = get_batch_idx();
    int        in_size     = in_size_batch[batch_idx];
    int        in0         = in0_batch[batch_idx];
    work_type  scale       = scale_batch[batch_idx];
    work_type  filterscale = filterscale_batch[batch_idx];
    work_type  support     = support_batch[batch_idx];
    int        out_size    = out_size_batch[batch_idx];
    int        k_size      = k_size_batch[batch_idx];
    int       *bounds_out  = bounds_out_batch + bound_out_offset[batch_idx];
    work_type *kk_out      = kk_out_batch + kk_out_offset[batch_idx];

    work_type *kk = kk_out + x_offset * k_size;
    if (use_share_mem)
    {
        extern __shared__ __align__(sizeof(work_type)) unsigned char smem_raw[];
        kk = reinterpret_cast<work_type *>(smem_raw);
    }

    if (xx < out_size)
    {
        int             x          = 0;
        int             xmin       = 0;
        int             xmax       = 0;
        work_type       center     = 0;
        work_type       ww         = 0;
        work_type       ss         = 0;
        const work_type half_pixel = 0.5;

        center = in0 + (xx + half_pixel) * scale;
        ww     = 0.0;
        ss     = 1.0 / filterscale;
        // Round the value.
        xmin = static_cast<int>(center - support + half_pixel);
        if (xmin < 0)
        {
            xmin = 0;
        }
        // Round the value.
        xmax = static_cast<int>(center + support + half_pixel);
        if (xmax > in_size)
        {
            xmax = in_size;
        }
        xmax -= xmin;
        work_type *k = &kk[local_id * k_size];
        for (x = 0; x < xmax; ++x)
        {
            work_type w = filterp.filter((x + xmin - center + half_pixel) * ss);
            k[x]        = w;
            ww += w;
        }
        for (x = 0; x < xmax; ++x)
        {
            if (std::fabs(ww) > 1e-5)
            {
                k[x] /= ww;
            }
        }
        // Remaining values should stay empty if they are used despite of xmax.
        for (; x < k_size; ++x)
        {
            k[x] = .0f;
        }
        if (normalize_coeff)
        {
            for (int i = 0; i < k_size; i++)
            {
                work_type val = k[i];
                if (val < 0)
                {
                    k[i] = static_cast<int>(-half_pixel + val * (1U << precision_bits_var_shape));
                }
                else
                {
                    k[i] = static_cast<int>(half_pixel + val * (1U << precision_bits_var_shape));
                }
            }
        }
        bounds_out[xx * 2]     = xmin;
        bounds_out[xx * 2 + 1] = xmax;
    }
    if (use_share_mem)
    {
        __syncthreads();
        for (int i = local_id; i < (out_size - x_offset) * k_size && i < blockDim.x * k_size; i += blockDim.x)
        {
            kk_out[x_offset * k_size + i] = kk[i];
        }
    }
}

template<class T1, class T2, class Filter>
__global__ void horizontal_pass_var_shape(const Ptr2dVarShapeNHWC<T1> src, Ptr2dNHWC<T2> dst, Filter &filterp,
                                          int *h_ksize_batch, int *v_ksize_batch, int *h_bounds_batch,
                                          int *h_bounds_offset, work_type *h_kk_batch, int *h_kk_offset,
                                          int *v_bounds_batch, int *v_bounds_offset, work_type *v_kk_batch,
                                          int *v_kk_offset, work_type init_buffer, bool round_up, bool use_share_mem)
{
    const int dst_x    = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y    = blockIdx.y * blockDim.y + threadIdx.y;
    const int local_x  = threadIdx.x;
    const int x_offset = blockIdx.x * blockDim.x;

    const int  batch_idx = get_batch_idx();
    int        h_ksize   = h_ksize_batch[batch_idx];
    int       *h_bounds  = h_bounds_batch + h_bounds_offset[batch_idx];
    work_type *h_kk      = h_kk_batch + h_kk_offset[batch_idx];

    int out_height = dst.at_rows(batch_idx), out_width = dst.at_cols(batch_idx);

    work_type *h_k_tmp = h_kk + x_offset * h_ksize;

    if (use_share_mem)
    {
        const int         local_tid = threadIdx.x + blockDim.x * threadIdx.y;
        extern __shared__ __align__(sizeof(work_type)) unsigned char kk_smem_h[];
        h_k_tmp = reinterpret_cast<work_type *>(kk_smem_h);

        for (int i = local_tid; i < blockDim.x * h_ksize && i < (out_width - x_offset) * h_ksize;
             i += blockDim.x * blockDim.y)
        {
            h_k_tmp[i] = h_kk[x_offset * h_ksize + i];
        }
        __syncthreads();
    }

    if (dst_x < out_width && dst_y < out_height)
    {
        int xmin = h_bounds[dst_x * 2];
        int xmax = h_bounds[dst_x * 2 + 1];

        work_type *h_k = &h_k_tmp[local_x * h_ksize];
        // int        offset_src = dst_y * src.at_cols(batch_idx) * src.nch + xmin * src.nch;
        // int        offset_dst = dst_y * dst.at_cols(batch_idx) * dst.nch + dst_x * dst.nch;
        for (int c = 0; c < src.nch; ++c)
        {
            work_type h_ss = 0.0;
            for (int x = 0; x < xmax; ++x)
            {
                // offset = offset_src + x * src.nch + c = (dst_y * src.at_cols(batch_idx) + xmin  + x) * src.nch + c
                h_ss = h_ss
                     + *src.ptr(batch_idx, dst_y + (xmin + x) / src.at_cols(batch_idx),
                                (xmin + x) % src.at_cols(batch_idx), c)
                           * h_k[x];
            }
            if (round_up)
                *dst.ptr(batch_idx, dst_y, dst_x, c) = cuda::SaturateCast<cuda::BaseType<T2>>(std::round(h_ss));
            else
                *dst.ptr(batch_idx, dst_y, dst_x, c) = cuda::SaturateCast<cuda::BaseType<T2>>(h_ss);
        }
    }
}

template<class T1, class T2, class Filter>
__global__ void vertical_pass_var_shape(const Ptr2dNHWC<T1> src, Ptr2dVarShapeNHWC<T2> dst, Filter &filterp,
                                        int *h_ksize_batch, int *v_ksize_batch, int *h_bounds_batch,
                                        int *h_bounds_offset, work_type *h_kk_batch, int *h_kk_offset,
                                        int *v_bounds_batch, int *v_bounds_offset, work_type *v_kk_batch,
                                        int *v_kk_offset, work_type init_buffer, bool round_up, bool use_share_mem)
{
    const int dst_x    = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y    = blockIdx.y * blockDim.y + threadIdx.y;
    const int local_y  = threadIdx.y;
    const int y_offset = blockIdx.y * blockDim.y;

    const int  batch_idx = get_batch_idx();
    int        v_ksize   = v_ksize_batch[batch_idx];
    int       *v_bounds  = v_bounds_batch + v_bounds_offset[batch_idx];
    work_type *v_kk      = v_kk_batch + v_kk_offset[batch_idx];

    int out_height = dst.at_rows(batch_idx), out_width = dst.at_cols(batch_idx);

    work_type *v_k_tmp = v_kk + y_offset * v_ksize;
    if (use_share_mem)
    {
        const int         local_tid = threadIdx.x + blockDim.x * threadIdx.y;
        extern __shared__ __align__(sizeof(work_type)) unsigned char kk_smem_v[];
        v_k_tmp = reinterpret_cast<work_type *>(kk_smem_v);

        for (int i = local_tid; i < blockDim.y * v_ksize && i < (out_height - y_offset) * v_ksize;
             i += blockDim.x * blockDim.y)
        {
            v_k_tmp[i] = v_kk[y_offset * v_ksize + i];
        }
        __syncthreads();
    }

    if (dst_x < out_width && dst_y < out_height)
    {
        int ymin = v_bounds[dst_y * 2];
        int ymax = v_bounds[dst_y * 2 + 1];

        work_type *v_k = &v_k_tmp[local_y * v_ksize];
        // int        offset_src     = ymin * src.at_cols(batch_idx) * src.nch + dst_x * src.nch;
        // int        col_offset_src = src.at_cols(batch_idx) * src.nch;
        // int        offset_dst     = dst_y * dst.at_cols(batch_idx) * dst.nch + dst_x * dst.nch;
        for (int c = 0; c < src.ch; ++c)
        {
            work_type ss = init_buffer;
            for (int y = 0; y < ymax; ++y)
            {
                // offset =  offset_src + y * col_offset_src + c = ((y + ymin)* src.at_cols(batch_idx) + dst_x) * src.nch + c
                ss = ss
                   + *src.ptr(batch_idx, y + ymin + (dst_x / src.at_cols(batch_idx)), dst_x % src.at_cols(batch_idx), c)
                         * v_k[y];
            }

            if (round_up)
                *dst.ptr(batch_idx, dst_y, dst_x, c) = cuda::SaturateCast<cuda::BaseType<T2>>(std::round(ss));
            else
                *dst.ptr(batch_idx, dst_y, dst_x, c) = cuda::SaturateCast<cuda::BaseType<T2>>(ss);
        }
    }
}

template<typename Filter, typename elem_type>
void pillow_resize_var_shape(const IImageBatchVarShape &inDataBase, const IImageBatchVarShape &outDataBase,
                             void *gpu_workspace, void *cpu_workspace, bool normalize_coeff, work_type init_buffer,
                             bool round_up, hipStream_t stream)
{
    auto *inDataPtr = dynamic_cast<const IImageBatchVarShapeDataStridedCuda *>(inDataBase.exportData(stream));
    if (inDataPtr == nullptr)
    {
        throw std::runtime_error("Something wrong happend during conversion of type...!!!");
    }

    auto *outDataPtr = dynamic_cast<const IImageBatchVarShapeDataStridedCuda *>(outDataBase.exportData(stream));
    if (outDataPtr == nullptr)
    {
        throw std::runtime_error("Something wrong happend during conversion of type...!!!");
    }

    const IImageBatchVarShapeDataStridedCuda &inData  = *inDataPtr;
    const IImageBatchVarShapeDataStridedCuda &outData = *outDataPtr;

    int channels = inData.uniqueFormat().numChannels();
    int batch    = inData.numImages();

    Filter filterp;

    Size2D outMaxSize = outData.maxSize();
    Size2D inMaxSize  = inData.maxSize();

    int max_height = outMaxSize.h, max_width = outMaxSize.w;
    int max_input_height = inMaxSize.h;

    const void **inputs              = (const void **)cpu_workspace;
    void       **outputs             = (void **)((char *)inputs + sizeof(void *) * batch);
    void       **hori                = (void **)((char *)outputs + sizeof(void *) * batch);
    int         *rows                = (int *)((char *)hori + sizeof(void *) * batch);
    int         *cols                = (int *)((char *)rows + sizeof(int) * batch);
    int         *out_rows            = (int *)((char *)cols + sizeof(int) * batch);
    int         *out_cols            = (int *)((char *)out_rows + sizeof(int) * batch);
    int         *roi_x               = (int *)((char *)out_cols + sizeof(int) * batch);
    int         *roi_y               = (int *)((char *)roi_x + sizeof(int) * batch);
    work_type   *h_scale_batch       = (work_type *)((char *)roi_y + sizeof(int) * batch);
    work_type   *v_scale_batch       = (work_type *)((char *)h_scale_batch + sizeof(work_type) * batch);
    work_type   *h_filterscale_batch = (work_type *)((char *)v_scale_batch + sizeof(work_type) * batch);
    work_type   *v_filterscale_batch = (work_type *)((char *)h_filterscale_batch + sizeof(work_type) * batch);
    work_type   *h_support_batch     = (work_type *)((char *)v_filterscale_batch + sizeof(work_type) * batch);
    work_type   *v_support_batch     = (work_type *)((char *)h_support_batch + sizeof(work_type) * batch);
    int         *h_k_size_batch      = (int *)((char *)v_support_batch + sizeof(work_type) * batch);
    int         *v_k_size_batch      = (int *)((char *)h_k_size_batch + sizeof(int) * batch);
    int         *h_bounds_offset     = (int *)((char *)v_k_size_batch + sizeof(int) * batch);
    int         *v_bounds_offset     = (int *)((char *)h_bounds_offset + sizeof(int) * batch);
    int         *h_kk_offset         = (int *)((char *)v_bounds_offset + sizeof(int) * batch);
    int         *v_kk_offset         = (int *)((char *)h_kk_offset + sizeof(int) * batch);

    int h_kk_total = 0, v_kk_total = 0;
    int max_h_k_size = 0, max_v_k_size = 0;
    int h_bounds_total = 0, v_bounds_total = 0;

    for (int i = 0; i < batch; i++)
    {
        rows[i]     = inDataBase[i].size().h;
        cols[i]     = inDataBase[i].size().w;
        out_rows[i] = outDataBase[i].size().h;
        out_cols[i] = outDataBase[i].size().w;

        roi_x[i] = 0;
        roi_y[i] = 0;

        work_type h_scale = 0, v_scale = 0;
        work_type h_filterscale = 0, v_filterscale = 0;
        h_filterscale = h_scale = static_cast<work_type>(inDataBase[i].size().w) / out_cols[i];
        v_filterscale = v_scale = static_cast<work_type>(inDataBase[i].size().h) / out_rows[i];
        if (h_filterscale < 1.0)
        {
            h_filterscale = 1.0;
        }
        if (v_filterscale < 1.0)
        {
            v_filterscale = 1.0;
        }
        h_scale_batch[i]       = h_scale;
        v_scale_batch[i]       = v_scale;
        h_filterscale_batch[i] = h_filterscale;
        v_filterscale_batch[i] = v_filterscale;

        // Determine support size (length of resampling filter).
        work_type h_support = filterp.support() * h_filterscale;
        work_type v_support = filterp.support() * v_filterscale;
        // Maximum number of coeffs.
        int       h_k_size = static_cast<int>(ceil(h_support)) * 2 + 1;
        int       v_k_size = static_cast<int>(ceil(v_support)) * 2 + 1;
        h_support_batch[i] = h_support;
        v_support_batch[i] = v_support;
        h_k_size_batch[i]  = h_k_size;
        v_k_size_batch[i]  = v_k_size;
        h_kk_offset[i]     = h_kk_total;
        v_kk_offset[i]     = v_kk_total;
        h_kk_total += out_cols[i] * h_k_size;
        v_kk_total += out_rows[i] * v_k_size;
        h_bounds_offset[i] = h_bounds_total;
        v_bounds_offset[i] = v_bounds_total;
        h_bounds_total += out_cols[i] * 2;
        v_bounds_total += out_rows[i] * 2;

        if (h_k_size > max_h_k_size)
            max_h_k_size = h_k_size;
        if (v_k_size > max_v_k_size)
            max_v_k_size = v_k_size;
    }

    const void **inputs_gpu              = (const void **)gpu_workspace;
    void       **outputs_gpu             = (void **)((char *)inputs_gpu + sizeof(void *) * batch);
    void       **hori_gpu                = (void **)((char *)outputs_gpu + sizeof(void *) * batch);
    int         *rows_gpu                = (int *)((char *)hori_gpu + sizeof(void *) * batch);
    int         *cols_gpu                = (int *)((char *)rows_gpu + sizeof(int) * batch);
    int         *out_rows_gpu            = (int *)((char *)cols_gpu + sizeof(int) * batch);
    int         *out_cols_gpu            = (int *)((char *)out_rows_gpu + sizeof(int) * batch);
    int         *roi_x_gpu               = (int *)((char *)out_cols_gpu + sizeof(int) * batch);
    int         *roi_y_gpu               = (int *)((char *)roi_x_gpu + sizeof(int) * batch);
    work_type   *h_scale_batch_gpu       = (work_type *)((char *)roi_y_gpu + sizeof(int) * batch);
    work_type   *v_scale_batch_gpu       = (work_type *)((char *)h_scale_batch_gpu + sizeof(work_type) * batch);
    work_type   *h_filterscale_batch_gpu = (work_type *)((char *)v_scale_batch_gpu + sizeof(work_type) * batch);
    work_type   *v_filterscale_batch_gpu = (work_type *)((char *)h_filterscale_batch_gpu + sizeof(work_type) * batch);
    work_type   *h_support_batch_gpu     = (work_type *)((char *)v_filterscale_batch_gpu + sizeof(work_type) * batch);
    work_type   *v_support_batch_gpu     = (work_type *)((char *)h_support_batch_gpu + sizeof(work_type) * batch);
    int         *h_k_size_batch_gpu      = (int *)((char *)v_support_batch_gpu + sizeof(work_type) * batch);
    int         *v_k_size_batch_gpu      = (int *)((char *)h_k_size_batch_gpu + sizeof(int) * batch);
    int         *h_bounds_offset_gpu     = (int *)((char *)v_k_size_batch_gpu + sizeof(int) * batch);
    int         *v_bounds_offset_gpu     = (int *)((char *)h_bounds_offset_gpu + sizeof(int) * batch);
    int         *h_kk_offset_gpu         = (int *)((char *)v_bounds_offset_gpu + sizeof(int) * batch);
    int         *v_kk_offset_gpu         = (int *)((char *)h_kk_offset_gpu + sizeof(int) * batch);

    work_type *h_kk_batch_gpu     = (work_type *)((char *)v_kk_offset_gpu + sizeof(int) * batch);
    work_type *v_kk_batch_gpu     = (work_type *)((char *)h_kk_batch_gpu + sizeof(work_type) * h_kk_total);
    int       *h_bounds_batch_gpu = (int *)((char *)v_kk_batch_gpu + sizeof(work_type) * v_kk_total);
    int       *v_bounds_batch_gpu = (int *)((char *)h_bounds_batch_gpu + sizeof(int) * h_bounds_total);

    int current_buffer_size = (sizeof(void *) * 3 + sizeof(int) * 12 + sizeof(work_type) * 6) * batch
                            + sizeof(work_type) * (h_kk_total + v_kk_total)
                            + sizeof(int) * (h_bounds_total + v_bounds_total);

    // buffer for storing results from horizontal pass
    void *hori_gpu_data = (void *)((char *)gpu_workspace + current_buffer_size);

    checkCudaErrors(hipMemcpyAsync((void *)gpu_workspace, (void *)cpu_workspace, current_buffer_size,
                                    hipMemcpyHostToDevice, stream));

    Ptr2dVarShapeNHWC<elem_type> src_ptr(inData);
    Ptr2dVarShapeNHWC<elem_type> dst_ptr(outData);
    Ptr2dNHWC<work_type>         ptr_h_out(batch, max_input_height, max_width, channels, (work_type *)hori_gpu_data);

    dim3 blockSize(BLOCK, BLOCK / 4, 1);
    dim3 gridSizeH(divUp(max_width, blockSize.x), divUp(max_input_height, blockSize.y), batch);
    dim3 gridSizeV(divUp(max_width, blockSize.x), divUp(max_height, blockSize.y), batch);

    dim3 coef_block(BLOCK * 2, 1, 1);
    dim3 h_coef_grid(divUp(max_width, coef_block.x), 1, batch);
    dim3 v_coef_grid(divUp(max_height, coef_block.x), 1, batch);

    size_t h_sm_size       = coef_block.x * (max_h_k_size * sizeof(work_type));
    size_t v_sm_size       = coef_block.x * (max_v_k_size * sizeof(work_type));
    size_t hv_sm_size1     = max_h_k_size * sizeof(work_type) * blockSize.x;
    size_t hv_sm_size2     = max_v_k_size * sizeof(work_type) * blockSize.y;
    bool   h_use_share_mem = h_sm_size <= SHARE_MEM_LIMIT;
    if (!h_use_share_mem)
    {
        h_sm_size = 0;
    }
    bool v_use_share_mem = v_sm_size <= SHARE_MEM_LIMIT;
    if (!v_use_share_mem)
    {
        v_sm_size = 0;
    }
    bool hv_use_share_mem = (hv_sm_size1 <= SHARE_MEM_LIMIT) && (hv_sm_size2 <= SHARE_MEM_LIMIT);
    if (!hv_use_share_mem)
    {
        hv_sm_size1 = 0;
        hv_sm_size2 = 0;
    }

    // compute horizontal coef
    _precomputeCoeffsVarShape<Filter><<<h_coef_grid, coef_block, h_sm_size, stream>>>(
        cols_gpu, roi_x_gpu, h_scale_batch_gpu, h_filterscale_batch_gpu, h_support_batch_gpu, out_cols_gpu,
        h_k_size_batch_gpu, filterp, h_bounds_batch_gpu, h_bounds_offset_gpu, h_kk_batch_gpu, h_kk_offset_gpu,
        normalize_coeff, h_use_share_mem);

    checkKernelErrors();
    // checkCudaErrors(hipStreamSynchronize(stream));
    // compute vertical coef
    _precomputeCoeffsVarShape<Filter><<<v_coef_grid, coef_block, v_sm_size, stream>>>(
        rows_gpu, roi_y_gpu, v_scale_batch_gpu, v_filterscale_batch_gpu, v_support_batch_gpu, out_rows_gpu,
        v_k_size_batch_gpu, filterp, v_bounds_batch_gpu, v_bounds_offset_gpu, v_kk_batch_gpu, v_kk_offset_gpu,
        normalize_coeff, v_use_share_mem);
    checkKernelErrors();
    // checkCudaErrors(hipStreamSynchronize(stream));
    horizontal_pass_var_shape<elem_type, work_type, Filter><<<gridSizeH, blockSize, hv_sm_size1, stream>>>(
        src_ptr, ptr_h_out, filterp, h_k_size_batch_gpu, v_k_size_batch_gpu, h_bounds_batch_gpu, h_bounds_offset_gpu,
        h_kk_batch_gpu, h_kk_offset_gpu, v_bounds_batch_gpu, v_bounds_offset_gpu, v_kk_batch_gpu, v_kk_offset_gpu,
        init_buffer, round_up, hv_use_share_mem);
    checkKernelErrors();
    // checkCudaErrors(hipStreamSynchronize(stream));
    vertical_pass_var_shape<work_type, elem_type, Filter><<<gridSizeV, blockSize, hv_sm_size2, stream>>>(
        ptr_h_out, dst_ptr, filterp, h_k_size_batch_gpu, v_k_size_batch_gpu, h_bounds_batch_gpu, h_bounds_offset_gpu,
        h_kk_batch_gpu, h_kk_offset_gpu, v_bounds_batch_gpu, v_bounds_offset_gpu, v_kk_batch_gpu, v_kk_offset_gpu,
        init_buffer, round_up, hv_use_share_mem);

    checkKernelErrors();
}

} // namespace

template<typename Filter>
void pillow_resize_filter_var_shape(const IImageBatchVarShape &inData, const IImageBatchVarShape &outData,
                                    void *gpu_workspace, void *cpu_workspace, NVCVInterpolationType interpolation,
                                    hipStream_t stream)
{
    DataType data_type = helpers::GetLegacyDataType(inData.uniqueFormat());
    switch (data_type)
    {
    case kCV_8U:
        pillow_resize_var_shape<Filter, unsigned char>(inData, outData, gpu_workspace, cpu_workspace, false, 0., false,
                                                       stream);
        break;
    case kCV_8S:
        pillow_resize_var_shape<Filter, signed char>(inData, outData, gpu_workspace, cpu_workspace, false, 0., true,
                                                     stream);
        break;
    case kCV_16U:
        pillow_resize_var_shape<Filter, std::uint16_t>(inData, outData, gpu_workspace, cpu_workspace, false, 0., false,
                                                       stream);
        break;
    case kCV_16S:
        pillow_resize_var_shape<Filter, std::int16_t>(inData, outData, gpu_workspace, cpu_workspace, false, 0., true,
                                                      stream);
        break;
    case kCV_32S:
        pillow_resize_var_shape<Filter, int>(inData, outData, gpu_workspace, cpu_workspace, false, 0., true, stream);
        break;
    case kCV_32F:
        pillow_resize_var_shape<Filter, float>(inData, outData, gpu_workspace, cpu_workspace, false, 0., false, stream);
        break;
    case kCV_64F:
    default:
        break;
    }
}

PillowResizeVarShape::PillowResizeVarShape(DataShape max_input_shape, DataShape max_output_shape,
                                           DataType max_data_type)
    : CudaBaseOp(max_input_shape, max_output_shape)
{
    int    max_support = 1; //3
    size_t size        = std::ceil(
               max_output_shape.H
                   * (((1.0 * max_input_shape.H / max_output_shape.H + 1) * max_support * 2 + 1) * sizeof(work_type)
               + 2 * sizeof(int))
               + max_output_shape.W
                     * (((1.0 * max_input_shape.W / max_output_shape.W + 1) * max_support * 2 + 1) * sizeof(work_type)
                 + 2 * sizeof(int)));
    size_t buffer_size = (sizeof(void *) * 3 + sizeof(int) * 12 + sizeof(work_type) * 6 + size) * max_input_shape.N;
    buffer_size += max_input_shape.N * max_input_shape.C * max_input_shape.H * max_output_shape.W * sizeof(float);

    NVCV_CHECK_LOG(hipMalloc(&gpu_workspace, buffer_size));

    cpu_workspace = malloc(buffer_size);
    if (!cpu_workspace)
    {
        LOG_ERROR("Memory allocation error of size: " << buffer_size);
        throw std::runtime_error("Memory allocation error!");
    }
}

PillowResizeVarShape::~PillowResizeVarShape()
{
    NVCV_CHECK_LOG(hipFree(gpu_workspace));
    free(cpu_workspace);
}

size_t PillowResizeVarShape::calBufferSize(DataShape max_input_shape, DataShape max_output_shape,
                                           DataType max_data_type)
{
    int    max_support = 1; //3
    size_t size        = std::ceil(
               max_output_shape.H
                   * (((1.0 * max_input_shape.H / max_output_shape.H + 1) * max_support * 2 + 1) * sizeof(work_type)
               + 2 * sizeof(int))
               + max_output_shape.W
                     * (((1.0 * max_input_shape.W / max_output_shape.W + 1) * max_support * 2 + 1) * sizeof(work_type)
                 + 2 * sizeof(int)));
    size_t buffer_size = (sizeof(void *) * 3 + sizeof(int) * 12 + sizeof(work_type) * 6 + size) * max_input_shape.N;
    buffer_size += max_input_shape.N * max_input_shape.C * max_input_shape.H * max_output_shape.W * sizeof(float);

    return buffer_size;
}

ErrorCode PillowResizeVarShape::infer(const nvcv::IImageBatchVarShape &inDataBase,
                                      const nvcv::IImageBatchVarShape &outDataBase,
                                      const NVCVInterpolationType interpolation, hipStream_t stream)
{
    if (!inDataBase.uniqueFormat() || !outDataBase.uniqueFormat())
    {
        LOG_ERROR("Images in input and outut batch must all have the same format");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (inDataBase.uniqueFormat() != outDataBase.uniqueFormat())
    {
        LOG_ERROR("Invalid DataFormat between input and output");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataFormat format = GetLegacyDataFormat(inDataBase);

    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    int channels = inDataBase.uniqueFormat().numChannels();

    if (channels > 4)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    DataType data_type = helpers::GetLegacyDataType(inDataBase.uniqueFormat());

    if (!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_16S || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    if (!(interpolation == NVCV_INTERP_LINEAR))
    {
        LOG_ERROR("Unsupported interpolation method " << interpolation);
        return ErrorCode::INVALID_PARAMETER;
    }

    switch (interpolation)
    {
    case NVCV_INTERP_LINEAR:
        pillow_resize_filter_var_shape<BilinearFilterVarShape>(inDataBase, outDataBase, gpu_workspace, cpu_workspace,
                                                               interpolation, stream);
        break;
    default:
        break;
    }
    return ErrorCode::SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
