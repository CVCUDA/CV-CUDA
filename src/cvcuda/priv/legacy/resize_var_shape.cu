#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
 * Copyright (C) 2009-2010, Willow Garage Inc., all rights reserved.
 * Copyright (C) 2014-2015, Itseez Inc., all rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

#include <nvcv/cuda/MathWrappers.hpp>
#include <nvcv/cuda/SaturateCast.hpp>

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;

namespace nvcv::legacy::cuda_op {

namespace {

#define MAX_BUFFER_BYTES_VS 128 //multiple of 4 for word-aligned read, multiple of 16 for cacheline alignment (float4)
#define MAX_BUFFER_WORDS_VS (MAX_BUFFER_BYTES_VS / 4) //extra bytes for cache alignment

#define LEGACY_BICUBIC_MATH_VS //apparently the legacy code has an abs() that needs to be matched

// Replaced below 15 to 0 due to a reported regression
#define CACHE_MEMORY_ALIGNMENT_VS 0 //this is 'M' for _cacheAlignedBufferedReadVS

//legal values for CACHE_MEMORY_ALIGNMENT_VS are:
// 31: 256-bit alignment
// 15: 128-bit alignment <-- should be ideal for Ampere
//  7:  64-bit alignment
//  3:  32-bit alignment (word)
//  0:  disable buffering
template<typename T, size_t M>
inline __device__ T *_cacheAlignedBufferedReadVS(cuda::ImageBatchVarShapeWrap<const T> srcImage, int width,
                                                 uint *pReadBuffer, uint nReadBufferWordsMax, int nBatch, int nYPos,
                                                 int nXPosMin, int nXPosMax)
{
    const T *lineStartPtr = srcImage.ptr(nBatch, nYPos, 0); //do not access prior to this address
    const T *pixSrcPtr    = &lineStartPtr[nXPosMin];
    if (M == 0)
        return (T *)pixSrcPtr; //return GMEM pointer instead
    else
    {
        uint     *memSrcPtr       = (uint *)(((size_t)pixSrcPtr) & (~M)); //(M+1) byte alignment
        const T  *pixBeyondPtr    = &lineStartPtr[nXPosMax + 1];
        const int functionalWidth = ((size_t)pixBeyondPtr + M) & (~M) - ((size_t)lineStartPtr);
        const int nWordsToRead    = (((size_t)pixBeyondPtr + M) & (~M) - (size_t)memSrcPtr) / 4;

        if (((size_t)memSrcPtr < (size_t)lineStartPtr) || (width * sizeof(T) < functionalWidth)
            || (nWordsToRead > nReadBufferWordsMax))
            return (T *)pixSrcPtr; //return GMEM pointer instead if running off the image
        else
        {                                             //copy out source data, aligned based upon M (31, 15, 7, 3)
            const int skew = ((size_t)pixSrcPtr) & M; //byte offset for nXPosMin
            int       i    = 0;
            if (M >= 31) //256-bit align, 32 bytes at a time
                for (; i < nWordsToRead; i += 8) *((double4 *)(&pReadBuffer[i])) = *((double4 *)(&memSrcPtr[i]));
            if (M == 15) //128-bit align, 16 bytes at a time
                for (; i < nWordsToRead; i += 4) *((float4 *)(&pReadBuffer[i])) = *((float4 *)(&memSrcPtr[i]));
            if (M == 7) //64-bit align, 8 bytes at a time
                for (; i < nWordsToRead; i += 2) *((float2 *)(&pReadBuffer[i])) = *((float2 *)(&memSrcPtr[i]));
            //32-bit align, 4 bytes at a time
            for (; i < nWordsToRead; ++i) pReadBuffer[i] = memSrcPtr[i];

            return (T *)(((size_t)pReadBuffer) + skew); //buffered pixel data
        }
    }
} //_cacheAlignedBufferedReadVS

//******************** NN = Nearest Neighbor

template<typename T>
__global__ void resize_NN(cuda::ImageBatchVarShapeWrap<const T> src, cuda::ImageBatchVarShapeWrap<T> dst)
{
    const int dst_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();
    const int dstWidth  = dst.width(batch_idx);
    const int dstHeight = dst.height(batch_idx);

    if ((dst_x < dstWidth) && (dst_y < dstHeight))
    { //generic copy pixel to pixel
        const int width  = src.width(batch_idx);
        const int height = src.height(batch_idx);

        const float scale_x = static_cast<float>(width) / dstWidth;
        const float scale_y = static_cast<float>(height) / dstHeight;
        const int   sx      = cuda::min(__float2int_rd((dst_x + 0.5f) * scale_x), width - 1);
        const int   sy      = cuda::min(__float2int_rd((dst_y + 0.5f) * scale_y), height - 1);

        *dst.ptr(batch_idx, dst_y, dst_x) = *src.ptr(batch_idx, sy, sx);
    }
} //resize_NN

//******************** Bilinear

template<typename T>
__global__ void resize_bilinear(cuda::ImageBatchVarShapeWrap<const T> src, cuda::ImageBatchVarShapeWrap<T> dst)
{
    const int dst_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();
    const int dstWidth  = dst.width(batch_idx);
    const int dstHeight = dst.height(batch_idx);

    if ((dst_x < dstWidth) && (dst_y < dstHeight))
    {
        const int width  = src.width(batch_idx);
        const int height = src.height(batch_idx);

        const float scale_x = static_cast<float>(width) / dstWidth;
        const float scale_y = static_cast<float>(height) / dstHeight;

        //float space for weighted addition
        using work_type = cuda::ConvertBaseTypeTo<float, T>;

        //y coordinate
        float fy = (float)((dst_y + 0.5f) * scale_y - 0.5f);
        int   sy = cuda::round<cuda::RoundMode::DOWN, int>(fy);
        fy -= sy;
        sy = cuda::max(0, cuda::min(sy, height - 2));

        //row pointers
        const T *aPtr = src.ptr(batch_idx, sy, 0);     //start of upper row
        const T *bPtr = src.ptr(batch_idx, sy + 1, 0); //start of lower row

        { //compute source data position and weight for [x0] components
            float fx = (float)((dst_x + 0.5f) * scale_x - 0.5f);
            int   sx = cuda::round<cuda::RoundMode::DOWN, int>(fx);
            fx -= sx;
            fx *= ((sx >= 0) && (sx < width - 1));
            sx = cuda::max(0, cuda::min(sx, width - 2));

            *dst.ptr(batch_idx, dst_y, dst_x)
                = cuda::SaturateCast<T>((1.0f - fx) * (aPtr[sx] * (1.0f - fy) + bPtr[sx] * fy)
                                        + fx * (aPtr[sx + 1] * (1.0f - fy) + bPtr[sx + 1] * fy));
        }
    }
} //resize_bilinear

//******************** Bicubic

template<typename T>
__global__ void resize_bicubic(cuda::ImageBatchVarShapeWrap<const T> src, cuda::ImageBatchVarShapeWrap<T> dst)
{ //optimized for aligned read
    const int dst_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();
    const int dstWidth  = dst.width(batch_idx);
    const int dstHeight = dst.height(batch_idx);

    if ((dst_x < dstWidth) & (dst_y < dstHeight))
    {
        const int width  = src.width(batch_idx);
        const int height = src.height(batch_idx);

        const float scale_x = static_cast<float>(width) / dstWidth;
        const float scale_y = static_cast<float>(height) / dstHeight;

        //float space for weighted addition
        using work_type = cuda::ConvertBaseTypeTo<float, T>;

        uint readBuffer[MAX_BUFFER_WORDS_VS];

        //y coordinate
        float fy = (float)((dst_y + 0.5f) * scale_y - 0.5f);
        int   sy = cuda::round<cuda::RoundMode::DOWN, int>(fy);
        fy -= sy;
        sy = cuda::max(1, cuda::min(sy, height - 3));

        const float A = -0.75f;

        float cY[4];
        cY[0] = ((A * (fy + 1) - 5 * A) * (fy + 1) + 8 * A) * (fy + 1) - 4 * A;
        cY[1] = ((A + 2) * fy - (A + 3)) * fy * fy + 1;
        cY[2] = ((A + 2) * (1 - fy) - (A + 3)) * (1 - fy) * (1 - fy) + 1;
        cY[3] = 1.f - cY[0] - cY[1] - cY[2];

        work_type accum = cuda::SetAll<work_type>(0);

        float fx = (float)((dst_x + 0.5f) * scale_x - 0.5f);
        int   sx = cuda::round<cuda::RoundMode::DOWN, int>(fx);
        fx -= sx;
        fx *= ((sx >= 1) && (sx < width - 3));
        sx = cuda::max(1, cuda::min(sx, width - 3));

        float cX[4];
        cX[0] = ((A * (fx + 1.0f) - 5.0f * A) * (fx + 1.0f) + 8.0f * A) * (fx + 1.0f) - 4.0f * A;
        cX[1] = ((A + 2.0f) * fx - (A + 3.0f)) * fx * fx + 1.0f;
        cX[2] = ((A + 2.0f) * (1.0f - fx) - (A + 3.0f)) * (1.0f - fx) * (1.0f - fx) + 1.0f;
        cX[3] = 1.0f - cX[0] - cX[1] - cX[2];
#pragma unroll
        for (int row = 0; row < 4; ++row)
        {
            //1 - load each sub row from sx-1 to sx+3 inclusive, aligned
            //const T * aPtr = src.ptr(batch_idx, sy + row - 1, sx-1);
            T *aPtr = _cacheAlignedBufferedReadVS<T, CACHE_MEMORY_ALIGNMENT_VS>(
                src, width, readBuffer, MAX_BUFFER_WORDS_VS, batch_idx, sy + row - 1, sx - 1, sx + 2);

            //2 - do a pixel's partial on this row
            accum += cY[row] * (cX[0] * aPtr[0] + cX[1] * aPtr[1] + cX[2] * aPtr[2] + cX[3] * aPtr[3]);
        } //for row
#ifndef LEGACY_BICUBIC_MATH_VS
        //correct math
        *dst.ptr(batch_idx, dst_y, dst_x) = cuda::SaturateCast<T>(accum);
#else
        //abs() needed to match legacy operator.
        *dst.ptr(batch_idx, dst_y, dst_x) = cuda::SaturateCast<T>(cuda::abs(accum));
#endif
    }
} //resize_bicubic

//******************** Integrate area

template<typename T>
__global__ void resize_area_ocv_align(const cuda::ImageBatchVarShapeWrap<const T>                   src,
                                      const cuda::BorderVarShapeWrap<const T, NVCV_BORDER_CONSTANT> brd_src,
                                      cuda::ImageBatchVarShapeWrap<T>                               dst)
{
    const int x         = blockDim.x * blockIdx.x + threadIdx.x;
    const int y         = blockDim.y * blockIdx.y + threadIdx.y;
    const int batch_idx = get_batch_idx();

    int dstWidth  = dst.width(batch_idx);
    int dstHeight = dst.height(batch_idx);

    if (x >= dstWidth || y >= dstHeight)
        return;
    int height = src.height(batch_idx), width = src.width(batch_idx);

    float scale_x = static_cast<float>(width) / dstWidth;
    float scale_y = static_cast<float>(height) / dstHeight;

    double inv_scale_x  = 1. / scale_x;
    double inv_scale_y  = 1. / scale_y;
    int    iscale_x     = cuda::SaturateCast<int>(scale_x);
    int    iscale_y     = cuda::SaturateCast<int>(scale_y);
    bool   is_area_fast = abs(scale_x - iscale_x) < DBL_EPSILON && abs(scale_y - iscale_y) < DBL_EPSILON;

    if (scale_x >= 1.0f && scale_y >= 1.0f) // zoom out
    {
        if (is_area_fast) // integer multiples
        {
            float scale = 1.f / (scale_x * scale_y);
            float fsx1  = x * scale_x;
            float fsx2  = fsx1 + scale_x;

            int sx1 = cuda::round<cuda::RoundMode::UP, int>(fsx1);
            int sx2 = cuda::round<cuda::RoundMode::DOWN, int>(fsx2);

            float fsy1 = y * scale_y;
            float fsy2 = fsy1 + scale_y;

            int sy1 = cuda::round<cuda::RoundMode::UP, int>(fsy1);
            int sy2 = cuda::round<cuda::RoundMode::DOWN, int>(fsy2);

            using work_type = cuda::ConvertBaseTypeTo<float, T>;
            work_type out   = {0};

            int3 srcCoord = {0, 0, batch_idx};

            for (int dy = sy1; dy < sy2; ++dy)
            {
                srcCoord.y = dy;

                for (int dx = sx1; dx < sx2; ++dx)
                {
                    srcCoord.x = dx;

                    out = out + brd_src[srcCoord] * scale;
                }
            }
            *dst.ptr(batch_idx, y, x) = cuda::SaturateCast<T>(out);
            return;
        }

        float fsx1 = x * scale_x;
        float fsx2 = fsx1 + scale_x;

        int sx1 = cuda::round<cuda::RoundMode::UP, int>(fsx1);
        int sx2 = cuda::round<cuda::RoundMode::DOWN, int>(fsx2);

        float fsy1 = y * scale_y;
        float fsy2 = fsy1 + scale_y;

        int sy1 = cuda::round<cuda::RoundMode::UP, int>(fsy1);
        int sy2 = cuda::round<cuda::RoundMode::DOWN, int>(fsy2);

        float scale
            = 1.f / (fminf(scale_x, src.width(batch_idx) - fsx1) * fminf(scale_y, src.height(batch_idx) - fsy1));

        using work_type = cuda::ConvertBaseTypeTo<float, T>;
        work_type out   = {0};

        int3 srcCoord = {0, 0, batch_idx};

        for (int dy = sy1; dy < sy2; ++dy)
        {
            srcCoord.y = dy;

            for (int dx = sx1; dx < sx2; ++dx)
            {
                srcCoord.x = dx;

                out = out + brd_src[srcCoord] * scale;
            }

            if (sx1 > fsx1)
            {
                srcCoord.x = sx1 - 1;
                out        = out + brd_src[srcCoord] * ((sx1 - fsx1) * scale);
            }

            if (sx2 < fsx2)
            {
                srcCoord.x = sx2;
                out        = out + brd_src[srcCoord] * ((fsx2 - sx2) * scale);
            }
        }

        if (sy1 > fsy1)
        {
            srcCoord.y = sy1 - 1;
            for (int dx = sx1; dx < sx2; ++dx)
            {
                srcCoord.x = dx;
                out        = out + brd_src[srcCoord] * ((sy1 - fsy1) * scale);
            }
        }

        if (sy2 < fsy2)
        {
            srcCoord.y = sy2;
            for (int dx = sx1; dx < sx2; ++dx)
            {
                srcCoord.x = dx;
                out        = out + brd_src[srcCoord] * ((fsy2 - sy2) * scale);
            }
        }

        if ((sy1 > fsy1) && (sx1 > fsx1))
        {
            srcCoord.y = (sy1 - 1);
            srcCoord.x = (sx1 - 1);
            out        = out + brd_src[srcCoord] * ((sy1 - fsy1) * (sx1 - fsx1) * scale);
        }

        if ((sy1 > fsy1) && (sx2 < fsx2))
        {
            srcCoord.y = (sy1 - 1);
            srcCoord.x = sx2;
            out        = out + brd_src[srcCoord] * ((sy1 - fsy1) * (fsx2 - sx2) * scale);
        }

        if ((sy2 < fsy2) && (sx2 < fsx2))
        {
            srcCoord.y = sy2;
            srcCoord.x = sx2;
            out        = out + brd_src[srcCoord] * ((fsy2 - sy2) * (fsx2 - sx2) * scale);
        }

        if ((sy2 < fsy2) && (sx1 > fsx1))
        {
            srcCoord.y = sy2;
            srcCoord.x = sx1 - 1;
            out        = out + brd_src[srcCoord] * ((fsy2 - sy2) * (sx1 - fsx1) * scale);
        }

        *dst.ptr(batch_idx, y, x) = cuda::SaturateCast<T>(out);
        return;
    }

    // zoom in, it is emulated using some variant of bilinear interpolation
    int   sy = cuda::round<cuda::RoundMode::DOWN, int>(y * scale_y);
    float fy = (float)((y + 1) - (sy + 1) * inv_scale_y);
    fy       = fy <= 0 ? 0.f : fy - cuda::round<cuda::RoundMode::DOWN, int>(fy);

    float cbufy[2];
    cbufy[0] = 1.f - fy;
    cbufy[1] = fy;

    int   sx = cuda::round<cuda::RoundMode::DOWN, int>(x * scale_x);
    float fx = (float)((x + 1) - (sx + 1) * inv_scale_x);
    fx       = fx < 0 ? 0.f : fx - cuda::round<cuda::RoundMode::DOWN, int>(fx);

    if (sx < 0)
    {
        fx = 0, sx = 0;
    }

    if (sx >= src.width(batch_idx) - 1)
    {
        fx = 0, sx = src.width(batch_idx) - 2;
    }
    if (sy >= src.height(batch_idx) - 1)
    {
        sy = src.height(batch_idx) - 2;
    }

    float cbufx[2];
    cbufx[0] = 1.f - fx;
    cbufx[1] = fx;

    *dst.ptr(batch_idx, y, x) = cuda::SaturateCast<T>((*src.ptr(batch_idx, sy, sx) * cbufx[0] * cbufy[0]
                                                       + *src.ptr(batch_idx, sy + 1, sx) * cbufx[0] * cbufy[1]
                                                       + *src.ptr(batch_idx, sy, sx + 1) * cbufx[1] * cbufy[0]
                                                       + *src.ptr(batch_idx, sy + 1, sx + 1) * cbufx[1] * cbufy[1]));
}

template<typename T>
void resize(const ImageBatchVarShapeDataStridedCuda &in, const ImageBatchVarShapeDataStridedCuda &out,
            const int interpolation, hipStream_t stream)
{
    NVCV_ASSERT(in.numImages() == out.numImages());

    cuda::ImageBatchVarShapeWrap<const T> src_ptr(in);
    cuda::ImageBatchVarShapeWrap<T>       dst_ptr(out);

    Size2D outMaxSize = out.maxSize();

    const int THREADS_PER_BLOCK = 256; //Performance degrades above 256 and below 16 (GMEM speed limited)
    const int BLOCK_WIDTH       = 8;   //as in 32x4 or 32x8 or 8x32.

    const dim3 blockSize(BLOCK_WIDTH, THREADS_PER_BLOCK / BLOCK_WIDTH, 1);
    const dim3 gridSize(divUp(outMaxSize.w, blockSize.x), divUp(outMaxSize.h, blockSize.y), in.numImages());

    //quad permits aligned writes to output image, if image is multiple of 4.  kernels in resize_varshape are smart
    const int  out_quad_width = outMaxSize.w / 4;
    const dim3 quadGridSize(divUp(out_quad_width, blockSize.x), divUp(outMaxSize.h, blockSize.y), in.numImages());

    switch (interpolation)
    {
    case NVCV_INTERP_NEAREST:
        resize_NN<T><<<gridSize, blockSize, 0, stream>>>(src_ptr, dst_ptr);
        break;

    case NVCV_INTERP_LINEAR:
        resize_bilinear<T><<<gridSize, blockSize, 0, stream>>>(src_ptr, dst_ptr);
        break;

    case NVCV_INTERP_CUBIC:
        resize_bicubic<T><<<gridSize, blockSize, 0, stream>>>(src_ptr, dst_ptr);
        break;

    case NVCV_INTERP_AREA:
        cuda::BorderVarShapeWrap<const T, NVCV_BORDER_CONSTANT> brdSrc(in);
        resize_area_ocv_align<T><<<gridSize, blockSize, 0, stream>>>(src_ptr, brdSrc, dst_ptr);
        break;

    } //switch interpolation
    checkKernelErrors();

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif
}

} // namespace

ErrorCode ResizeVarShape::infer(const ImageBatchVarShapeDataStridedCuda &inData,
                                const ImageBatchVarShapeDataStridedCuda &outData,
                                const NVCVInterpolationType interpolation, hipStream_t stream)
{
    DataFormat input_format  = helpers::GetLegacyDataFormat(inData);
    DataFormat output_format = helpers::GetLegacyDataFormat(outData);

    if (input_format != output_format)
    {
        LOG_ERROR("Invalid DataFormat between input (" << input_format << ") and output (" << output_format << ")");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataFormat format = input_format;

    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid input DataFormat " << format << ", the valid DataFormats are: \"NHWC\", \"HWC\"");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!inData.uniqueFormat())
    {
        LOG_ERROR("Images in input batch must all have the same format ");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    int channels = inData.uniqueFormat().numChannels();

    if (channels > 4)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    DataType data_type = helpers::GetLegacyDataType(inData.uniqueFormat());

    if (!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_16S || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    if (!(interpolation == NVCV_INTERP_LINEAR || interpolation == NVCV_INTERP_NEAREST
          || interpolation == NVCV_INTERP_CUBIC || interpolation == NVCV_INTERP_AREA))
    {
        LOG_ERROR("Invalid interpolation " << interpolation);
        return ErrorCode::INVALID_PARAMETER;
    }

    typedef void (*func_t)(const ImageBatchVarShapeDataStridedCuda &in, const ImageBatchVarShapeDataStridedCuda &out,
                           const int interpolation, hipStream_t stream);

    static const func_t funcs[6][4] = {
        {      resize<uchar>,  0 /*resize<uchar2>*/,      resize<uchar3>,      resize<uchar4>},
        {0 /*resize<schar>*/,   0 /*resize<char2>*/, 0 /*resize<char3>*/, 0 /*resize<char4>*/},
        {     resize<ushort>, 0 /*resize<ushort2>*/,     resize<ushort3>,     resize<ushort4>},
        {      resize<short>,  0 /*resize<short2>*/,      resize<short3>,      resize<short4>},
        {  0 /*resize<int>*/,    0 /*resize<int2>*/,  0 /*resize<int3>*/,  0 /*resize<int4>*/},
        {      resize<float>,  0 /*resize<float2>*/,      resize<float3>,      resize<float4>}
    };

    const func_t func = funcs[data_type][channels - 1];

    assert(func != 0);
    func(inData, outData, interpolation, stream);
    return ErrorCode::SUCCESS;
} // namespace

} // namespace nvcv::legacy::cuda_op
