#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

//#include <cvcuda/OpNormalize.h> // for CVCUDA_NORMALIZE_SCALE_IS_STDDEV, etc.
#include <cvcuda/cuda_tools/MathWrappers.hpp>

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;

template<class SrcWrapper, class DstWrapper>
__global__ void hist_kernel(const SrcWrapper src, DstWrapper histogram, int channels)
{
    const int src_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int src_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();
    const int local_id
        = threadIdx.y * blockDim.x + threadIdx.x; // just the thread id within the block of 256 * channels threads

    //initialize the block local memory
    extern __shared__ int shist[];
    if (local_id < 256 * channels)
    {
        shist[local_id] = 0;
    }
    __syncthreads(); // wait for all threads to finish initialization

    //check if we are in the image.
    if (src_x < src.width(batch_idx) && src_y < src.height(batch_idx))
    {
        for (int ch = 0; ch < channels; ch++)
        {
            int4  coordImg{batch_idx, src_y, src_x, ch};
            uchar out = src[coordImg];
            int   idx = out + (256 * ch);
            atomicAdd(&shist[idx], 1);
        }
    }
    __syncthreads();

    // copy to the final destination histogram from block local memory
    if (local_id < 256 * channels)
    {
        int hist_val = shist[local_id];
        if (hist_val > 0)
        {
            int2 coordHisto{local_id, batch_idx};
            atomicAdd(&histogram[coordHisto], hist_val);
        }
    }
}

template<class CdfWrapper, class SrcWrapper>
__global__ void prefix_sum_with_norm_kernel(CdfWrapper histogram, SrcWrapper dst)
{
    const int tid       = threadIdx.x; // thread id in the block 0-255
    const int batch_idx = get_batch_idx();

    const int  hist_idx = threadIdx.x + (blockIdx.x * 256); // index into the histogram 0-255*channels
    const int2 coordHisto{hist_idx, batch_idx};             // index into the histogram 0-255*channels

    __shared__ int temp[256 * 2]; // temp block shared buffer
    int           *reduce_buf = &temp[256];

    // Set block shared memory
    temp[tid] = histogram[coordHisto]; // copy the histogram for this thred to the shared buffer

    // check if there is a histogram value for this index
    if (temp[tid])
    {
        reduce_buf[tid] = tid; // set the reduce buffer to the index of this histogram bin
    }
    else
    {
        reduce_buf[tid] = 255;
    }
    __syncthreads();

    //min-reduce
    for (int s = 128; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            reduce_buf[tid] = min(reduce_buf[tid], reduce_buf[tid + s]);
        }
        __syncthreads();
    }

    const int min_idx      = reduce_buf[0]; // this is the first non-zero element in the histogram
    const int total_pixels = dst.width(batch_idx) * dst.height(batch_idx);

    // compute and normalize cdf put into histogram
    if (temp[min_idx] == total_pixels)
    {
        // all pixels have same value
        histogram[coordHisto] = min_idx;
    }
    else
    {
        int pout = 0, pin = 1;
        temp[min_idx] = 0;
        for (int offset = 1; offset < 256; offset *= 2)
        {
            pout = 1 - pout;
            pin  = 1 - pout;
            if (tid >= offset)
                temp[pout * 256 + tid] = temp[pin * 256 + tid] + temp[pin * 256 + tid - offset];
            else
                temp[pout * 256 + tid] = temp[pin * 256 + tid];
            __syncthreads();
        }
        histogram[coordHisto]
            = nvcv::cuda::SaturateCast<int>(1.0 * temp[pout * 256 + tid] / temp[pout * 256 + 255] * 255);
    }
}

template<class SrcWrapper, class DstWrapper, class CdfWrapper>
__global__ void lookup(const SrcWrapper src, DstWrapper dst, CdfWrapper cdf, int channels)

{
    const int             src_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int             src_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int             batch_idx = get_batch_idx();
    const int             local_id  = threadIdx.y * blockDim.x + threadIdx.x;
    extern __shared__ int temp[];

    //copy the cdf to the block shared memory
    if (local_id < 256 * channels)
    {
        int2 coordHisto{local_id, batch_idx};
        temp[local_id] = cdf[coordHisto];
    }
    __syncthreads();

    //check if we are in the image.
    if (src_x < dst.width(batch_idx) && src_y < dst.height(batch_idx))
    {
        int offset = 0;
        for (int ch = 0; ch < channels; ch++)
        {
            offset = 256 * ch;
            int4 coordImg{batch_idx, src_y, src_x, ch};
            int2 coordHisto{src[coordImg] + offset, batch_idx};
            dst[coordImg] = nvcv::cuda::SaturateCast<uchar>((temp[src[coordImg] + offset]));
        }
    }
}

namespace nvcv::legacy::cuda_op {

HistogramEqVarShape::HistogramEqVarShape(int maxBatchSize)
{
    if (maxBatchSize < 0)
    {
        LOG_ERROR("Invalid num of max batch size " << maxBatchSize);
        throw nvcv::Exception(nvcv::Status::ERROR_INVALID_ARGUMENT, "maxBatchSize must be >= 0");
    }

    m_maxBatchSize    = maxBatchSize;
    m_maxChannelCount = 4;
    //histogram is 256 * channels per image
    m_sizeOfHisto = m_maxBatchSize * m_maxChannelCount * 256 * sizeof(int);

    if (m_maxBatchSize > 0 && m_maxChannelCount > 0)
    {
        NVCV_CHECK_THROW(hipMalloc(&m_histoArray, m_sizeOfHisto));
    }
}

HistogramEqVarShape::~HistogramEqVarShape()
{
    if (m_histoArray)
    {
        hipFree(m_histoArray);
        m_histoArray = nullptr;
    }
}

ErrorCode HistogramEqVarShape::infer(const nvcv::ImageBatchVarShapeDataStridedCuda &inData,
                                     const nvcv::ImageBatchVarShapeDataStridedCuda &outData, hipStream_t stream)
{
    DataFormat input_format  = helpers::GetLegacyDataFormat(inData);
    DataFormat output_format = helpers::GetLegacyDataFormat(outData);
    if (input_format != output_format)
    {
        LOG_ERROR("Invalid DataFormat between input (" << input_format << ") and output (" << output_format << ")");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataFormat format = input_format;

    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invliad DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!inData.uniqueFormat())
    {
        LOG_ERROR("Images in the input batch must all have the same format");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!outData.uniqueFormat())
    {
        LOG_ERROR("Images in the output batch must all have the same format");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataType data_type     = helpers::GetLegacyDataType(inData.uniqueFormat());
    DataType out_data_type = helpers::GetLegacyDataType(outData.uniqueFormat());
    int      channels      = inData.uniqueFormat().numChannels();
    int      batch         = inData.numImages();

    if (data_type != out_data_type)
    {
        LOG_ERROR("Input and Output formats must be same data type");
        return ErrorCode::INVALID_DATA_TYPE;
    }

    if (!(data_type == kCV_8U))
    {
        LOG_ERROR("Invalid IDataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    if (channels > 4 || channels < 1)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    if (inData.numImages() != outData.numImages())
    {
        LOG_ERROR("Input and Output batch size must be same");
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    cuda::ImageBatchVarShapeWrapNHWC<uchar> dst(outData, channels);
    cuda::ImageBatchVarShapeWrapNHWC<uchar> src(inData, channels);
    auto histo = nvcv::cuda::Tensor2DWrap<int, int32_t>(m_histoArray, (int)(256 * channels * sizeof(int)));

    {
        //compute the histogram for each image in the batch into m_histoArray
        int bsX = 32; //1024 ( 4 ch of 256 bins)
        int bsY = 32;

        switch (channels)
        {
        case 1:
            bsX = 16; // 256 (1 ch)
            bsY = 16;
            break;
        case 2:
            bsX = 32; // 512 (2 ch)
            bsY = 16;
            break;
        case 3:
            bsX = 32; // 768 (3 ch)
            bsY = 24;
            break;
        default:
            break;
        }

        // each block is going to be 256bins * channels = threads
        dim3   histBlockSize(bsX, bsY, 1);
        dim3   histGridSize(divUp(inData.maxSize().w, histBlockSize.x), divUp(inData.maxSize().h, histBlockSize.y),
                            batch);
        size_t sharedMemSize = 256 * channels * sizeof(int);
        hist_kernel<<<histGridSize, histBlockSize, sharedMemSize, stream>>>(src, histo, channels);
        checkKernelErrors();
    }
    //compute cfd
    {
        int  bsX = 256;
        int  bsY = 1;
        int  bsZ = 1;
        dim3 prefixSumBlockSize(bsX, bsY, bsZ);
        dim3 prefixSumGridSize(channels, 1, batch);
        prefix_sum_with_norm_kernel<<<prefixSumGridSize, prefixSumBlockSize, 0, stream>>>(histo, dst);
        checkKernelErrors();
    }
    //lookup
    {
        dim3 lookupBlockSize(32, 32, 1);
        dim3 lookupGridSize(divUp(inData.maxSize().w, lookupBlockSize.x), divUp(inData.maxSize().h, lookupBlockSize.y),
                            batch);
        lookup<<<lookupGridSize, lookupBlockSize, 256 * channels * sizeof(int), stream>>>(src, dst, histo, channels);
        checkKernelErrors();
    }

    return ErrorCode::SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
