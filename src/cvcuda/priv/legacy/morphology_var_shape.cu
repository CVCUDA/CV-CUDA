#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

#include <nvcv/cuda/MathWrappers.hpp>
#include <nvcv/cuda/SaturateCast.hpp>

using namespace nvcv::legacy::helpers;
using namespace nvcv::legacy::cuda_op;

namespace nvcv::legacy::cuda_op {

__global__ void UpdateMasksAnchors(cuda::Tensor1DWrap<int2> masks, cuda::Tensor1DWrap<int2> anchors, int numImages,
                                   int iteration)
{
    int1 coord;
    coord.x = cuda::StaticCast<int>(blockIdx.x * blockDim.x + threadIdx.x);
    if (coord.x >= numImages)
        return;

    int2 mask_size = masks[coord];
    int2 anchor    = anchors[coord];

    if (mask_size.x == -1 || mask_size.y == -1)
        mask_size.x = mask_size.y = 3;
    if (anchor.x < 0)
        anchor.x = mask_size.x / 2;
    if (anchor.y < 0)
        anchor.y = mask_size.y / 2;

    mask_size = mask_size + (iteration - 1) * (mask_size - 1);
    anchor    = anchor * iteration;

    masks[coord]   = mask_size;
    anchors[coord] = anchor;
}

template<class SrcWrapper, class DstWrapper, typename D = typename DstWrapper::ValueType,
         typename BT = typename cuda::BaseType<D>>
__global__ void dilate(const SrcWrapper src, DstWrapper dst, cuda::Tensor1DWrap<int2> kernelSizeArr,
                       cuda::Tensor1DWrap<int2> kernelAnchorArr, BT maxmin)
{
    D         res       = cuda::SetAll<D>(maxmin);
    const int x         = blockIdx.x * blockDim.x + threadIdx.x;
    const int y         = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();

    if (x >= dst.width(batch_idx) || y >= dst.height(batch_idx))
        return;

    int2 kernelSize = kernelSizeArr[batch_idx];
    int2 anchor     = kernelAnchorArr[batch_idx];

    int3 srcCoord = {0, 0, batch_idx};

    for (int i = 0; i < kernelSize.y; ++i)
    {
        srcCoord.y = y - anchor.y + i;

        for (int j = 0; j < kernelSize.x; ++j)
        {
            srcCoord.x = x - anchor.x + j;

            res = cuda::max(res, src[srcCoord]);
        }
    }

    *dst.ptr(batch_idx, y, x) = cuda::SaturateCast<D>(res);
}

template<class SrcWrapper, class DstWrapper, typename D = typename DstWrapper::ValueType,
         typename BT = typename cuda::BaseType<D>>
__global__ void erode(const SrcWrapper src, DstWrapper dst, cuda::Tensor1DWrap<int2> kernelSizeArr,
                      cuda::Tensor1DWrap<int2> kernelAnchorArr, BT maxmin)
{
    D         res       = cuda::SetAll<D>(maxmin);
    const int x         = blockIdx.x * blockDim.x + threadIdx.x;
    const int y         = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();

    if (x >= dst.width(batch_idx) || y >= dst.height(batch_idx))
        return;

    int2 kernelSize = kernelSizeArr[batch_idx];
    int2 anchor     = kernelAnchorArr[batch_idx];

    int3 srcCoord = {0, 0, batch_idx};

    for (int i = 0; i < kernelSize.y; ++i)
    {
        srcCoord.y = y - anchor.y + i;

        for (int j = 0; j < kernelSize.x; ++j)
        {
            srcCoord.x = x - anchor.x + j;

            res = cuda::min(res, src[srcCoord]);
        }
    }

    *dst.ptr(batch_idx, y, x) = cuda::SaturateCast<D>(res);
}

template<typename D, NVCVBorderType B>
void MorphFilter2DCaller(const ImageBatchVarShapeDataStridedCuda &inData,
                         const ImageBatchVarShapeDataStridedCuda &outData, const TensorDataStridedCuda &kMasks,
                         const TensorDataStridedCuda &kAnchors, NVCVMorphologyType morph_type, hipStream_t stream)
{
    cuda::Tensor1DWrap<int2> kernelSizeTensor(kMasks);
    cuda::Tensor1DWrap<int2> kernelAnchorTensor(kAnchors);

    Size2D outMaxSize = outData.maxSize();
    int    maxWidth   = outMaxSize.w;
    int    maxHeight  = outMaxSize.h;

    dim3 block(16, 16);
    dim3 grid(divUp(maxWidth, block.x), divUp(maxHeight, block.y), outData.numImages());

    using BT = nvcv::cuda::BaseType<D>;
    BT val   = (morph_type == NVCVMorphologyType::NVCV_DILATE) ? std::numeric_limits<BT>::min()
                                                               : std::numeric_limits<BT>::max();

    cuda::BorderVarShapeWrap<const D, B> src(inData, cuda::SetAll<D>(val));
    cuda::ImageBatchVarShapeWrap<D>      dst(outData);

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif

    if (morph_type == NVCVMorphologyType::NVCV_ERODE)
    {
        erode<<<grid, block, 0, stream>>>(src, dst, kernelSizeTensor, kernelAnchorTensor, val);
        checkKernelErrors();
    }
    else if (morph_type == NVCVMorphologyType::NVCV_DILATE)
    {
        dilate<<<grid, block, 0, stream>>>(src, dst, kernelSizeTensor, kernelAnchorTensor, val);
        checkKernelErrors();
    }

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif
}

template<typename D>
void MorphFilter2D(const ImageBatchVarShapeDataStridedCuda &inData, const ImageBatchVarShapeDataStridedCuda &outData,
                   const TensorDataStridedCuda &kMasks, const TensorDataStridedCuda &kAnchors,
                   NVCVMorphologyType morph_type, NVCVBorderType borderMode, hipStream_t stream)
{
    typedef void (*func_t)(const ImageBatchVarShapeDataStridedCuda &inData,
                           const ImageBatchVarShapeDataStridedCuda &outData, const TensorDataStridedCuda &kMasks,
                           const TensorDataStridedCuda &kAnchors, NVCVMorphologyType morph_type, hipStream_t stream);

    static const func_t funcs[]
        = {MorphFilter2DCaller<D, NVCV_BORDER_CONSTANT>, MorphFilter2DCaller<D, NVCV_BORDER_REPLICATE>,
           MorphFilter2DCaller<D, NVCV_BORDER_REFLECT>, MorphFilter2DCaller<D, NVCV_BORDER_WRAP>,
           MorphFilter2DCaller<D, NVCV_BORDER_REFLECT101>};

    funcs[borderMode](inData, outData, kMasks, kAnchors, morph_type, stream);
}

MorphologyVarShape::MorphologyVarShape(const int maxBatchSize)
    : CudaBaseOp()
    , m_maxBatchSize(maxBatchSize)
    , m_kernelMaskSizes(maxBatchSize)
    , m_kernelAnchors(maxBatchSize)
{
    if (m_maxBatchSize > 0)
    {
        // {Width, Height} per image for mask and anchor
        size_t totalNumElements = m_maxBatchSize * 2;

        m_kernelMaskSizes.resize(totalNumElements);
        if (m_kernelMaskSizes.size() != totalNumElements)
        {
            throw std::runtime_error("Host memory allocation error!");
        }

        m_kernelAnchors.resize(totalNumElements);
        if (m_kernelAnchors.size() != totalNumElements)
        {
            throw std::runtime_error("Host memory allocation error!");
        }
    }
}

MorphologyVarShape::~MorphologyVarShape() {}

ErrorCode MorphologyVarShape::infer(const nvcv::IImageBatchVarShape &inBatch, const nvcv::IImageBatchVarShape &outBatch,
                                    NVCVMorphologyType morph_type, const TensorDataStridedCuda &masks,
                                    const TensorDataStridedCuda &anchors, int iteration, NVCVBorderType borderMode,
                                    hipStream_t stream)
{
    auto inData = inBatch.exportData<nvcv::ImageBatchVarShapeDataStridedCuda>(stream);
    if (inData == nullptr)
    {
        LOG_ERROR("Input must be varshape image batch");
    }
    auto outData = outBatch.exportData<nvcv::ImageBatchVarShapeDataStridedCuda>(stream);
    if (outData == nullptr)
    {
        LOG_ERROR("Output must be varshape image batch");
    }

    DataFormat input_format  = GetLegacyDataFormat(*inData);
    DataFormat output_format = GetLegacyDataFormat(*outData);
    DataType   data_type     = GetLegacyDataType(inData->uniqueFormat());

    if (inData->numImages() > m_maxBatchSize)
    {
        LOG_ERROR("Number of VarShape Images exceeds configured max size");
        return ErrorCode::INVALID_PARAMETER;
    }

    if (input_format != output_format)
    {
        LOG_ERROR("Invalid DataFormat between input (" << input_format << ") and output (" << output_format << ")");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (input_format != output_format)
    {
        LOG_ERROR("Invalid DataFormat between input (" << input_format << ") and output (" << output_format << ")");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataFormat format = input_format;
    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    if (!(borderMode == NVCV_BORDER_REFLECT101 || borderMode == NVCV_BORDER_REPLICATE
          || borderMode == NVCV_BORDER_CONSTANT || borderMode == NVCV_BORDER_REFLECT || borderMode == NVCV_BORDER_WRAP))
    {
        LOG_ERROR("Invalid borderMode " << borderMode);
        return ErrorCode::INVALID_PARAMETER;
    }

    const int channels = inData->uniqueFormat().numChannels();

    if (!(channels == 1 || channels == 3 || channels == 4))
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_PARAMETER;
    }

    if (iteration == 0)
    {
        for (auto init = inBatch.begin(), outit = outBatch.begin(); init != inBatch.end(), outit != outBatch.end();
             ++init, ++outit)
        {
            const IImage            &inimg      = *init;
            const IImage            &outimg     = *outit;
            auto                     inimgdata  = inimg.exportData<ImageDataStridedCuda>();
            auto                     outimgdata = outimg.exportData<ImageDataStridedCuda>();
            const ImagePlaneStrided &inplane    = inimgdata->plane(0);
            const ImagePlaneStrided &outplane   = outimgdata->plane(0);
            checkCudaErrors(hipMemcpy2DAsync(outplane.basePtr, outplane.rowStride, inplane.basePtr, inplane.rowStride,
                                              inplane.rowStride, inplane.height, hipMemcpyDeviceToDevice, stream));
        }
        return ErrorCode::SUCCESS;
    }

    dim3                     block(32), grid(divUp(inData->numImages(), 32));
    cuda::Tensor1DWrap<int2> kmasks(masks), kanchors(anchors);
    UpdateMasksAnchors<<<grid, block, 0, stream>>>(kmasks, kanchors, inData->numImages(), iteration);

    typedef void (*filter2D_t)(const ImageBatchVarShapeDataStridedCuda &inData,
                               const ImageBatchVarShapeDataStridedCuda &outData, const TensorDataStridedCuda &kMasks,
                               const TensorDataStridedCuda &kAnchors, NVCVMorphologyType morph_type,
                               NVCVBorderType borderMode, hipStream_t stream);

    static const filter2D_t funcs[6][4] = {
        { MorphFilter2D<uchar>, 0,  MorphFilter2D<uchar3>,  MorphFilter2D<uchar4>},
        {                    0, 0,                      0,                      0},
        {MorphFilter2D<ushort>, 0, MorphFilter2D<ushort3>, MorphFilter2D<ushort4>},
        {                    0, 0,                      0,                      0},
        {                    0, 0,                      0,                      0},
        { MorphFilter2D<float>, 0,  MorphFilter2D<float3>,  MorphFilter2D<float4>},
    };

    funcs[data_type][channels - 1](*inData, *outData, masks, anchors, morph_type, borderMode, stream);

    return ErrorCode::SUCCESS;
}
} // namespace nvcv::legacy::cuda_op
