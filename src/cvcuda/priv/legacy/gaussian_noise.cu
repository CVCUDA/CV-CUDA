#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"
#include "gaussian_noise_util.cuh"

#include <hiprand/hiprand_kernel.h>
using namespace nvcv::legacy::helpers;

using namespace nvcv::legacy::cuda_op;

using namespace nvcv::cuda;

#define BLOCK 512

template<typename T, typename StrideType>
__global__ void gaussian_noise_kernel(const Tensor3DWrap<T, StrideType> src, Tensor3DWrap<T, StrideType> dst,
                                      hiprandState *state, Tensor1DWrap<float, int32_t> mu,
                                      Tensor1DWrap<float, int32_t> sigma, int rows, int cols)
{
    int         offset     = threadIdx.x;
    int         batch_idx  = blockIdx.x;
    int         id         = threadIdx.x + blockIdx.x * blockDim.x;
    int         total_size = rows * cols;
    hiprandState localState = state[id];
    while (offset < total_size)
    {
        int   dst_x                       = offset % cols;
        int   dst_y                       = offset / cols;
        float rand                        = hiprand_normal(&localState);
        float delta                       = mu[batch_idx] + rand * sigma[batch_idx];
        *dst.ptr(batch_idx, dst_y, dst_x) = SaturateCast<T>(*src.ptr(batch_idx, dst_y, dst_x) + delta);
        offset += blockDim.x;
    }
    state[id] = localState;
}

template<typename T, typename StrideType>
__global__ void gaussian_noise_per_channel_kernel(const Tensor4DWrap<T, StrideType> src,
                                                  Tensor4DWrap<T, StrideType> dst, hiprandState *state,
                                                  Tensor1DWrap<float, int32_t> mu, Tensor1DWrap<float, int32_t> sigma,
                                                  int rows, int cols, int channel)
{
    int         offset     = threadIdx.x;
    int         batch_idx  = blockIdx.x;
    int         id         = threadIdx.x + blockIdx.x * blockDim.x;
    int         total_size = rows * cols;
    hiprandState localState = state[id];
    while (offset < total_size)
    {
        int dst_x = offset % cols;
        int dst_y = offset / cols;
        for (int ch = 0; ch < channel; ch++)
        {
            float rand                            = hiprand_normal(&localState);
            float delta                           = mu[batch_idx] + rand * sigma[batch_idx];
            *dst.ptr(batch_idx, dst_y, dst_x, ch) = SaturateCast<T>(*src.ptr(batch_idx, dst_y, dst_x, ch) + delta);
        }
        offset += blockDim.x;
    }
    state[id] = localState;
}

template<typename T, typename StrideType>
__global__ void gaussian_noise_float_kernel(const Tensor3DWrap<T, StrideType> src, Tensor3DWrap<T, StrideType> dst,
                                            hiprandState *state, Tensor1DWrap<float, int32_t> mu,
                                            Tensor1DWrap<float, int32_t> sigma, int rows, int cols)
{
    int         offset     = threadIdx.x;
    int         batch_idx  = blockIdx.x;
    int         id         = threadIdx.x + blockIdx.x * blockDim.x;
    int         total_size = rows * cols;
    hiprandState localState = state[id];
    while (offset < total_size)
    {
        int   dst_x                       = offset % cols;
        int   dst_y                       = offset / cols;
        float rand                        = hiprand_normal(&localState);
        float delta                       = mu[batch_idx] + rand * sigma[batch_idx];
        T     out                         = SaturateCast<T>(*src.ptr(batch_idx, dst_y, dst_x) + delta);
        *dst.ptr(batch_idx, dst_y, dst_x) = clamp(StaticCast<float>(out), 0.f, 1.f);
        offset += blockDim.x;
    }
    state[id] = localState;
}

template<typename T, typename StrideType>
__global__ void gaussian_noise_float_per_channel_kernel(const Tensor4DWrap<T, StrideType> src,
                                                        Tensor4DWrap<T, StrideType> dst, hiprandState *state,
                                                        Tensor1DWrap<float, int32_t> mu,
                                                        Tensor1DWrap<float, int32_t> sigma, int rows, int cols,
                                                        int channel)
{
    int         offset     = threadIdx.x;
    int         batch_idx  = blockIdx.x;
    int         id         = threadIdx.x + blockIdx.x * blockDim.x;
    int         total_size = rows * cols;
    hiprandState localState = state[id];
    while (offset < total_size)
    {
        int dst_x = offset % cols;
        int dst_y = offset / cols;
        for (int ch = 0; ch < channel; ch++)
        {
            float rand                            = hiprand_normal(&localState);
            float delta                           = mu[batch_idx] + rand * sigma[batch_idx];
            T     out                             = SaturateCast<T>(*src.ptr(batch_idx, dst_y, dst_x, ch) + delta);
            *dst.ptr(batch_idx, dst_y, dst_x, ch) = clamp(StaticCast<float>(out), 0.f, 1.f);
        }
        offset += blockDim.x;
    }
    state[id] = localState;
}

template<typename T, typename StrideType = int32_t>
void gaussian_noise(const nvcv::TensorDataStridedCuda &d_in, const nvcv::TensorDataStridedCuda &d_out, int batch,
                    int rows, int cols, hiprandState *m_states, const nvcv::TensorDataStridedCuda &_mu,
                    const nvcv::TensorDataStridedCuda &_sigma, hipStream_t stream)
{
    auto                         src_ptr = CreateTensorWrapNHW<T, StrideType>(d_in);
    auto                         dst_ptr = CreateTensorWrapNHW<T, StrideType>(d_out);
    Tensor1DWrap<float, int32_t> mu(_mu);
    Tensor1DWrap<float, int32_t> sigma(_sigma);

    gaussian_noise_kernel<T><<<batch, BLOCK, 0, stream>>>(src_ptr, dst_ptr, m_states, mu, sigma, rows, cols);
    checkKernelErrors();
}

template<typename T, typename StrideType = int32_t>
void gaussian_noise_per_channel(const nvcv::TensorDataStridedCuda &d_in, const nvcv::TensorDataStridedCuda &d_out,
                                int batch, int channels, int rows, int cols, hiprandState *m_states,
                                const nvcv::TensorDataStridedCuda &_mu, const nvcv::TensorDataStridedCuda &_sigma,
                                hipStream_t stream)
{
    auto                         src_ptr = CreateTensorWrapNHWC<T, StrideType>(d_in);
    auto                         dst_ptr = CreateTensorWrapNHWC<T, StrideType>(d_out);
    Tensor1DWrap<float, int32_t> mu(_mu);
    Tensor1DWrap<float, int32_t> sigma(_sigma);

    gaussian_noise_per_channel_kernel<T>
        <<<batch, BLOCK, 0, stream>>>(src_ptr, dst_ptr, m_states, mu, sigma, rows, cols, channels);
    checkKernelErrors();
}

template<typename T, typename StrideType = int32_t>
void gaussian_noise_float(const nvcv::TensorDataStridedCuda &d_in, const nvcv::TensorDataStridedCuda &d_out, int batch,
                          int rows, int cols, hiprandState *m_states, const nvcv::TensorDataStridedCuda &_mu,
                          const nvcv::TensorDataStridedCuda &_sigma, hipStream_t stream)
{
    auto                         src_ptr = CreateTensorWrapNHW<T, StrideType>(d_in);
    auto                         dst_ptr = CreateTensorWrapNHW<T, StrideType>(d_out);
    Tensor1DWrap<float, int32_t> mu(_mu);
    Tensor1DWrap<float, int32_t> sigma(_sigma);

    gaussian_noise_float_kernel<T><<<batch, BLOCK, 0, stream>>>(src_ptr, dst_ptr, m_states, mu, sigma, rows, cols);
    checkKernelErrors();
}

template<typename T, typename StrideType = int32_t>
void gaussian_noise_float_per_channel(const nvcv::TensorDataStridedCuda &d_in, const nvcv::TensorDataStridedCuda &d_out,
                                      int batch, int channels, int rows, int cols, hiprandState *m_states,
                                      const nvcv::TensorDataStridedCuda &_mu, const nvcv::TensorDataStridedCuda &_sigma,
                                      hipStream_t stream)
{
    auto                         src_ptr = CreateTensorWrapNHWC<T, StrideType>(d_in);
    auto                         dst_ptr = CreateTensorWrapNHWC<T, StrideType>(d_out);
    Tensor1DWrap<float, int32_t> mu(_mu);
    Tensor1DWrap<float, int32_t> sigma(_sigma);

    gaussian_noise_float_per_channel_kernel<T>
        <<<batch, BLOCK, 0, stream>>>(src_ptr, dst_ptr, m_states, mu, sigma, rows, cols, channels);
    checkKernelErrors();
}

namespace nvcv::legacy::cuda_op {

GaussianNoise::GaussianNoise(DataShape max_input_shape, DataShape max_output_shape, int maxBatchSize)
    : CudaBaseOp(max_input_shape, max_output_shape)
    , m_states(nullptr)
    , m_seed(0)
    , m_maxBatchSize(maxBatchSize)
    , m_setupDone(false)
{
    if (maxBatchSize < 0)
    {
        LOG_ERROR("Invalid num of max batch size " << maxBatchSize);
        throw nvcv::Exception(nvcv::Status::ERROR_INVALID_ARGUMENT, "Parameter error!");
    }
    hipError_t err = hipMalloc((void **)&m_states, sizeof(hiprandState) * BLOCK * maxBatchSize);
    if (err != hipSuccess)
    {
        LOG_ERROR("CUDA memory allocation error of size: " << sizeof(hiprandState) * BLOCK * maxBatchSize);
        throw std::runtime_error("CUDA memory allocation error!");
    }
}

GaussianNoise::~GaussianNoise()
{
    hipError_t err = hipFree(m_states);
    if (err != hipSuccess)
        LOG_ERROR("CUDA memory free error, possible memory leak!");
}

ErrorCode GaussianNoise::infer(const TensorDataStridedCuda &inData, const TensorDataStridedCuda &outData,
                               const TensorDataStridedCuda &mu, const TensorDataStridedCuda &sigma, bool per_channel,
                               unsigned long long seed, hipStream_t stream)
{
    DataFormat in_format  = GetLegacyDataFormat(inData.layout());
    DataFormat out_format = GetLegacyDataFormat(outData.layout());
    if (!(in_format == kNHWC || in_format == kHWC))
    {
        LOG_ERROR("Invalid input DataFormat " << in_format << ", the valid DataFormats are: \"NHWC\", \"HWC\"");
        return ErrorCode::INVALID_DATA_FORMAT;
    }
    if (!(out_format == kNHWC || out_format == kHWC))
    {
        LOG_ERROR("Invalid output DataFormat " << out_format << ", the valid DataFormats are: \"NHWC\", \"HWC\"");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto inAccess = TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);
    int channels = inAccess->numChannels();
    if (channels > 4)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    auto inMaxStride = inAccess->sampleStride() * inAccess->numSamples();
    if (inMaxStride > cuda::TypeTraits<int32_t>::max)
    {
        LOG_ERROR("Input size exceeds " << nvcv::cuda::TypeTraits<int32_t>::max << ". Tensor is too large.");
        return ErrorCode::INVALID_PARAMETER;
    }

    auto outAccess = TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    auto outMaxStride = outAccess->sampleStride() * outAccess->numSamples();
    if (outMaxStride > cuda::TypeTraits<int32_t>::max)
    {
        LOG_ERROR("Output size exceeds " << nvcv::cuda::TypeTraits<int32_t>::max << ". Tensor is too large.");
        return ErrorCode::INVALID_PARAMETER;
    }

    DataType in_data_type = GetLegacyDataType(inData.dtype());
    if (!(in_data_type == kCV_8U || in_data_type == kCV_16U || in_data_type == kCV_16S || in_data_type == kCV_32S
          || in_data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << in_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType out_data_type = GetLegacyDataType(outData.dtype());
    if (in_data_type != out_data_type)
    {
        LOG_ERROR("DataType of input and output must be equal, but got " << in_data_type << " and " << out_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType mu_data_type = GetLegacyDataType(mu.dtype());
    if (mu_data_type != kCV_32F)
    {
        LOG_ERROR("Invalid mu DataType " << mu_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }
    int mu_dim = mu.layout().rank();
    if (mu_dim != 1)
    {
        LOG_ERROR("Invalid mu Dim " << mu_dim);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataType sigma_data_type = GetLegacyDataType(sigma.dtype());
    if (sigma_data_type != kCV_32F)
    {
        LOG_ERROR("Invalid sigma DataType " << sigma_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }
    int sigma_dim = sigma.layout().rank();
    if (sigma_dim != 1)
    {
        LOG_ERROR("Invalid sigma Dim " << sigma_dim);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!m_setupDone || m_seed != seed)
    {
        m_seed = seed;
        setup_gaussian_rand_kernel<<<m_maxBatchSize, BLOCK, 0, stream>>>(m_states, m_seed);
        m_setupDone = true;
    }

    if (per_channel)
    {
        typedef void (*func_t)(const TensorDataStridedCuda &d_in, const TensorDataStridedCuda &d_out, int batch,
                               int channels, int rows, int cols, hiprandState *m_states, const TensorDataStridedCuda &mu,
                               const TensorDataStridedCuda &sigma, hipStream_t stream);

        static const func_t funcs[5] = {
            gaussian_noise_per_channel<uchar>, 0, gaussian_noise_per_channel<ushort>, gaussian_noise_per_channel<short>,
            gaussian_noise_per_channel<int>,
        };

        static const func_t float_funcs[1] = {
            gaussian_noise_float_per_channel<float>,
        };

        if (in_data_type == kCV_32F)
        {
            const func_t func = float_funcs[0];
            assert(func != 0);
            func(inData, outData, inAccess->numSamples(), channels, inAccess->numRows(), inAccess->numCols(), m_states,
                 mu, sigma, stream);
        }
        else
        {
            const func_t func = funcs[in_data_type];
            assert(func != 0);
            func(inData, outData, inAccess->numSamples(), channels, inAccess->numRows(), inAccess->numCols(), m_states,
                 mu, sigma, stream);
        }
    }
    else
    {
        typedef void (*func_t)(const TensorDataStridedCuda &d_in, const TensorDataStridedCuda &d_out, int batch,
                               int rows, int cols, hiprandState *m_states, const TensorDataStridedCuda &mu,
                               const TensorDataStridedCuda &sigma, hipStream_t stream);

        static const func_t funcs[5][4] = {
            {      gaussian_noise<uchar>,      gaussian_noise<uchar2>,      gaussian_noise<uchar3>,gaussian_noise<uchar4>                                                                                                   },
            {0 /*gaussian_noise<schar>*/, 0 /*gaussian_noise<char2>*/, 0 /*gaussian_noise<char3>*/,
             0 /*gaussian_noise<char4>*/                                                                                   },
            {     gaussian_noise<ushort>,     gaussian_noise<ushort2>,     gaussian_noise<ushort3>, gaussian_noise<ushort4>},
            {      gaussian_noise<short>,      gaussian_noise<short2>,      gaussian_noise<short3>,  gaussian_noise<short4>},
            {        gaussian_noise<int>,        gaussian_noise<int2>,        gaussian_noise<int3>,    gaussian_noise<int4>},
        };

        static const func_t float_funcs[4] = {gaussian_noise_float<float>, gaussian_noise_float<float2>,
                                              gaussian_noise_float<float3>, gaussian_noise_float<float4>};

        if (in_data_type == kCV_32F)
        {
            const func_t func = float_funcs[channels - 1];
            assert(func != 0);
            func(inData, outData, inAccess->numSamples(), inAccess->numRows(), inAccess->numCols(), m_states, mu, sigma,
                 stream);
        }
        else
        {
            const func_t func = funcs[in_data_type][channels - 1];
            assert(func != 0);
            func(inData, outData, inAccess->numSamples(), inAccess->numRows(), inAccess->numCols(), m_states, mu, sigma,
                 stream);
        }
    }
    return SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
