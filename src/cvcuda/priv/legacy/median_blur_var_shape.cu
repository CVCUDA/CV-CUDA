#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

#define GENERAL_KERNEL_BLOCK 32
#define SMALL_KERNEL_BLOCK   16

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;

namespace nvcv::legacy::cuda_op {

/**
 * This function fetches the pixel from the shared if possible.
 * Otherwise, the pixel is read from global memory.
 * If the given index is out of bound, then based on the rule of nvcv::BORDER_REPLICATE,
 * this function fetches the nearest valid pixel.
 * @tparam T The type of the pixels stored.
 * @param shared a pointer of type T to shared memory,
 * @param src a Ptr2dNHWC <T> stored in global memory.
 * @param batchIdx the index of the image.
 * @param h the height of the image.
 * @param w the width of the image.
 * @param c the channel being processed.
 * @param sxOffset the x offset that is subtracted from gx to calculate the corresponding pixel index in shared memory.
 * @param syOffset the y offset that is subtracted from gy to calculate the corresponding pixel index in shared memory.
 * @param gx the horizontal index of the desired pixel in the image.
 * @param gy the vertical index of the desired pixel in the image.
 * @return the pixel at given index
 */
template<typename T>
__device__ T fetch(T *shared, const Ptr2dVarShapeNHWC<T> src, int batchIdx, int h, int w, int c, int sxOffset,
                   int syOffset, int gx, int gy, int block_size)
{
    // check for nvcv::BORDER_REPLICATE.
    if (gx < 0)
    {
        gx = 0;
    }
    if (gx >= w)
    {
        gx = w - 1;
    }
    if (gy < 0)
    {
        gy = 0;
    }
    if (gy >= h)
    {
        gy = h - 1;
    }
    // check if the desired pixel is not in shared memory.
    if (gy - syOffset < 0 || gy - syOffset >= blockDim.y || gx - sxOffset < 0 || gx - sxOffset >= blockDim.x)
    {
        return *src.ptr(batchIdx, gy, gx, c); // fetch from global memory.
    }
    else
    {
        return shared[(gy - syOffset) * block_size + gx - sxOffset]; // fetch from shared memory.
    }
}

/**
 * Perform median fliter on the image
 * @tparam T The type of the pixels stored.
 * @param src a Ptr2dNHWC <T> stored in global memory.
 * @param dst a Ptr2dNHWC <T> stored in global memory.
 * @param kWidth width of the kernel.
 * @param kHeight height of the kernel.
 */
template<typename T>
__global__ void median(const Ptr2dVarShapeNHWC<T> src, Ptr2dVarShapeNHWC<T> dst, const cuda::Tensor2DWrap<int> ksize)
{
#define fetch_(gx, gy, block_size) \
    fetch<T>(tails, src, batchIdx, h, w, channel, blockX, blockY, (gx), (gy), (block_size))
#define fetchAs1d(idx, block_size) \
    fetch_(x - (kWidth / 2) + ((idx) % kWidth), y - (kHeight / 2) + ((idx) / kWidth), (block_size))
    int tx = threadIdx.x, ty = threadIdx.y;
    int blockX   = blockIdx.x * blockDim.x;
    int blockY   = blockIdx.y * blockDim.y;
    int x        = blockX + threadIdx.x;
    int y        = blockY + threadIdx.y;
    int channel  = blockIdx.z % dst.nch;
    int batchIdx = blockIdx.z / dst.nch;
    int h = src.at_rows(batchIdx), w = src.at_cols(batchIdx);
    int kWidth  = *ksize.ptr(batchIdx, 0); //kWidths[batchIdx];
    int kHeight = *ksize.ptr(batchIdx, 1); //kHeights[batchIdx];

    __shared__ T tails[GENERAL_KERNEL_BLOCK * GENERAL_KERNEL_BLOCK];
    if (x < w && y < h)
    {
        tails[ty * GENERAL_KERNEL_BLOCK + tx] = *src.ptr(batchIdx, y, x, channel);
    }
    __syncthreads();

    if ((x < w && y < h))
    {
        // min_ and max_ set up a range that we are looking for
        // only elements in that range could be median
        T    tmp, pivot0, pivot1, pivot2, min_, max_;
        // In the 1st and possibly several following iterations, min_ or max_ is not assigned.
        // use isMinReady and isMaxReady to control from comparison on them.
        bool isMinReady = false, isMaxReady = false;
        int  numOfEq = 0, numOfGt = 0, numOfLt = 0, numOfTaken = 0;
        int  median = (kWidth * kHeight) / 2;
        int  start = 0, end = kWidth * kHeight, t;
        bool isAllPreviousOutOfRange = true;

        // loop until we rule out all possible elements, and the last pivot is the median.
        while (numOfTaken < (kWidth * kHeight))
        {
            pivot0 = fetchAs1d(start, GENERAL_KERNEL_BLOCK);
            while ((isMinReady && (min_ >= pivot0)) || (isMaxReady && (max_ <= pivot0)))
            {
                start++;
                pivot0 = fetchAs1d(start, GENERAL_KERNEL_BLOCK);
            }

            pivot2 = fetchAs1d(end - 1, GENERAL_KERNEL_BLOCK);
            while ((isMinReady && (min_ >= pivot2)) || (isMaxReady && (max_ <= pivot2)))
            {
                end--;
                pivot2 = fetchAs1d(end - 1, GENERAL_KERNEL_BLOCK);
            }

            int idx = (start + end) / 2;
            pivot1  = fetchAs1d(idx, GENERAL_KERNEL_BLOCK);
            // check if the pivot is in the range defined by min_ and max_.
            // if not, go to the next until we find one that is in the range.
            while ((isMinReady && (min_ >= pivot1)) || (isMaxReady && (max_ <= pivot1)))
            {
                idx++;
                if (idx >= end)
                {
                    idx = start;
                }
                pivot1 = fetchAs1d(idx, GENERAL_KERNEL_BLOCK);
            }

            if (pivot0 < pivot1 && pivot1 < pivot2)
            {
                pivot0 = pivot1;
            }
            else if (pivot0 < pivot2 && pivot2 < pivot1)
            {
                pivot0 = pivot2;
            }

            // use the pivot to partition the array.
            t = end;
            for (int i = start; i < t; i++)
            {
                tmp = fetchAs1d(i, GENERAL_KERNEL_BLOCK);
                // only consider the element in the range defined by min_ and max_.
                // because others are already ruled out.
                if ((!isMinReady || min_ < tmp) && (!isMaxReady || tmp < max_))
                {
                    if (tmp > pivot0)
                    {
                        numOfGt++;
                    }
                    else if (tmp < pivot0)
                    {
                        numOfLt++;
                    }
                    else
                    {
                        numOfEq++;
                    }
                    if (isAllPreviousOutOfRange)
                    {
                        start                   = i;
                        isAllPreviousOutOfRange = false;
                    }
                    end = i + 1;
                }
            }

            // if the index of median is less than numOfLt,
            // use max_ to rule out elements greater than or equal to pivot.
            if (median < numOfLt)
            {
                max_       = pivot0;
                numOfTaken = numOfTaken + numOfEq + numOfGt;
                isMaxReady = true;
                // if the index of median is in between numOfLt and (numOfLt + numOfEq).
                // the median is found. we are lucky:)
            }
            else if (median < (numOfLt + numOfEq))
            {
                break;
                // if the index of median is greater than (numOfLt + numOfEq),
                // use min_ to rule out elements greater than or equal to pivot.
            }
            else
            {
                min_       = pivot0;
                median     = median - numOfLt - numOfEq;
                numOfTaken = numOfTaken + numOfLt + numOfEq;
                isMinReady = true;
            }
            numOfLt = 0;
            numOfEq = 0;
            numOfGt = 0;
        }
        *dst.ptr(batchIdx, y, x, channel) = pivot0;
    }
}

template<typename T>
__device__ int partition(T *arr, int length, T pvt, int *numOfEq)
{
    T val;
    *numOfEq = 1;
    int i    = 1;
    for (int j = 1; j < length; j++)
    {
        val = arr[j];
        if (val == pvt)
        {
            (*numOfEq) += 1;
        }
        if (val < pvt)
        {
            arr[j] = arr[i];
            arr[i] = val;
            i += 1;
        }
    }
    val        = arr[0];
    arr[0]     = arr[i - 1];
    arr[i - 1] = val;
    return i - 1;
}

template<typename T>
__inline__ __device__ T placePivot(T *arr, int length)
{
    int mid    = length / 2;
    T   pivot0 = arr[0], pivot1 = arr[mid], pivot2 = arr[length - 1];
    if (pivot0 < pivot1 && pivot1 <= pivot2)
    {
        arr[0]   = pivot1;
        arr[mid] = pivot0;
        return pivot1;
    }
    if (pivot0 < pivot2 && pivot2 <= pivot1)
    {
        arr[0]          = pivot2;
        arr[length - 1] = pivot0;
        return pivot2;
    }
    return pivot0;
}

template<typename T>
__global__ void medianForSmallKernel(const Ptr2dVarShapeNHWC<T> src, Ptr2dVarShapeNHWC<T> dst,
                                     const cuda::Tensor2DWrap<int> ksize)
{
    int tx = threadIdx.x, ty = threadIdx.y;
    int blockX   = blockIdx.x * blockDim.x;
    int blockY   = blockIdx.y * blockDim.y;
    int x        = blockX + threadIdx.x;
    int y        = blockY + threadIdx.y;
    int channel  = blockIdx.z % dst.nch;
    int batchIdx = blockIdx.z / dst.nch;
    int h = src.at_rows(batchIdx), w = src.at_cols(batchIdx);
    int kWidth  = *ksize.ptr(batchIdx, 0); //kWidths[batchIdx];
    int kHeight = *ksize.ptr(batchIdx, 1); //kHeights[batchIdx];

    __shared__ T tails[SMALL_KERNEL_BLOCK * SMALL_KERNEL_BLOCK];
    if (x < w && y < h)
    {
        tails[ty * SMALL_KERNEL_BLOCK + tx] = *src.ptr(batchIdx, y, x, channel);
    }
    __syncthreads();

    extern __shared__ char _arrays[];
    int                    length = kWidth * kHeight;
    T                     *arr    = ((T *)_arrays) + ((tx * SMALL_KERNEL_BLOCK) + ty) * length;
    T                      pivot;
    int                    numOfEq, k = length / 2;

    if ((x < w && y < h))
    {
        for (int i = 0; i < length; i++)
        {
            arr[i] = fetchAs1d(i, SMALL_KERNEL_BLOCK);
        }
        while (length > 1)
        {
            pivot      = placePivot(arr, length);
            int middle = partition(arr, length, pivot, &numOfEq);
            if (k < middle)
            {
                length = middle;
            }
            else if (k < (middle + numOfEq))
            {
                *dst.ptr(batchIdx, y, x, channel) = pivot;
                return;
            }
            else
            {
                k      = k - middle - 1;
                length = length - middle - 1;
                arr    = arr + middle + 1;
            }
        }
        *dst.ptr(batchIdx, y, x, channel) = arr[0];
    }
}

#undef fetch_
#undef fetchAs1d

template<typename T>
void median(const IImageBatchVarShapeDataStridedCuda &in, const IImageBatchVarShapeDataStridedCuda &out,
            const ITensorDataStridedCuda &ksize, int maxKHeight, int maxKWidth, hipStream_t stream)
{
    Size2D outMaxSize = out.maxSize();

    int maxWidth  = outMaxSize.w;
    int maxHeight = outMaxSize.h;

    Ptr2dVarShapeNHWC<T> src(in);  //(batch, height, width, channels, (T **) input);
    Ptr2dVarShapeNHWC<T> dst(out); //(batch, height, width, channels, (T **) output);

    cuda::Tensor2DWrap<int> ksizePtr(ksize);

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif

    long unsigned int sharedMemSize = SMALL_KERNEL_BLOCK * SMALL_KERNEL_BLOCK * maxKWidth * maxKHeight * sizeof(T);
    if (sharedMemSize < 48 * 1024)
    {
        dim3 block(SMALL_KERNEL_BLOCK, SMALL_KERNEL_BLOCK);
        dim3 grid(divUp(maxWidth, block.x), divUp(maxHeight, block.y), dst.nch * dst.batches);
        medianForSmallKernel<T><<<grid, block, sharedMemSize, stream>>>(src, dst, ksize);
        checkKernelErrors();
    }
    else
    {
        dim3 block(GENERAL_KERNEL_BLOCK, GENERAL_KERNEL_BLOCK);
        dim3 grid(divUp(maxWidth, block.x), divUp(maxHeight, block.y), dst.nch * dst.batches);
        median<T><<<grid, block, 0, stream>>>(src, dst, ksize);
        checkKernelErrors();
    }

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif
}

MedianBlurVarShape::MedianBlurVarShape(const int maxBatchSize)
    : CudaBaseOp()
    , m_maxBatchSize(maxBatchSize)
{
    if (m_maxBatchSize > 0)
    {
        // {Width, Height} per image in var shape
        size_t totalNumElements = m_maxBatchSize * 2;

        m_kernelSizes.resize(totalNumElements);
        if (m_kernelSizes.size() != totalNumElements)
        {
            LOG_ERROR("Host memory allocation error!!!");
        }
    }
}

MedianBlurVarShape::~MedianBlurVarShape()
{
    m_kernelSizes.clear();
    m_kernelSizes.shrink_to_fit();
}

ErrorCode MedianBlurVarShape::infer(const IImageBatchVarShapeDataStridedCuda &inData,
                                    const IImageBatchVarShapeDataStridedCuda &outData,
                                    const ITensorDataStridedCuda &ksize, hipStream_t stream)
{
    if (m_maxBatchSize <= 0)
    {
        LOG_ERROR(
            "Operator median blur var shape is not initialized properly, maxVarShapeBatchSize: " << m_maxBatchSize);
        return ErrorCode::INVALID_PARAMETER;
    }

    if (m_maxBatchSize < inData.numImages())
    {
        LOG_ERROR("Invalid number of images, it should not exceed " << m_maxBatchSize);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    DataFormat input_format  = helpers::GetLegacyDataFormat(inData);
    DataFormat output_format = helpers::GetLegacyDataFormat(outData);

    if (input_format != output_format)
    {
        LOG_ERROR("Invalid DataFormat between input (" << input_format << ") and output (" << output_format << ")");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataFormat format = input_format;

    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataType data_type = helpers::GetLegacyDataType(inData.uniqueFormat());

    if (!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    int channels = inData.uniqueFormat().numChannels();

    if (channels > 4)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    auto ksizeDataAccess = nvcv::TensorDataAccessStrided::Create(ksize);
    NVCV_ASSERT(ksizeDataAccess);

    // Copy the data to host
    checkCudaErrors(hipMemcpy2DAsync(m_kernelSizes.data(), sizeof(int) * 2, ksizeDataAccess->sampleData(0),
                                      ksizeDataAccess->sampleStride(), sizeof(int) * 2, inData.numImages(),
                                      hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    // Compute the max width & height of kernel sizes
    int maxKHeight = 0, maxKWidth = 0;
    for (int b = 0; b < inData.numImages(); b++)
    {
        int wIndex = b * 2;
        int hIndex = b * 2 + 1;
        if (!(m_kernelSizes[wIndex] > 0 && m_kernelSizes[wIndex] % 2 == 1 && m_kernelSizes[hIndex] > 0
              && m_kernelSizes[hIndex] % 2 == 1))
        {
            LOG_ERROR("Invalid ksize " << m_kernelSizes[wIndex] << " " << m_kernelSizes[hIndex]);
            return ErrorCode::INVALID_PARAMETER;
        }

        if (m_kernelSizes[wIndex] > maxKWidth)
        {
            maxKWidth = m_kernelSizes[wIndex];
        }

        if (m_kernelSizes[hIndex] > maxKHeight)
        {
            maxKHeight = m_kernelSizes[hIndex];
        }
    }

    typedef void (*median_t)(const IImageBatchVarShapeDataStridedCuda &in,
                             const IImageBatchVarShapeDataStridedCuda &out, const ITensorDataStridedCuda &ksize,
                             int maxKHeight, int maxKWidth, hipStream_t stream);

    static const median_t funcs[6] = {
        median<uchar>, 0, median<ushort>, 0, median<int>, median<float>,

    };
    funcs[data_type](inData, outData, ksize, maxKHeight, maxKWidth, stream);
    return SUCCESS;
}

} // namespace nvcv::legacy::cuda_op

#undef GENERAL_KERNEL_BLOCK
#undef SMALL_KERNEL_BLOCK
