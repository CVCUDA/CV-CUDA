#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
 * Copyright (C) 2009-2010, Willow Garage Inc., all rights reserved.
 * Copyright (C) 2014-2015, Itseez Inc., all rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;

namespace nvcv::legacy::cuda_op {

static __device__ __forceinline__ float norm1(const float &a)
{
    return std::abs(a);
}

static __device__ __forceinline__ float norm1(const float2 &a)
{
    return cuda::abs(a.x) + cuda::abs(a.y);
}

static __device__ __forceinline__ float norm1(const float3 &a)
{
    return cuda::abs(a.x) + cuda::abs(a.y) + cuda::abs(a.z);
}

static __device__ __forceinline__ float norm1(const float4 &a)
{
    return cuda::abs(a.x) + cuda::abs(a.y) + cuda::abs(a.z) + cuda::abs(a.w);
}

template<typename T, typename BrdRd>
__global__ void BilateralFilterVarShapeKernel(const BrdRd src, Ptr2dVarShapeNHWC<T> dst,
                                              const cuda::Tensor1DWrap<int>   inDiameter,
                                              const cuda::Tensor1DWrap<float> inSigmaColor,
                                              const cuda::Tensor1DWrap<float> inSigmaSpace)
{
    const int batch_idx = get_batch_idx();
    const int rows      = dst.at_rows(batch_idx);
    const int columns   = dst.at_cols(batch_idx);

    // Preprocessing moved here because tensors are GPU resident
    float sigmaColor = *inSigmaColor.ptr(batch_idx);
    if (sigmaColor <= 0)
    {
        sigmaColor = 1;
    }
    float sigmaSpace = *inSigmaSpace.ptr(batch_idx);
    if (sigmaSpace <= 0)
    {
        sigmaSpace = 1;
    }

    int radius;
    int diameter = *inDiameter.ptr(batch_idx);
    if (diameter <= 0)
    {
        radius = std::roundf(sigmaSpace * 1.5f);
    }
    else
    {
        radius = diameter / 2;
    }
    if (radius < 1)
    {
        radius = 1;
    }
    assert(radius < 10000);

    const int colIdx = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    const int rowIdx = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    using work_type  = cuda::ConvertBaseTypeTo<float, T>;
    int3      coord0{colIdx, rowIdx, batch_idx};
    int3      coord1{colIdx + 1, rowIdx, batch_idx};
    int3      coord2{colIdx, rowIdx + 1, batch_idx};
    int3      coord3{colIdx + 1, rowIdx + 1, batch_idx};
    work_type center0 = cuda::StaticCast<float>(src(coord0.z, coord0.y, coord0.x));
    work_type center1 = cuda::StaticCast<float>(src(coord1.z, coord1.y, coord1.x));
    work_type center2 = cuda::StaticCast<float>(src(coord2.z, coord2.y, coord2.x));
    work_type center3 = cuda::StaticCast<float>(src(coord3.z, coord3.y, coord3.x));

    int       squared_radius    = radius * radius;
    float     space_coefficient = -1 / (2 * sigmaSpace * sigmaSpace);
    float     color_coefficient = -1 / (2 * sigmaColor * sigmaColor);
    work_type numerator0        = cuda::SetAll<work_type>(0);
    work_type numerator1        = cuda::SetAll<work_type>(0);
    work_type numerator2        = cuda::SetAll<work_type>(0);
    work_type numerator3        = cuda::SetAll<work_type>(0);
    float     denominator0      = 0;
    float     denominator1      = 0;
    float     denominator2      = 0;
    float     denominator3      = 0;

    for (int c = colIdx - radius; c < colIdx + radius + 2; c++)
    {
        for (int r = rowIdx - radius; r < rowIdx + radius + 2; r++)
        {
            int t0 = std::abs(c - colIdx), t1 = cuda::abs(r - rowIdx);
            int t2 = std::abs(c - (colIdx + 1)), t3 = cuda::abs(r - (rowIdx + 1));
            int squared_dis0 = t0 * t0 + t1 * t1;
            int squared_dis1 = t2 * t2 + t1 * t1;
            int squared_dis2 = t0 * t0 + t3 * t3;
            int squared_dis3 = t2 * t2 + t3 * t3;

            if (!(squared_dis0 <= squared_radius || squared_dis1 <= squared_radius || squared_dis2 <= squared_radius
                  || squared_dis3 <= squared_radius))
            {
                continue;
            }

            work_type curr = cuda::StaticCast<float>(src(batch_idx, r, c));

            if (squared_dis0 <= squared_radius)
            {
                float e_space       = squared_dis0 * space_coefficient;
                float one_norm_size = norm1(curr - center0);
                float e_color       = one_norm_size * one_norm_size * color_coefficient;
                float weight        = cuda::exp(e_space + e_color);
                denominator0 += weight;
                numerator0 += weight * curr;
            }

            if (squared_dis1 <= squared_radius)
            {
                float e_space       = squared_dis1 * space_coefficient;
                float one_norm_size = norm1(curr - center1);
                float e_color       = one_norm_size * one_norm_size * color_coefficient;
                float weight        = cuda::exp(e_space + e_color);
                denominator1 += weight;
                numerator1 = numerator1 + (weight * curr);
            }

            if (squared_dis2 <= squared_radius)
            {
                float e_space       = squared_dis2 * space_coefficient;
                float one_norm_size = norm1(curr - center2);
                float e_color       = one_norm_size * one_norm_size * color_coefficient;
                float weight        = cuda::exp(e_space + e_color);
                denominator2 += weight;
                numerator2 = numerator2 + (weight * curr);
            }

            if (squared_dis3 <= squared_radius)
            {
                float e_space       = squared_dis3 * space_coefficient;
                float one_norm_size = norm1(curr - center3);
                float e_color       = one_norm_size * one_norm_size * color_coefficient;
                float weight        = cuda::exp(e_space + e_color);
                denominator3 += weight;
                numerator3 = numerator3 + (weight * curr);
            }
        }
    }
    if (colIdx < columns && rowIdx < rows)
    {
        *dst.ptr(coord0.z, coord0.y, coord0.x) = nvcv::cuda::SaturateCast<cuda::BaseType<T>>(numerator0 / denominator0);
    }
    if (colIdx + 1 < columns && rowIdx < rows)
    {
        *dst.ptr(coord1.z, coord1.y, coord1.x) = nvcv::cuda::SaturateCast<cuda::BaseType<T>>(numerator1 / denominator1);
    }
    if (colIdx < columns && rowIdx + 1 < rows)
    {
        *dst.ptr(coord2.z, coord2.y, coord2.x) = nvcv::cuda::SaturateCast<cuda::BaseType<T>>(numerator2 / denominator2);
    }
    if (colIdx + 1 < columns && rowIdx + 1 < rows)
    {
        *dst.ptr(coord3.z, coord3.y, coord3.x) = nvcv::cuda::SaturateCast<cuda::BaseType<T>>(numerator3 / denominator3);
    }
}

template<typename T, template<typename> class Brd>
void BilateralFilterVarShapeCaller(const IImageBatchVarShapeDataStridedCuda &inData,
                                   const IImageBatchVarShapeDataStridedCuda &outData, int batch,
                                   const cuda::Tensor1DWrap<int>   &inDiameter,
                                   const cuda::Tensor1DWrap<float> &inSigmaColor,
                                   const cuda::Tensor1DWrap<float> &inSigmaSpace, hipStream_t stream)
{
    Ptr2dVarShapeNHWC<T> src(inData);
    Ptr2dVarShapeNHWC<T> dst(outData);
    using work_type = cuda::ConvertBaseTypeTo<float, T>;
    Brd<work_type>                                     brd(0, 0, cuda::SetAll<work_type>(0.0f));
    BorderReader<Ptr2dVarShapeNHWC<T>, Brd<work_type>> brdSrc(src, brd);
    Size2D                                             outMaxSize = outData.maxSize();
    dim3                                               block(8, 8);
    dim3 grid(divUp(outMaxSize.w, block.x * 2), divUp(outMaxSize.h, block.y * 2), batch);

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif

    BilateralFilterVarShapeKernel<<<grid, block, 0, stream>>>(brdSrc, dst, inDiameter, inSigmaColor, inSigmaSpace);

#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif
}

ErrorCode BilateralFilterVarShape::infer(const IImageBatchVarShapeDataStridedCuda &inData,
                                         const IImageBatchVarShapeDataStridedCuda &outData,
                                         const ITensorDataStridedCuda             &diameterData,
                                         const ITensorDataStridedCuda             &sigmaColorData,
                                         const ITensorDataStridedCuda &sigmaSpaceData, NVCVBorderType borderMode,
                                         hipStream_t stream)
{
    cuda_op::DataFormat input_format  = GetLegacyDataFormat(inData);
    cuda_op::DataFormat output_format = GetLegacyDataFormat(outData);

    if (input_format != output_format)
    {
        LOG_ERROR("Input data format (" << input_format << ") and output data format (" << output_format
                                        << ") must be the same.");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if ((input_format != kNHWC) && (input_format != kHWC))
    {
        LOG_ERROR("Invalid DataFormat both Input and Output must be kHWC or kNHWC");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!inData.uniqueFormat())
    {
        LOG_ERROR("Images in the input varshape must all have the same format");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (inData.uniqueFormat() != outData.uniqueFormat())
    {
        LOG_ERROR("Input and Output formats must be same input format ="
                  << helpers::GetLegacyDataType(inData.uniqueFormat())
                  << " output format = " << helpers::GetLegacyDataType(outData.uniqueFormat()));
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!(borderMode == NVCV_BORDER_CONSTANT || borderMode == NVCV_BORDER_REPLICATE || borderMode == NVCV_BORDER_REFLECT
          || borderMode == NVCV_BORDER_WRAP || borderMode == NVCV_BORDER_REFLECT101))
    {
        LOG_ERROR("[Error] Invalid borderMode " << borderMode);
        return ErrorCode::INVALID_PARAMETER;
    }

    DataType data_type = GetLegacyDataType(outData.uniqueFormat());
    if (!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_16S || data_type == kCV_32S
          || data_type == kCV_32F))
    {
        LOG_ERROR("[Error] Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType diameter_data_type = GetLegacyDataType(diameterData.dtype());
    if (diameter_data_type != kCV_32S)
    {
        LOG_ERROR("[Error] Invalid diameterData DataType " << diameter_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType sigmaColor_data_type = GetLegacyDataType(sigmaColorData.dtype());
    if (sigmaColor_data_type != kCV_32F)
    {
        LOG_ERROR("[Error] Invalid sigmaColorData DataType " << sigmaColor_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType sigmaSpace_data_type = GetLegacyDataType(sigmaSpaceData.dtype());
    if (sigmaSpace_data_type != kCV_32F)
    {
        LOG_ERROR("[Error] Invalid sigmaSpaceData DataType " << sigmaSpace_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    if (inData.numImages() != outData.numImages())
    {
        LOG_ERROR("Input and Output data must have the same number of images (" << inData.numImages()
                                                                                << " != " << outData.numImages());
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    int batch    = inData.numImages();
    int channels = inData.uniqueFormat().numChannels();
    if (channels > 4 || channels < 1)
    {
        LOG_ERROR("Invalid channel number ch = " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    // Create Tensor wrappers for parameter arrays
    cuda::Tensor1DWrap<int>   inDiameter(diameterData);
    cuda::Tensor1DWrap<float> inSigmaColor(sigmaColorData);
    cuda::Tensor1DWrap<float> inSigmaSpace(sigmaSpaceData);

    typedef void (*bilateral_filter_var_shape_t)(
        const IImageBatchVarShapeDataStridedCuda &inData, const IImageBatchVarShapeDataStridedCuda &outData, int batch,
        const cuda::Tensor1DWrap<int> &inDiameter, const cuda::Tensor1DWrap<float> &inSigmaColor,
        const cuda::Tensor1DWrap<float> &inSigmaSpace, hipStream_t stream);

    // All templated functions instantiated here to remove one level of indirection that just hides the same lookup
    // table in 5 parts
    static const bilateral_filter_var_shape_t funcs[5][6][4] = {
        {
         {BilateralFilterVarShapeCaller<uchar, BrdConstant>, BilateralFilterVarShapeCaller<uchar2, BrdConstant>,
         BilateralFilterVarShapeCaller<uchar3, BrdConstant>, BilateralFilterVarShapeCaller<uchar4, BrdConstant>},
         {BilateralFilterVarShapeCaller<char, BrdConstant>, BilateralFilterVarShapeCaller<char2, BrdConstant>,
         BilateralFilterVarShapeCaller<char3, BrdConstant>, BilateralFilterVarShapeCaller<char4, BrdConstant>},
         {BilateralFilterVarShapeCaller<ushort, BrdConstant>, BilateralFilterVarShapeCaller<ushort2, BrdConstant>,
         BilateralFilterVarShapeCaller<ushort3, BrdConstant>, BilateralFilterVarShapeCaller<ushort4, BrdConstant>},
         {BilateralFilterVarShapeCaller<short, BrdConstant>, BilateralFilterVarShapeCaller<short2, BrdConstant>,
         BilateralFilterVarShapeCaller<short3, BrdConstant>, BilateralFilterVarShapeCaller<short4, BrdConstant>},
         {BilateralFilterVarShapeCaller<int, BrdConstant>, BilateralFilterVarShapeCaller<int2, BrdConstant>,
         BilateralFilterVarShapeCaller<int3, BrdConstant>, BilateralFilterVarShapeCaller<int4, BrdConstant>},
         {BilateralFilterVarShapeCaller<float, BrdConstant>, BilateralFilterVarShapeCaller<float2, BrdConstant>,
         BilateralFilterVarShapeCaller<float3, BrdConstant>, BilateralFilterVarShapeCaller<float4, BrdConstant>},
         },
        {
         {BilateralFilterVarShapeCaller<uchar, BrdReplicate>, BilateralFilterVarShapeCaller<uchar2, BrdReplicate>,
         BilateralFilterVarShapeCaller<uchar3, BrdReplicate>, BilateralFilterVarShapeCaller<uchar4, BrdReplicate>},
         {BilateralFilterVarShapeCaller<char, BrdReplicate>, BilateralFilterVarShapeCaller<char2, BrdReplicate>,
         BilateralFilterVarShapeCaller<char3, BrdReplicate>, BilateralFilterVarShapeCaller<char4, BrdReplicate>},
         {BilateralFilterVarShapeCaller<ushort, BrdReplicate>, BilateralFilterVarShapeCaller<ushort2, BrdReplicate>,
         BilateralFilterVarShapeCaller<ushort3, BrdReplicate>,
         BilateralFilterVarShapeCaller<ushort4, BrdReplicate>},
         {BilateralFilterVarShapeCaller<short, BrdReplicate>, BilateralFilterVarShapeCaller<short2, BrdReplicate>,
         BilateralFilterVarShapeCaller<short3, BrdReplicate>, BilateralFilterVarShapeCaller<short4, BrdReplicate>},
         {BilateralFilterVarShapeCaller<int, BrdReplicate>, BilateralFilterVarShapeCaller<int2, BrdReplicate>,
         BilateralFilterVarShapeCaller<int3, BrdReplicate>, BilateralFilterVarShapeCaller<int4, BrdReplicate>},
         {BilateralFilterVarShapeCaller<float, BrdReplicate>, BilateralFilterVarShapeCaller<float2, BrdReplicate>,
         BilateralFilterVarShapeCaller<float3, BrdReplicate>, BilateralFilterVarShapeCaller<float4, BrdReplicate>},
         },
        {
         {BilateralFilterVarShapeCaller<uchar, BrdReflect>, BilateralFilterVarShapeCaller<uchar2, BrdReflect>,
         BilateralFilterVarShapeCaller<uchar3, BrdReflect>, BilateralFilterVarShapeCaller<uchar4, BrdReflect>},
         {BilateralFilterVarShapeCaller<char, BrdReflect>, BilateralFilterVarShapeCaller<char2, BrdReflect>,
         BilateralFilterVarShapeCaller<char3, BrdReflect>, BilateralFilterVarShapeCaller<char4, BrdReflect>},
         {BilateralFilterVarShapeCaller<ushort, BrdReflect>, BilateralFilterVarShapeCaller<ushort2, BrdReflect>,
         BilateralFilterVarShapeCaller<ushort3, BrdReflect>, BilateralFilterVarShapeCaller<ushort4, BrdReflect>},
         {BilateralFilterVarShapeCaller<short, BrdReflect>, BilateralFilterVarShapeCaller<short2, BrdReflect>,
         BilateralFilterVarShapeCaller<short3, BrdReflect>, BilateralFilterVarShapeCaller<short4, BrdReflect>},
         {BilateralFilterVarShapeCaller<int, BrdReflect>, BilateralFilterVarShapeCaller<int2, BrdReflect>,
         BilateralFilterVarShapeCaller<int3, BrdReflect>, BilateralFilterVarShapeCaller<int4, BrdReflect>},
         {BilateralFilterVarShapeCaller<float, BrdReflect>, BilateralFilterVarShapeCaller<float2, BrdReflect>,
         BilateralFilterVarShapeCaller<float3, BrdReflect>, BilateralFilterVarShapeCaller<float4, BrdReflect>},
         },
        {
         {BilateralFilterVarShapeCaller<uchar, BrdWrap>, BilateralFilterVarShapeCaller<uchar2, BrdWrap>,
         BilateralFilterVarShapeCaller<uchar3, BrdWrap>, BilateralFilterVarShapeCaller<uchar4, BrdWrap>},
         {BilateralFilterVarShapeCaller<char, BrdWrap>, BilateralFilterVarShapeCaller<char2, BrdWrap>,
         BilateralFilterVarShapeCaller<char3, BrdWrap>, BilateralFilterVarShapeCaller<char4, BrdWrap>},
         {BilateralFilterVarShapeCaller<ushort, BrdWrap>, BilateralFilterVarShapeCaller<ushort2, BrdWrap>,
         BilateralFilterVarShapeCaller<ushort3, BrdWrap>, BilateralFilterVarShapeCaller<ushort4, BrdWrap>},
         {BilateralFilterVarShapeCaller<short, BrdWrap>, BilateralFilterVarShapeCaller<short2, BrdWrap>,
         BilateralFilterVarShapeCaller<short3, BrdWrap>, BilateralFilterVarShapeCaller<short4, BrdWrap>},
         {BilateralFilterVarShapeCaller<int, BrdWrap>, BilateralFilterVarShapeCaller<int2, BrdWrap>,
         BilateralFilterVarShapeCaller<int3, BrdWrap>, BilateralFilterVarShapeCaller<int4, BrdWrap>},
         {BilateralFilterVarShapeCaller<float, BrdWrap>, BilateralFilterVarShapeCaller<float2, BrdWrap>,
         BilateralFilterVarShapeCaller<float3, BrdWrap>, BilateralFilterVarShapeCaller<float4, BrdWrap>},
         },
        {
         {BilateralFilterVarShapeCaller<uchar, BrdReflect101>, BilateralFilterVarShapeCaller<uchar2, BrdReflect101>,
         BilateralFilterVarShapeCaller<uchar3, BrdReflect101>,
         BilateralFilterVarShapeCaller<uchar4, BrdReflect101>},
         {BilateralFilterVarShapeCaller<char, BrdReflect101>, BilateralFilterVarShapeCaller<char2, BrdReflect101>,
         BilateralFilterVarShapeCaller<char3, BrdReflect101>, BilateralFilterVarShapeCaller<char4, BrdReflect101>},
         {BilateralFilterVarShapeCaller<ushort, BrdReflect101>,
         BilateralFilterVarShapeCaller<ushort2, BrdReflect101>,
         BilateralFilterVarShapeCaller<ushort3, BrdReflect101>,
         BilateralFilterVarShapeCaller<ushort4, BrdReflect101>},
         {BilateralFilterVarShapeCaller<short, BrdReflect101>, BilateralFilterVarShapeCaller<short2, BrdReflect101>,
         BilateralFilterVarShapeCaller<short3, BrdReflect101>,
         BilateralFilterVarShapeCaller<short4, BrdReflect101>},
         {BilateralFilterVarShapeCaller<int, BrdReflect101>, BilateralFilterVarShapeCaller<int2, BrdReflect101>,
         BilateralFilterVarShapeCaller<int3, BrdReflect101>, BilateralFilterVarShapeCaller<int4, BrdReflect101>},
         {BilateralFilterVarShapeCaller<float, BrdReflect101>, BilateralFilterVarShapeCaller<float2, BrdReflect101>,
         BilateralFilterVarShapeCaller<float3, BrdReflect101>,
         BilateralFilterVarShapeCaller<float4, BrdReflect101>},
         },
    };

    funcs[borderMode][data_type][channels - 1](inData, outData, batch, inDiameter, inSigmaColor, inSigmaSpace, stream);
    return ErrorCode::SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
