#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

#define BLOCK 32
using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;
using namespace nvcv::cuda;

template<class Transform, class Filter, typename T>
__global__ void warp(const Filter src, Ptr2dNHWC<T> dst, Transform transform)
{
    const int               x         = blockDim.x * blockIdx.x + threadIdx.x;
    const int               y         = blockDim.y * blockIdx.y + threadIdx.y;
    const int               lid       = get_lid();
    const int               batch_idx = get_batch_idx();
    extern __shared__ float coeff[];
    if (lid < 9)
    {
        coeff[lid] = transform.xform[lid];
    }
    __syncthreads();
    if (x < dst.cols && y < dst.rows)
    {
        const float2 coord        = Transform::calcCoord(coeff, x, y);
        *dst.ptr(batch_idx, y, x) = nvcv::cuda::SaturateCast<nvcv::cuda::BaseType<T>>(src(batch_idx, coord.y, coord.x));
    }
}

template<class Transform, template<typename> class Filter, template<typename> class B, typename T>
struct WarpDispatcher
{
    static void call(const Ptr2dNHWC<T> src, Ptr2dNHWC<T> dst, Transform transform, const float4 borderValue,
                     hipStream_t stream)
    {
        using work_type = nvcv::cuda::ConvertBaseTypeTo<float, T>;

        dim3 block(BLOCK, BLOCK / 4);
        dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y), dst.batches);

        work_type                                borderVal = nvcv::cuda::DropCast<NumComponents<T>>(borderValue);
        B<work_type>                             brd(src.rows, src.cols, borderVal);
        BorderReader<Ptr2dNHWC<T>, B<work_type>> brdSrc(src, brd);
        Filter<BorderReader<Ptr2dNHWC<T>, B<work_type>>> filter_src(brdSrc);
        size_t                                           smem_size = 9 * sizeof(float);
        warp<Transform><<<grid, block, smem_size, stream>>>(filter_src, dst, transform);
        checkKernelErrors();
    }
};

template<class Transform, typename T>
void warp_caller(const Ptr2dNHWC<T> src, Ptr2dNHWC<T> dst, Transform transform, int interpolation, int borderMode,
                 const float4 borderValue, hipStream_t stream)
{
    typedef void (*func_t)(const Ptr2dNHWC<T> src, Ptr2dNHWC<T> dst, Transform transform, const float4 borderValue,
                           hipStream_t stream);

    static const func_t funcs[3][5] = {
        {WarpDispatcher<Transform,  PointFilter, BrdConstant, T>::call,
         WarpDispatcher<Transform,  PointFilter, BrdReplicate, T>::call,
         WarpDispatcher<Transform,  PointFilter, BrdReflect, T>::call,
         WarpDispatcher<Transform,  PointFilter, BrdWrap, T>::call,
         WarpDispatcher<Transform,  PointFilter, BrdReflect101, T>::call},
        {WarpDispatcher<Transform, LinearFilter, BrdConstant, T>::call,
         WarpDispatcher<Transform, LinearFilter, BrdReplicate, T>::call,
         WarpDispatcher<Transform, LinearFilter, BrdReflect, T>::call,
         WarpDispatcher<Transform, LinearFilter, BrdWrap, T>::call,
         WarpDispatcher<Transform, LinearFilter, BrdReflect101, T>::call},
        {WarpDispatcher<Transform,  CubicFilter, BrdConstant, T>::call,
         WarpDispatcher<Transform,  CubicFilter, BrdReplicate, T>::call,
         WarpDispatcher<Transform,  CubicFilter, BrdReflect, T>::call,
         WarpDispatcher<Transform,  CubicFilter, BrdWrap, T>::call,
         WarpDispatcher<Transform,  CubicFilter, BrdReflect101, T>::call}
    };

    funcs[interpolation][borderMode](src, dst, transform, borderValue, stream);
}

template<typename T>
void warpAffine(const nvcv::TensorDataAccessStridedImagePlanar &inData,
                const nvcv::TensorDataAccessStridedImagePlanar &outData, WarpAffineTransform transform,
                const int interpolation, int borderMode, const float4 borderValue, hipStream_t stream)
{
    Ptr2dNHWC<T> src_ptr(inData);
    Ptr2dNHWC<T> dst_ptr(outData);
    warp_caller<WarpAffineTransform, T>(src_ptr, dst_ptr, transform, interpolation, borderMode, borderValue, stream);
}

template<typename T>
void warpPerspective(const nvcv::TensorDataAccessStridedImagePlanar &inData,
                     const nvcv::TensorDataAccessStridedImagePlanar &outData, PerspectiveTransform transform,
                     const int interpolation, int borderMode, const float4 borderValue, hipStream_t stream)
{
    Ptr2dNHWC<T> src_ptr(inData);
    Ptr2dNHWC<T> dst_ptr(outData);
    warp_caller<PerspectiveTransform, T>(src_ptr, dst_ptr, transform, interpolation, borderMode, borderValue, stream);
}

static void invertMat(const float *M, float *h_aCoeffs)
{
    // M is stored in row-major format M[0,0], M[0,1], M[0,2], M[1,0], M[1,1], M[1,2]
    float den    = M[0] * M[4] - M[1] * M[3];
    den          = std::abs(den) > 1e-5 ? 1. / den : .0;
    h_aCoeffs[0] = (float)M[5] * den;
    h_aCoeffs[1] = (float)-M[1] * den;
    h_aCoeffs[2] = (float)(M[1] * M[5] - M[4] * M[2]) * den;
    h_aCoeffs[3] = (float)-M[3] * den;
    h_aCoeffs[4] = (float)M[0] * den;
    h_aCoeffs[5] = (float)(M[3] * M[2] - M[0] * M[5]) * den;
}

namespace nvcv::legacy::cuda_op {

ErrorCode WarpAffine::infer(const ITensorDataStridedCuda &inData, const ITensorDataStridedCuda &outData,
                            const float *xform, const int32_t flags, const NVCVBorderType borderMode,
                            const float4 borderValue, hipStream_t stream)
{
    DataFormat input_format  = GetLegacyDataFormat(inData.layout());
    DataFormat output_format = GetLegacyDataFormat(outData.layout());

    if (input_format != output_format)
    {
        LOG_ERROR("Invalid DataFormat between input (" << input_format << ") and output (" << output_format << ")");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataFormat format = input_format;

    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto inAccess = TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);

    auto outAccess = TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    cuda_op::DataType  data_type   = GetLegacyDataType(inData.dtype());
    cuda_op::DataShape input_shape = GetLegacyDataShape(inAccess->infoShape());

    int       channels      = input_shape.C;
    const int interpolation = flags & NVCV_INTERP_MAX;

    if (channels > 4)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    if (!(data_type == kCV_8U || data_type == kCV_8S || data_type == kCV_16U || data_type == kCV_16S
          || data_type == kCV_32S || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    NVCV_ASSERT(interpolation == NVCV_INTERP_NEAREST || interpolation == NVCV_INTERP_LINEAR
                || interpolation == NVCV_INTERP_CUBIC);
    NVCV_ASSERT(borderMode == NVCV_BORDER_REFLECT101 || borderMode == NVCV_BORDER_REPLICATE
                || borderMode == NVCV_BORDER_CONSTANT || borderMode == NVCV_BORDER_REFLECT
                || borderMode == NVCV_BORDER_WRAP);

    typedef void (*func_t)(const nvcv::TensorDataAccessStridedImagePlanar &inData,
                           const nvcv::TensorDataAccessStridedImagePlanar &outData, WarpAffineTransform transform,
                           const int interpolation, int borderMode, const float4 borderValue, hipStream_t stream);

    static const func_t funcs[6][4] = {
        { warpAffine<uchar>, 0,  warpAffine<uchar3>,  warpAffine<uchar4>},
        {                 0, 0,                   0,                   0},
        {warpAffine<ushort>, 0, warpAffine<ushort3>, warpAffine<ushort4>},
        { warpAffine<short>, 0,  warpAffine<short3>,  warpAffine<short4>},
        {                 0, 0,                   0,                   0},
        { warpAffine<float>, 0,  warpAffine<float3>,  warpAffine<float4>}
    };

    const func_t func = funcs[data_type][channels - 1];
    NVCV_ASSERT(func != 0);

    WarpAffineTransform transform;

    // initialize affine transform
    for (int i = 0; i < 9; i++)
    {
        transform.xform[i] = i < 6 ? (float)(xform[i]) : 0.0f;
    }

    if (flags & NVCV_WARP_INVERSE_MAP)
    {
        invertMat(xform, transform.xform);
    }

    func(*inAccess, *outAccess, transform, interpolation, borderMode, borderValue, stream);

    return ErrorCode::SUCCESS;
}

size_t WarpPerspective::calBufferSize(DataShape max_input_shape, DataShape max_output_shape, DataType max_data_type)
{
    return 9 * sizeof(float);
}

ErrorCode WarpPerspective::infer(const ITensorDataStridedCuda &inData, const ITensorDataStridedCuda &outData,
                                 const float *transMatrix, const int32_t flags, const NVCVBorderType borderMode,
                                 const float4 borderValue, hipStream_t stream)
{
    DataFormat input_format  = GetLegacyDataFormat(inData.layout());
    DataFormat output_format = GetLegacyDataFormat(outData.layout());

    if (input_format != output_format)
    {
        LOG_ERROR("Invalid DataFormat between input (" << input_format << ") and output (" << output_format << ")");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataFormat format = input_format;

    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto inAccess = TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);

    auto outAccess = TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    cuda_op::DataType  data_type   = GetLegacyDataType(inData.dtype());
    cuda_op::DataShape input_shape = GetLegacyDataShape(inAccess->infoShape());

    int       channels      = input_shape.C;
    const int interpolation = flags & NVCV_INTERP_MAX;

    if (channels > 4)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    if (!(data_type == kCV_8U || data_type == kCV_8S || data_type == kCV_16U || data_type == kCV_16S
          || data_type == kCV_32S || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    NVCV_ASSERT(interpolation == NVCV_INTERP_NEAREST || interpolation == NVCV_INTERP_LINEAR
                || interpolation == NVCV_INTERP_CUBIC);
    NVCV_ASSERT(borderMode == NVCV_BORDER_REFLECT101 || borderMode == NVCV_BORDER_REPLICATE
                || borderMode == NVCV_BORDER_CONSTANT || borderMode == NVCV_BORDER_REFLECT
                || borderMode == NVCV_BORDER_WRAP);

    typedef void (*func_t)(const nvcv::TensorDataAccessStridedImagePlanar &inData,
                           const nvcv::TensorDataAccessStridedImagePlanar &outData, PerspectiveTransform transform,
                           const int interpolation, int borderMode, const float4 borderValue, hipStream_t stream);

    static const func_t funcs[6][4] = {
        {      warpPerspective<uchar>,  0 /*warpPerspective<uchar2>*/,      warpPerspective<uchar3>,warpPerspective<uchar4>                                                                                                    },
        {0 /*warpPerspective<schar>*/,   0 /*warpPerspective<char2>*/, 0 /*warpPerspective<char3>*/,
         0 /*warpPerspective<char4>*/                                                                                        },
        {     warpPerspective<ushort>, 0 /*warpPerspective<ushort2>*/,     warpPerspective<ushort3>, warpPerspective<ushort4>},
        {      warpPerspective<short>,  0 /*warpPerspective<short2>*/,      warpPerspective<short3>,  warpPerspective<short4>},
        {  0 /*warpPerspective<int>*/,    0 /*warpPerspective<int2>*/,  0 /*warpPerspective<int3>*/,
         0 /*warpPerspective<int4>*/                                                                                         },
        {      warpPerspective<float>,  0 /*warpPerspective<float2>*/,      warpPerspective<float3>,  warpPerspective<float4>}
    };

    const func_t func = funcs[data_type][channels - 1];
    NVCV_ASSERT(func != 0);

    PerspectiveTransform transform(transMatrix);

    if (flags & NVCV_WARP_INVERSE_MAP)
    {
        cuda::math::Matrix<float, 3, 3> tempMatrixForInverse;

        tempMatrixForInverse[0][0] = (float)(transMatrix[0]);
        tempMatrixForInverse[0][1] = (float)(transMatrix[1]);
        tempMatrixForInverse[0][2] = (float)(transMatrix[2]);
        tempMatrixForInverse[1][0] = (float)(transMatrix[3]);
        tempMatrixForInverse[1][1] = (float)(transMatrix[4]);
        tempMatrixForInverse[1][2] = (float)(transMatrix[5]);
        tempMatrixForInverse[2][0] = (float)(transMatrix[6]);
        tempMatrixForInverse[2][1] = (float)(transMatrix[7]);
        tempMatrixForInverse[2][2] = (float)(transMatrix[8]);

        math::inv_inplace(tempMatrixForInverse);

        transform.xform[0] = tempMatrixForInverse[0][0];
        transform.xform[1] = tempMatrixForInverse[0][1];
        transform.xform[2] = tempMatrixForInverse[0][2];
        transform.xform[3] = tempMatrixForInverse[1][0];
        transform.xform[4] = tempMatrixForInverse[1][1];
        transform.xform[5] = tempMatrixForInverse[1][2];
        transform.xform[6] = tempMatrixForInverse[2][0];
        transform.xform[7] = tempMatrixForInverse[2][1];
        transform.xform[8] = tempMatrixForInverse[2][2];
    }

    func(*inAccess, *outAccess, transform, interpolation, borderMode, borderValue, stream);

    return ErrorCode::SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
