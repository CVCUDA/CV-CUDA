#include "hip/hip_runtime.h"
/* Copyright (c) 2021-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * SPDX-FileCopyrightText: NVIDIA CORPORATION & AFFILIATES
 * SPDX-License-Identifier: Apache-2.0
 *
 * Copyright (C) 2021-2022, Bytedance Inc. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

namespace nvcv::legacy::cuda_op {

template<class SrcWrapper, class DstWrapper, class VecWrapper>
__global__ void padAndStack(SrcWrapper src, DstWrapper dst, VecWrapper topVec, VecWrapper leftVec, int2 dstSize)
{
    int3 dstCoord = cuda::StaticCast<int>(blockDim * blockIdx + threadIdx);

    const int top  = *topVec.ptr(0, 0, dstCoord.z);
    const int left = *leftVec.ptr(0, 0, dstCoord.z);

    int3 srcCoord = {dstCoord.x - left, dstCoord.y - top, dstCoord.z};

    if (dstCoord.x < dstSize.x && dstCoord.y < dstSize.y)
    {
        dst[dstCoord] = src[srcCoord];
    }
}

template<typename T, NVCVBorderType B>
void padAndStackCaller(const ImageBatchVarShapeDataStridedCuda &inData, const TensorDataStridedCuda &outData,
                       const TensorDataStridedCuda &top, const TensorDataStridedCuda &left, const float borderValue,
                       hipStream_t stream)
{
    cuda::BorderVarShapeWrap<const T, B> src(inData, cuda::SetAll<T>(borderValue));

    auto dst = cuda::CreateTensorWrapNHW<T>(outData);

    auto topVec  = cuda::CreateTensorWrapNHW<const int>(top);
    auto leftVec = cuda::CreateTensorWrapNHW<const int>(left);

    auto outAccess = TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    int2 dstSize{outAccess->numCols(), outAccess->numRows()};

    dim3 block(16, 16);
    dim3 grid(divUp(dstSize.x, block.x), divUp(dstSize.y, block.y), outAccess->numSamples());

    padAndStack<<<grid, block, 0, stream>>>(src, dst, topVec, leftVec, dstSize);
}

template<typename T>
void padAndStack(const ImageBatchVarShapeDataStridedCuda &inData, const TensorDataStridedCuda &outData,
                 const TensorDataStridedCuda &top, const TensorDataStridedCuda &left, const NVCVBorderType borderMode,
                 const float borderValue, hipStream_t stream)
{
    typedef void (*padAndStack_caller)(const ImageBatchVarShapeDataStridedCuda &inData,
                                       const TensorDataStridedCuda &outData, const TensorDataStridedCuda &top,
                                       const TensorDataStridedCuda &left, const float borderValue, hipStream_t stream);

    static const padAndStack_caller funcs[]
        = {padAndStackCaller<T, NVCV_BORDER_CONSTANT>, padAndStackCaller<T, NVCV_BORDER_REPLICATE>,
           padAndStackCaller<T, NVCV_BORDER_REFLECT>, padAndStackCaller<T, NVCV_BORDER_WRAP>,
           padAndStackCaller<T, NVCV_BORDER_REFLECT101>};

    funcs[borderMode](inData, outData, top, left, borderValue, stream);
}

size_t PadAndStack::calBufferSize(int batch_size)
{
    return 0;
}

ErrorCode PadAndStack::infer(const ImageBatchVarShapeDataStridedCuda &inData, const TensorDataStridedCuda &outData,
                             const TensorDataStridedCuda &top, const TensorDataStridedCuda &left,
                             const NVCVBorderType borderMode, const float borderValue, hipStream_t stream)
{
    DataFormat format    = helpers::GetLegacyDataFormat(outData.layout());
    DataType   data_type = helpers::GetLegacyDataType(outData.dtype());

    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    if (!(borderMode == NVCV_BORDER_REFLECT101 || borderMode == NVCV_BORDER_REPLICATE
          || borderMode == NVCV_BORDER_CONSTANT || borderMode == NVCV_BORDER_REFLECT || borderMode == NVCV_BORDER_WRAP))
    {
        LOG_ERROR("Invalid borderMode " << borderMode);
        return ErrorCode::INVALID_PARAMETER;
    }

    if (!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_16S || data_type == kCV_32S
          || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    auto outAccess = TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    DataType   left_data_type = helpers::GetLegacyDataType(left.dtype());
    DataFormat left_format    = helpers::GetLegacyDataFormat(left.layout());
    if (left_data_type != kCV_32S)
    {
        LOG_ERROR("Invalid Left DataType " << left_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }
    if (!(left_format == kNHWC || left_format == kHWC))
    {
        LOG_ERROR("Invalid Left DataFormat " << left_format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto leftAccess = TensorDataAccessStridedImagePlanar::Create(left);
    if (!leftAccess)
    {
        return ErrorCode::INVALID_DATA_TYPE;
    }

    DataType   top_data_type = helpers::GetLegacyDataType(top.dtype());
    DataFormat top_format    = helpers::GetLegacyDataFormat(top.layout());
    if (top_data_type != kCV_32S)
    {
        LOG_ERROR("Invalid Top DataType " << top_data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }
    if (!(top_format == kNHWC || top_format == kHWC))
    {
        LOG_ERROR("Invalid Top DataFormat " << top_format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto topAccess = TensorDataAccessStridedImagePlanar::Create(top);
    if (!topAccess)
    {
        return ErrorCode::INVALID_DATA_TYPE;
    }

    const int channels = outAccess->numChannels();
    if (channels > 4)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    typedef void (*func_t)(const ImageBatchVarShapeDataStridedCuda &inData, const TensorDataStridedCuda &outData,
                           const TensorDataStridedCuda &top, const TensorDataStridedCuda &left,
                           const NVCVBorderType borderMode, const float borderValue, hipStream_t stream);

    static const func_t funcs[6][4] = {
        { padAndStack<uchar1>, padAndStack<uchar2>,  padAndStack<uchar3>,  padAndStack<uchar4>},
        {                   0,                   0,                    0,                    0},
        {padAndStack<ushort1>,                   0, padAndStack<ushort3>, padAndStack<ushort4>},
        { padAndStack<short1>,                   0,  padAndStack<short3>,  padAndStack<short4>},
        {   padAndStack<int1>,                   0,    padAndStack<int3>,    padAndStack<int4>},
        { padAndStack<float1>,                   0,  padAndStack<float3>,  padAndStack<float4>}
    };

    const func_t func = funcs[data_type][channels - 1];
    NVCV_ASSERT(func != 0);

    func(inData, outData, top, left, borderMode, borderValue, stream);

    return SUCCESS;
}

} // namespace nvcv::legacy::cuda_op
