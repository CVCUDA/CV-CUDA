#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

//$$$ replace these with the new (non-legacy) nvcv approach

#include "CvCudaLegacy.h"
#include "CvCudaLegacyHelpers.hpp"

#include "CvCudaUtils.cuh"

#include <nvcv/cuda/MathWrappers.hpp>

using namespace nvcv::legacy::cuda_op;
using namespace nvcv::legacy::helpers;

namespace nvcv::legacy::cuda_op {

//private internal API

#define MAX_BUFFER_BYTES 128 //multiple of 4 for word-aligned read, multiple of 16 for cacheline alignment (float4)
#define MAX_BUFFER_WORDS (MAX_BUFFER_BYTES / 4) //extra bytes for cache alignment

#define LEGACY_BICUBIC_MATH //apparently the legacy code has an abs() that needs to be matched

#define CACHE_MEMORY_ALIGNMENT 15 //this is 'M' for _cacheAlignedBufferedRead

//legal values for CACHE_MEMORY_ALIGNMENT are:
// 31: 256-bit alignment
// 15: 128-bit alignment <-- should be ideal for Ampere
//  7:  64-bit alignment
//  3:  32-bit alignment (word)
//  0:  disable buffering
template<typename T, size_t M>
inline __device__ T *_cacheAlignedBufferedRead(cuda::Tensor3DWrap<const T> srcImage, int2 srcSize, uint *pReadBuffer,
                                               uint nReadBufferWordsMax, int nBatch, int nYPos, int nXPosMin,
                                               int nXPosMax)
{
    const T *lineStartPtr = srcImage.ptr(nBatch, nYPos); //do not access prior to this address
    const T *pixSrcPtr    = &lineStartPtr[nXPosMin];
    if (M == 0)
        return (T *)pixSrcPtr; //return GMEM pointer instead
    else
    {
        uint     *memSrcPtr       = (uint *)(((size_t)pixSrcPtr) & (~M)); //(M+1) byte alignment
        const T  *pixBeyondPtr    = &lineStartPtr[nXPosMax + 1];
        const int functionalWidth = ((size_t)pixBeyondPtr + M) & (~M) - ((size_t)lineStartPtr);
        const int nWordsToRead    = (((size_t)pixBeyondPtr + M) & (~M) - (size_t)memSrcPtr) / 4;

        if (((size_t)memSrcPtr < (size_t)lineStartPtr) || (srcSize.x * sizeof(T) < functionalWidth)
            || (nWordsToRead > nReadBufferWordsMax))
            return (T *)pixSrcPtr; //return GMEM pointer instead if running off the image
        else
        {                                             //copy out source data, aligned based upon M (31, 15, 7, 3)
            const int skew = ((size_t)pixSrcPtr) & M; //byte offset for nXPosMin
            int       i    = 0;
            if (M >= 31) //256-bit align, 32 bytes at a time
                for (; i < nWordsToRead; i += 8) *((double4 *)(&pReadBuffer[i])) = *((double4 *)(&memSrcPtr[i]));
            if (M == 15) //128-bit align, 16 bytes at a time
                for (; i < nWordsToRead; i += 4) *((float4 *)(&pReadBuffer[i])) = *((float4 *)(&memSrcPtr[i]));
            if (M == 7) //64-bit align, 8 bytes at a time
                for (; i < nWordsToRead; i += 2) *((float2 *)(&pReadBuffer[i])) = *((float2 *)(&memSrcPtr[i]));
            //32-bit align, 4 bytes at a time
            for (; i < nWordsToRead; ++i) pReadBuffer[i] = memSrcPtr[i];

            return (T *)(((size_t)pReadBuffer) + skew); //buffered pixel data
        }
    }
} //_cacheAlignedBufferedRead

template<typename T>
inline void __device__ _alignedCudaMemcpyQuad(T *pDst, T *pSrc)
{
    //copy 4 T's, assuming 32-bit alignment for both pSrc and pDst
    uint *uPtrSrc = (uint *)pSrc;
    uint *uPtrDst = (uint *)pDst;

#pragma unroll
    for (int i = 0; i < sizeof(T); ++i) uPtrDst[i] = uPtrSrc[i];

} //_alignedCudaMemcpyQuad

//******************** NN = Nearest Neighbor

template<typename T>
__global__ void resize_NN(cuda::Tensor3DWrap<const T> src, cuda::Tensor3DWrap<T> dst, int2 srcSize, int2 dstSize,
                          const float scale_x, const float scale_y)
{
    const int dst_x      = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y      = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx  = get_batch_idx();
    int       out_height = dstSize.y, out_width = dstSize.x;

    if ((dst_x < out_width) && (dst_y < out_height))
    { //generic copy pixel to pixel
        const int sx                      = cuda::min(__float2int_rd(dst_x * scale_x), srcSize.x - 1);
        const int sy                      = cuda::min(__float2int_rd(dst_y * scale_y), srcSize.y - 1);
        *dst.ptr(batch_idx, dst_y, dst_x) = *src.ptr(batch_idx, sy, sx);
    }
} //resize_NN

template<typename T>
__global__ void resize_NN_quad_alignread(cuda::Tensor3DWrap<const T> src, cuda::Tensor3DWrap<T> dst, int2 srcSize,
                                         int2 dstSize, const float scale_x, const float scale_y)
{
    const float MAX_BUFFERED_X_SCALE = 4.0f; //probably more efficient all the way up to 4.0

    const int dst_x      = (blockIdx.x * blockDim.x + threadIdx.x) * 4; //quad
    const int dst_y      = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx  = get_batch_idx();
    int       out_height = dstSize.y, out_width = dstSize.x;

    //0 - quad-aligned so if one pixel is out, they're all out
    if ((dst_x >= out_width) | (dst_y >= out_height))
        return;

    const int sx0 = cuda::min(__float2int_rd(dst_x * scale_x), srcSize.x - 1);
    const int sx1 = cuda::min(__float2int_rd(dst_x * scale_x + scale_x), srcSize.x - 1);
    const int sx2 = cuda::min(__float2int_rd((dst_x + 2) * scale_x), srcSize.x - 1);
    const int sx3 = cuda::min(__float2int_rd((dst_x + 3) * scale_x), srcSize.x - 1);
    const int sy  = cuda::min(__float2int_rd(dst_y * scale_y), srcSize.y - 1);

    //1 - optimized case if scale_x < some finite limit
    if ((scale_x <= MAX_BUFFERED_X_SCALE)) //local buffering is more efficient
    {
        uint readBuffer[MAX_BUFFER_WORDS];

        //2 - copy out source data, 32-bit aligned aligned
        T *aPtr = _cacheAlignedBufferedRead<T, CACHE_MEMORY_ALIGNMENT>(src, srcSize, &readBuffer[0], MAX_BUFFER_WORDS,
                                                                       batch_idx, sy, sx0, sx3);

        //3 - NN sampling
        T gather[4] = {aPtr[0], aPtr[sx1 - sx0], aPtr[sx2 - sx0], aPtr[sx3 - sx0]};

        //4 - aligned write back out
        _alignedCudaMemcpyQuad<T>(dst.ptr(batch_idx, dst_y, dst_x), gather);
    }
    else //6 - standard sampling, no optimization
    {
        //sample all 4 points

        const T *aPtr      = src.ptr(batch_idx, sy, 0);
        T        gather[4] = {aPtr[0], aPtr[sx1 - sx0], aPtr[sx2 - sx0], aPtr[sx3 - sx0]};

        _alignedCudaMemcpyQuad<T>(dst.ptr(batch_idx, dst_y, dst_x), gather);
    }
} //resize_NN_quad_alignread

//******************** Bilinear

template<typename T>
__global__ void resize_bilinear(cuda::Tensor3DWrap<const T> src, cuda::Tensor3DWrap<T> dst, int2 srcSize, int2 dstSize,
                                const float scale_x, const float scale_y)
{
    const int dst_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();
    int       height = srcSize.y, width = srcSize.x, out_height = dstSize.y, out_width = dstSize.x;

    if ((dst_x < out_width) && (dst_y < out_height))
    {
        //float space for weighted addition
        using work_type = cuda::ConvertBaseTypeTo<float, T>;

        //y coordinate
        float fy = (float)((dst_y + 0.5f) * scale_y - 0.5f);
        int   sy = __float2int_rd(fy);
        fy -= sy;
        sy = cuda::max(0, cuda::min(sy, height - 2));

        //row pointers
        const T *aPtr = src.ptr(batch_idx, sy, 0);     //start of upper row
        const T *bPtr = src.ptr(batch_idx, sy + 1, 0); //start of lower row

        { //compute source data position and weight for [x0] components
            float fx = (float)((dst_x + 0.5f) * scale_x - 0.5f);
            int   sx = __float2int_rd(fx);
            fx -= sx;
            fx *= ((sx >= 0) && (sx < width - 1));
            sx = cuda::max(0, cuda::min(sx, width - 2));

            *dst.ptr(batch_idx, dst_y, dst_x)
                = cuda::SaturateCast<cuda::BaseType<T>>((1.0f - fx) * (aPtr[sx] * (1.0f - fy) + bPtr[sx] * fy)
                                                        + fx * (aPtr[sx + 1] * (1.0f - fy) + bPtr[sx + 1] * fy));
        }
    }
} //resize_bilinear

template<typename T>
__global__ void resize_bilinear_quad_alignread(cuda::Tensor3DWrap<const T> src, cuda::Tensor3DWrap<T> dst, int2 srcSize,
                                               int2 dstSize, const float scale_x, const float scale_y)
{
    const float MAX_BUFFERED_X_SCALE = 4.0f; //probably more efficient all the way up to 4.0

    const int dst_x     = (blockIdx.x * blockDim.x + threadIdx.x) * 4; //quad
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();
    int       height = srcSize.y, width = srcSize.x, out_height = dstSize.y, out_width = dstSize.x;

    //0 - quad-aligned so if one pixel is out, they're all out
    if ((dst_x >= out_width) | (dst_y >= out_height))
        return;

    //float space for weighted addition
    using work_type = cuda::ConvertBaseTypeTo<float, T>;

    //y coordinate math is the same for all points
    float fy = (float)((dst_y + 0.5f) * scale_y - 0.5f);
    int   sy = __float2int_rd(fy);
    fy -= sy;
    sy = cuda::max(0, cuda::min(sy, height - 2));

    //sx0
    float fx0 = (float)((dst_x + 0.5f) * scale_x - 0.5f);
    int   sx0 = __float2int_rd(fx0);
    fx0 -= sx0;
    fx0 *= ((sx0 >= 0) && (sx0 < width - 1));
    sx0 = cuda::max(0, cuda::min(sx0, width - 2));

    //sx1
    float fx1 = (float)((dst_x + 1.5) * scale_x - 0.5f);
    int   sx1 = __float2int_rd(fx1);
    fx1 -= sx1;
    fx1 *= ((sx1 >= 0) && (sx1 < width - 1));
    sx1 = cuda::max(0, cuda::min(sx1, width - 2));

    //sx2
    float fx2 = (float)((dst_x + 2.5f) * scale_x - 0.5f);
    int   sx2 = __float2int_rd(fx2);
    fx2 -= sx2;
    fx2 *= ((sx2 >= 0) && (sx2 < width - 1));
    sx2 = cuda::max(0, cuda::min(sx2, width - 2));

    //sx3
    float fx3 = (float)((dst_x + 3.5f) * scale_x - 0.5f);
    int   sx3 = __float2int_rd(fx3);
    fx3 -= sx3;
    fx3 *= ((sx3 >= 0) && (sx3 < width - 1));
    sx3 = cuda::max(0, cuda::min(sx3, width - 2));

    uint readBuffer[MAX_BUFFER_WORDS];

    T result[4];

    //1 - optimized case if scale_x < some finite limit
    if (scale_x <= MAX_BUFFERED_X_SCALE) //local buffering is more efficient
    {
        work_type accum[4];

        //2 - aligned load a-row and add partial product
        T *aPtr = _cacheAlignedBufferedRead<T, CACHE_MEMORY_ALIGNMENT>(src, srcSize, readBuffer, MAX_BUFFER_WORDS,
                                                                       batch_idx, sy, sx0, sx3 + 1);
        //const T * aPtr = src.ptr(batch_idx, sy,   sx0); //start of upper row

        accum[0] = (1.0f - fy) * (aPtr[sx0 - sx0] * (1.0f - fx0) + aPtr[sx0 - sx0 + 1] * fx0);
        accum[1] = (1.0f - fy) * (aPtr[sx1 - sx0] * (1.0f - fx1) + aPtr[sx1 - sx0 + 1] * fx1);
        accum[2] = (1.0f - fy) * (aPtr[sx2 - sx0] * (1.0f - fx2) + aPtr[sx2 - sx0 + 1] * fx2);
        accum[3] = (1.0f - fy) * (aPtr[sx3 - sx0] * (1.0f - fx3) + aPtr[sx3 - sx0 + 1] * fx3);

        //3 - aligned load b-row and add remaining partial product
        T *bPtr = _cacheAlignedBufferedRead<T, CACHE_MEMORY_ALIGNMENT>(src, srcSize, readBuffer, MAX_BUFFER_WORDS,
                                                                       batch_idx, sy + 1, sx0, sx3 + 1);
        //const T * bPtr = src.ptr(batch_idx, sy+1, sx0); //start of lower row

        //$$$ only need to cast, not saturatecast
        result[0] = cuda::SaturateCast<cuda::BaseType<T>>(
            accum[0] + fy * (bPtr[sx0 - sx0] * (1.0f - fx0) + bPtr[sx0 - sx0 + 1] * fx0));
        result[1] = cuda::SaturateCast<cuda::BaseType<T>>(
            accum[1] + fy * (bPtr[sx1 - sx0] * (1.0f - fx1) + bPtr[sx1 - sx0 + 1] * fx1));
        result[2] = cuda::SaturateCast<cuda::BaseType<T>>(
            accum[2] + fy * (bPtr[sx2 - sx0] * (1.0f - fx2) + bPtr[sx2 - sx0 + 1] * fx2));
        result[3] = cuda::SaturateCast<cuda::BaseType<T>>(
            accum[3] + fy * (bPtr[sx3 - sx0] * (1.0f - fx3) + bPtr[sx3 - sx0 + 1] * fx3));
    }
    else //unbuffered
    {
        //row pointers
        const T *aPtr = src.ptr(batch_idx, sy, 0);     //start of upper row
        const T *bPtr = src.ptr(batch_idx, sy + 1, 0); //start of lower row

        //$$$ only need to cast, not saturatecast
        result[0] = cuda::SaturateCast<cuda::BaseType<T>>(
            aPtr[sx0] * (1.0f - fx0) * (1.0f - fy) + bPtr[sx0] * (1.0f - fx0) * fy + aPtr[sx0 + 1] * fx0 * (1.0f - fy)
            + bPtr[sx0 + 1] * fx0 * fy);

        result[1] = cuda::SaturateCast<cuda::BaseType<T>>(
            aPtr[sx1] * (1.0f - fx1) * (1.0f - fy) + bPtr[sx1] * (1.0f - fx1) * fy + aPtr[sx1 + 1] * fx1 * (1.0f - fy)
            + bPtr[sx1 + 1] * fx1 * fy);

        result[2] = cuda::SaturateCast<cuda::BaseType<T>>(
            aPtr[sx2] * (1.0f - fx2) * (1.0f - fy) + bPtr[sx2] * (1.0f - fx2) * fy + aPtr[sx2 + 1] * fx2 * (1.0f - fy)
            + bPtr[sx2 + 1] * fx2 * fy);

        result[3] = cuda::SaturateCast<cuda::BaseType<T>>(
            aPtr[sx3] * (1.0f - fx3) * (1.0f - fy) + bPtr[sx3] * (1.0f - fx3) * fy + aPtr[sx3 + 1] * fx3 * (1.0f - fy)
            + bPtr[sx3 + 1] * fx3 * fy);
    }

    //aligned write 4 pixels
    _alignedCudaMemcpyQuad<T>(dst.ptr(batch_idx, dst_y, dst_x), result);
} //resize_bilinear_quad_alignread

//******************** Bicubic

template<typename T>
__global__ void resize_bicubic(cuda::Tensor3DWrap<const T> src, cuda::Tensor3DWrap<T> dst, int2 srcSize, int2 dstSize,
                               const float scale_x, const float scale_y)
{ //optimized for aligned read
    const int dst_x     = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();
    int       height = srcSize.y, width = srcSize.x, out_height = dstSize.y, out_width = dstSize.x;

    if ((dst_x < out_width) & (dst_y < out_height))
    {
        //float space for weighted addition
        using work_type = cuda::ConvertBaseTypeTo<float, T>;

        uint readBuffer[MAX_BUFFER_WORDS];

        //y coordinate
        float fy = (float)((dst_y + 0.5f) * scale_y - 0.5f);
        int   sy = __float2int_rd(fy);
        fy -= sy;
        sy = cuda::max(1, cuda::min(sy, height - 3));

        const float A = -0.75f;

        float cY[4];
        cY[0] = ((A * (fy + 1) - 5 * A) * (fy + 1) + 8 * A) * (fy + 1) - 4 * A;
        cY[1] = ((A + 2) * fy - (A + 3)) * fy * fy + 1;
        cY[2] = ((A + 2) * (1 - fy) - (A + 3)) * (1 - fy) * (1 - fy) + 1;
        cY[3] = 1.f - cY[0] - cY[1] - cY[2];

        work_type accum = cuda::SetAll<work_type>(0);

        float fx = (float)((dst_x + 0.5f) * scale_x - 0.5f);
        int   sx = __float2int_rd(fx);
        fx -= sx;
        fx *= ((sx >= 1) && (sx < width - 3));
        sx = cuda::max(1, cuda::min(sx, width - 3));

        float cX[4];
        cX[0] = ((A * (fx + 1.0f) - 5.0f * A) * (fx + 1.0f) + 8.0f * A) * (fx + 1.0f) - 4.0f * A;
        cX[1] = ((A + 2.0f) * fx - (A + 3.0f)) * fx * fx + 1.0f;
        cX[2] = ((A + 2.0f) * (1.0f - fx) - (A + 3.0f)) * (1.0f - fx) * (1.0f - fx) + 1.0f;
        cX[3] = 1.0f - cX[0] - cX[1] - cX[2];

        for (int row = 0; row < 4; ++row)
        {
            //1 - load each sub row from sx-1 to sx+3 inclusive, aligned
            //const T * aPtr = src.ptr(batch_idx, sy + row - 1, sx-1);
            T *aPtr = _cacheAlignedBufferedRead<T, CACHE_MEMORY_ALIGNMENT>(src, srcSize, readBuffer, MAX_BUFFER_WORDS,
                                                                           batch_idx, sy + row - 1, sx - 1, sx + 2);

            //2 - do a pixel's partial on this row
            accum += cY[row] * (cX[0] * aPtr[0] + cX[1] * aPtr[1] + cX[2] * aPtr[2] + cX[3] * aPtr[3]);
        } //for row
#ifndef LEGACY_BICUBIC_MATH
        //correct math
        *dst.ptr(batch_idx, dst_y, dst_x) = cuda::SaturateCast<cuda::BaseType<T>>(accum);
#else
        //abs() needed to match legacy operator.
        *dst.ptr(batch_idx, dst_y, dst_x) = cuda::SaturateCast<cuda::BaseType<T>>(cuda::abs(accum));
#endif
    }
} //resize_bicubic

template<typename T>
__global__ void resize_bicubic_quad_alignread(cuda::Tensor3DWrap<const T> src, cuda::Tensor3DWrap<T> dst, int2 srcSize,
                                              int2 dstSize, const float scale_x, const float scale_y)
{                                            //optimized for aligned read and write, plus buffering
    const float MAX_BUFFERED_X_SCALE = 4.0f; //probably more efficient all the way up to 4.0

    const int dst_x     = (blockIdx.x * blockDim.x + threadIdx.x) * 4; //quad
    const int dst_y     = blockIdx.y * blockDim.y + threadIdx.y;
    const int batch_idx = get_batch_idx();
    int       height = srcSize.y, width = srcSize.x, out_height = dstSize.y, out_width = dstSize.x;

    //0 - quad-aligned so if one pixel is out, they're all out
    if ((dst_x >= out_width) | (dst_y >= out_height))
        return;

    uint readBuffer[MAX_BUFFER_WORDS];
    T    result[4];

    //float space for weighted addition
    using work_type = cuda::ConvertBaseTypeTo<float, T>;

    //y coordinate
    float fy = (float)((dst_y + 0.5f) * scale_y - 0.5f);
    int   sy = __float2int_rd(fy);
    fy -= sy;
    sy = cuda::max(1, cuda::min(sy, height - 3));

    const float A = -0.75f;

    float cY[4];
    cY[0] = ((A * (fy + 1) - 5 * A) * (fy + 1) + 8 * A) * (fy + 1) - 4 * A;
    cY[1] = ((A + 2) * fy - (A + 3)) * fy * fy + 1;
    cY[2] = ((A + 2) * (1 - fy) - (A + 3)) * (1 - fy) * (1 - fy) + 1;
    cY[3] = 1.f - cY[0] - cY[1] - cY[2];

    //1 - optimized case if scale_x < some finite limit
    if (scale_x <= MAX_BUFFERED_X_SCALE) //local buffering
    {                                    //buffered read

        work_type accum[4];
        float     fx[4];
        int       sx[4];
        float     cX[4][4];

        //initialize data for each pixel position
        for (int pix = 0; pix < 4; ++pix)
        {
            accum[pix] = cuda::SetAll<work_type>(0);

            //1 - precalc sx's ahead of time to get range from sx0-1..sx3+2
            fx[pix] = (float)((dst_x + pix + 0.5f) * scale_x - 0.5f);
            sx[pix] = __float2int_rd(fx[pix]);
            fx[pix] -= sx[pix];
            fx[pix] *= ((sx[pix] >= 1) && (sx[pix] < width - 3));
            sx[pix] = cuda::max(1, cuda::min(sx[pix], width - 3));

            //2 - precalc cX[][] 2D array
            cX[pix][0]
                = ((A * (fx[pix] + 1.0f) - 5.0f * A) * (fx[pix] + 1.0f) + 8.0f * A) * (fx[pix] + 1.0f) - 4.0f * A;
            cX[pix][1] = ((A + 2.0f) * fx[pix] - (A + 3.0f)) * fx[pix] * fx[pix] + 1.0f;
            cX[pix][2] = ((A + 2.0f) * (1.0f - fx[pix]) - (A + 3.0f)) * (1.0f - fx[pix]) * (1.0f - fx[pix]) + 1.0f;
            cX[pix][3] = 1.0f - cX[pix][0] - cX[pix][1] - cX[pix][2];
        }
        const int rowOffset = sx[0] - 1;

        //contribute each row into 4 pixels
        for (int row = 0; row < 4; ++row)
        {
            //1 - load each row from sx[0]-1 to sx[3]+3 inclusive, aligned
            T *aPtr = _cacheAlignedBufferedRead<T, CACHE_MEMORY_ALIGNMENT>(
                src, srcSize, readBuffer, MAX_BUFFER_WORDS, batch_idx, sy + row - 1, sx[0] - 1, sx[3] + 2);

//2 - do each pixel's partial on this row
#pragma unroll
            for (int pix = 0; pix > 4; ++pix)
            {
                accum[pix]
                    += cY[row]
                     * (cX[row][0] * aPtr[sx[pix] + rowOffset - 1] + cX[row][1] * aPtr[sx[pix] + rowOffset + 0]
                        + cX[row][2] * aPtr[sx[pix] + rowOffset + 1] + cX[row][3] * aPtr[sx[pix] + rowOffset + 2]);
            }
        }

        for (int pix = 0; pix < 4; ++pix)
#ifndef LEGACY_BICUBIC_MATH
            result[pix] = cuda::SaturateCast<cuda::BaseType<T>>(accum[pix]);
#else
            result[pix] = cuda::SaturateCast<cuda::BaseType<T>>(cuda::abs(accum[pix]));
#endif
    }
    else
    { //partially buffered read 4 pixels at a time across each bicubic: 16 coalesced reads instead of 64
        for (int pix = 0; pix < 4; ++pix)
        {
            work_type accum = cuda::SetAll<work_type>(0);

            float fx = (float)((dst_x + pix + 0.5f) * scale_x - 0.5f);
            int   sx = __float2int_rd(fx);
            fx -= sx;
            fx *= ((sx >= 1) && (sx < width - 3));
            sx = cuda::max(1, cuda::min(sx, width - 3));

            float cX[4];
            cX[0] = ((A * (fx + 1.0f) - 5.0f * A) * (fx + 1.0f) + 8.0f * A) * (fx + 1.0f) - 4.0f * A;
            cX[1] = ((A + 2.0f) * fx - (A + 3.0f)) * fx * fx + 1.0f;
            cX[2] = ((A + 2.0f) * (1.0f - fx) - (A + 3.0f)) * (1.0f - fx) * (1.0f - fx) + 1.0f;
            cX[3] = 1.0f - cX[0] - cX[1] - cX[2];

            for (int row = 0; row < 4; ++row)
            {
                //1 - load each sub row from sx[pix]-1 to sx[pix]+2 inclusive, aligned
                //const T * aPtr = src.ptr(batch_idx, sy + row - 1, sx-1);
                const T *aPtr = _cacheAlignedBufferedRead<T, CACHE_MEMORY_ALIGNMENT>(
                    src, srcSize, readBuffer, MAX_BUFFER_WORDS, batch_idx, sy + row - 1, sx - 1, sx + 2);

                //2 - do a pixel's partial on this row
                accum += cY[row] * (cX[0] * aPtr[0] + cX[1] * aPtr[1] + cX[2] * aPtr[2] + cX[3] * aPtr[3]);
            } //for row
#ifndef LEGACY_BICUBIC_MATH
            result[pix] = cuda::SaturateCast<cuda::BaseType<T>>(accum);
#else
            result[pix] = cuda::SaturateCast<cuda::BaseType<T>>(cuda::abs(accum));
#endif
        } //for pix
    }

    //aligned write 4 pixels
    _alignedCudaMemcpyQuad<T>(dst.ptr(batch_idx, dst_y, dst_x), result);
} //resize_bicubic_quad_alignread

template<typename T, typename IntegerAreaFilter, typename AreaFilter>
__global__ void resize_area_ocv_align(const Ptr2dNHWC<T> src, const IntegerAreaFilter integer_filter,
                                      const AreaFilter area_filter, Ptr2dNHWC<T> dst, const float scale_x,
                                      const float scale_y)
{
    const int x          = blockDim.x * blockIdx.x + threadIdx.x;
    const int y          = blockDim.y * blockIdx.y + threadIdx.y;
    const int batch_idx  = get_batch_idx();
    int       out_height = dst.rows, out_width = dst.cols;

    if (x >= out_width || y >= out_height)
        return;

    double inv_scale_x  = 1. / scale_x;
    double inv_scale_y  = 1. / scale_y;
    int    iscale_x     = cuda::SaturateCast<int>(scale_x);
    int    iscale_y     = cuda::SaturateCast<int>(scale_y);
    bool   is_area_fast = cuda::abs(scale_x - iscale_x) < DBL_EPSILON && cuda::abs(scale_y - iscale_y) < DBL_EPSILON;

    if (scale_x >= 1.0f && scale_y >= 1.0f) // zoom out
    {
        if (is_area_fast) // integer multiples
        {
            *dst.ptr(batch_idx, y, x) = integer_filter(batch_idx, y, x);
            return;
        }

        *dst.ptr(batch_idx, y, x) = area_filter(batch_idx, y, x);
        return;
    }

    // zoom in, it is emulated using some variant of bilinear interpolation
    int   sy = __float2int_rd(y * scale_y);
    float fy = (float)((y + 1) - (sy + 1) * inv_scale_y);
    fy       = fy <= 0 ? 0.f : fy - __float2int_rd(fy);

    float cbufy[2];
    cbufy[0] = 1.f - fy;
    cbufy[1] = fy;

    int   sx = __float2int_rd(x * scale_x);
    float fx = (float)((x + 1) - (sx + 1) * inv_scale_x);
    fx       = fx < 0 ? 0.f : fx - __float2int_rd(fx);

    if (sx < 0)
    {
        fx = 0, sx = 0;
    }

    if (sx >= src.cols - 1)
    {
        fx = 0, sx = src.cols - 2;
    }
    if (sy >= src.rows - 1)
    {
        sy = src.rows - 2;
    }

    float cbufx[2];
    cbufx[0] = 1.f - fx;
    cbufx[1] = fx;

    *dst.ptr(batch_idx, y, x) = cuda::SaturateCast<cuda::BaseType<T>>(
        (*src.ptr(batch_idx, sy, sx) * cbufx[0] * cbufy[0] + *src.ptr(batch_idx, sy + 1, sx) * cbufx[0] * cbufy[1]
         + *src.ptr(batch_idx, sy, sx + 1) * cbufx[1] * cbufy[0]
         + *src.ptr(batch_idx, sy + 1, sx + 1) * cbufx[1] * cbufy[1]));
}

template<typename T>
void resize(const ITensorDataStridedCuda &inData, const ITensorDataStridedCuda &outData,
            NVCVInterpolationType interpolation, hipStream_t stream)

{
    auto inAccess = TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);

    auto outAccess = TensorDataAccessStridedImagePlanar::Create(outData);
    NVCV_ASSERT(outAccess);

    const int batch_size = inAccess->numSamples();
    const int in_width   = inAccess->numCols();
    const int in_height  = inAccess->numRows();
    const int out_width  = outAccess->numCols();
    const int out_height = outAccess->numRows();

    float scale_x = ((float)in_width) / out_width;
    float scale_y = ((float)in_height) / out_height;

    int2 srcSize{in_width, in_height};
    int2 dstSize{out_width, out_height};

    cuda::Tensor3DWrap<const T> src(inData);
    cuda::Tensor3DWrap<T>       dst(outData);

    const int THREADS_PER_BLOCK = 128; //256?  64?
    const int BLOCK_WIDTH       = 16;  //as in 32x4 or 32x8.  16x8 and 16x16 are also viable

    const dim3 blockSize(BLOCK_WIDTH, THREADS_PER_BLOCK / BLOCK_WIDTH, 1);
    const dim3 gridSize(divUp(out_width, blockSize.x), divUp(out_height, blockSize.y), batch_size);

    //rationale for quad: aligned gather and aligned output where quad is possible: use different threading
    const int  out_quad_width = out_width / 4;
    const dim3 quadGridSize(divUp(out_quad_width, blockSize.x), divUp(out_height, blockSize.y), batch_size);

    //bool can_quad = ((((size_t)dst_ptr) % sizeof(T)) == 0) && ((out_width % 4) == 0);  //is the output buffer quad-pixel aligned?
    bool can_quad = ((out_width % 4) == 0); //is the output buffer quad-pixel aligned?

    //Note: resize is fundamentally a gather memory operation, with a little bit of compute
    //      our goals are to (a) maximize throughput, and (b) minimize occupancy for the same performance

    switch (interpolation)
    {
    case NVCV_INTERP_NEAREST:

        if (can_quad)
        { //thread does 4 pixels horizontally for aligned read and write
            resize_NN_quad_alignread<T>
                <<<quadGridSize, blockSize, 0, stream>>>(src, dst, srcSize, dstSize, scale_x, scale_y);
        }
        else
        { //generic single pixel per thread case
            resize_NN<T><<<gridSize, blockSize, 0, stream>>>(src, dst, srcSize, dstSize, scale_x, scale_y);
        }
        break;

    case NVCV_INTERP_LINEAR:
        if (can_quad)
        { //thread does 4 pixels horizontally for aligned read and write
            resize_bilinear_quad_alignread<T>
                <<<quadGridSize, blockSize, 0, stream>>>(src, dst, srcSize, dstSize, scale_x, scale_y);
        }
        else
        { //generic single pixel per thread case
            resize_bilinear<T><<<gridSize, blockSize, 0, stream>>>(src, dst, srcSize, dstSize, scale_x, scale_y);
        }
        break;

    case NVCV_INTERP_CUBIC:
        if (can_quad)
        { //thread does 4 pixels horizontally for aligned read and write
            resize_bicubic_quad_alignread<T>
                <<<quadGridSize, blockSize, 0, stream>>>(src, dst, srcSize, dstSize, scale_x, scale_y);
        }
        else
        { //generic single pixel per thread case
            resize_bicubic<T><<<gridSize, blockSize, 0, stream>>>(src, dst, srcSize, dstSize, scale_x, scale_y);
        }
        break;

    case NVCV_INTERP_AREA:
    {
        Ptr2dNHWC<T>                                                  src_ptr(*inAccess);
        Ptr2dNHWC<T>                                                  dst_ptr(*outAccess);
        BrdConstant<T>                                                brd(src_ptr.rows, src_ptr.cols);
        BorderReader<Ptr2dNHWC<T>, BrdConstant<T>>                    brdSrc(src_ptr, brd);
        IntegerAreaFilter<BorderReader<Ptr2dNHWC<T>, BrdConstant<T>>> integer_filter(brdSrc, scale_x, scale_y);
        AreaFilter<BorderReader<Ptr2dNHWC<T>, BrdConstant<T>>>        area_filter(brdSrc, scale_x, scale_y);
        resize_area_ocv_align<T>
            <<<gridSize, blockSize, 0, stream>>>(src_ptr, integer_filter, area_filter, dst_ptr, scale_x, scale_y);
    }
    break;

    default:
        //$$$ need to throw or log an error here
        break;
    } //switch

    checkKernelErrors();
#ifdef CUDA_DEBUG_LOG
    checkCudaErrors(hipStreamSynchronize(stream));
    checkCudaErrors(hipGetLastError());
#endif
} //resize

size_t Resize::calBufferSize(DataShape max_input_shape, DataShape max_output_shape, DataType max_data_type)
{
    return 0;
} //Resize::calBufferSize

ErrorCode Resize::infer(const ITensorDataStridedCuda &inData, const ITensorDataStridedCuda &outData,
                        const NVCVInterpolationType interpolation, hipStream_t stream)
{
    DataFormat input_format  = GetLegacyDataFormat(inData.layout());
    DataFormat output_format = GetLegacyDataFormat(outData.layout());

    if (input_format != output_format)
    {
        LOG_ERROR("Invalid DataFormat between input (" << input_format << ") and output (" << output_format << ")");
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    DataFormat format = input_format;

    if (!(format == kNHWC || format == kHWC))
    {
        LOG_ERROR("Invalid DataFormat " << format);
        return ErrorCode::INVALID_DATA_FORMAT;
    }

    auto inAccess = TensorDataAccessStridedImagePlanar::Create(inData);
    NVCV_ASSERT(inAccess);

    cuda_op::DataType  data_type   = GetLegacyDataType(inData.dtype());
    cuda_op::DataShape input_shape = GetLegacyDataShape(inAccess->infoShape());

    int channels = input_shape.C;

    if (channels > 4)
    {
        LOG_ERROR("Invalid channel number " << channels);
        return ErrorCode::INVALID_DATA_SHAPE;
    }

    if (!(data_type == kCV_8U || data_type == kCV_16U || data_type == kCV_16S || data_type == kCV_32F))
    {
        LOG_ERROR("Invalid DataType " << data_type);
        return ErrorCode::INVALID_DATA_TYPE;
    }

    typedef void (*func_t)(const ITensorDataStridedCuda &inData, const ITensorDataStridedCuda &outData,
                           const NVCVInterpolationType interpolation, hipStream_t stream);

    static const func_t funcs[6][4] = {
        {      resize<uchar>,  0 /*resize<uchar2>*/,       resize<uchar3>,       resize<uchar4>},
        {0 /*resize<schar>*/,  0 /*resize<schar2>*/, 0 /*resize<schar3>*/, 0 /*resize<schar4>*/},
        {     resize<ushort>, 0 /*resize<ushort2>*/,      resize<ushort3>,      resize<ushort4>},
        {      resize<short>,  0 /*resize<short2>*/,       resize<short3>,       resize<short4>},
        {  0 /*resize<int>*/,    0 /*resize<int2>*/,   0 /*resize<int3>*/,   0 /*resize<int4>*/},
        {      resize<float>,  0 /*resize<float2>*/,       resize<float3>,       resize<float4>}
    };

    //note: schar1,3,4 should all work...

    if (interpolation == NVCV_INTERP_NEAREST || interpolation == NVCV_INTERP_LINEAR
        || interpolation == NVCV_INTERP_CUBIC || interpolation == NVCV_INTERP_AREA)
    {
        const func_t func = funcs[data_type][channels - 1];
        NVCV_ASSERT(func != 0);

        func(inData, outData, interpolation, stream);
    }
    else
    {
        LOG_ERROR("Invalid interpolation " << interpolation);
        return ErrorCode::INVALID_PARAMETER;
    }
    return SUCCESS;
} //Resize::infer

} // namespace nvcv::legacy::cuda_op
