#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "DeviceBorderWrap.hpp" // to test in the device

#include <gtest/gtest.h>            // for EXPECT_EQ, etc.
#include <nvcv/cuda/BorderWrap.hpp> // the object of this test
#include <nvcv/cuda/MathOps.hpp>    // for operator *, etc.
#include <nvcv/cuda/StaticCast.hpp> // for StaticCast, etc.
#include <nvcv/cuda/TensorWrap.hpp> // for Tensor3DWrap, etc.

namespace cuda = nvcv::cuda;

// ------------------ To allow testing device-side BorderWrap ------------------

template<class DstWrapper, class SrcWrapper>
__global__ void FillBorder(DstWrapper dst, SrcWrapper src, int3 dstSize, int3 borderSize)
{
    int3 dstCoord = cuda::StaticCast<int>(blockIdx * blockDim + threadIdx);

    if (dstCoord.z >= dstSize.z || dstCoord.y >= dstSize.y || dstCoord.x >= dstSize.x)
    {
        return;
    }

    int3 srcCoord = {dstCoord.x - borderSize.x, dstCoord.y - borderSize.y, dstCoord.z};

    dst[dstCoord] = src[srcCoord];
}

template<class DstWrapper, class SrcWrapper>
__global__ void FillBorder(DstWrapper dst, SrcWrapper src, int4 dstSize, int4 borderSize)
{
    int3 dstCoord = cuda::StaticCast<int>(blockIdx * blockDim + threadIdx);

    if (dstCoord.z >= dstSize.z || dstCoord.y >= dstSize.y || dstCoord.x >= dstSize.x)
    {
        return;
    }

    int3 srcCoord = {dstCoord.x - borderSize.x, dstCoord.y - borderSize.y, dstCoord.z};

    typename DstWrapper::ValueType *dstPtr = dst.ptr(dstCoord.z, dstCoord.y, dstCoord.x);
    typename SrcWrapper::ValueType *srcPtr = src.ptr(srcCoord.z, srcCoord.y, srcCoord.x);

    for (int c = 0; c < dstSize.w; ++c)
    {
        *dstPtr++ = (srcPtr == nullptr) ? src.borderValue() : *srcPtr++;
    }
}

template<class DstWrapper, class SrcWrapper, typename DimType>
void DeviceRunFillBorder(DstWrapper &dstWrap, SrcWrapper &srcWrap, DimType dstSize, DimType srcSize,
                         hipStream_t &stream)
{
    dim3 block{32, 2, 2};
    dim3 grid{(dstSize.x + block.x - 1) / block.x, (dstSize.y + block.y - 1) / block.y,
              (dstSize.z + block.z - 1) / block.z};

    DimType borderSize = (dstSize - srcSize) / 2;

    FillBorder<<<grid, block, 0, stream>>>(dstWrap, srcWrap, dstSize, borderSize);
}

// Need to instantiate each test on TestBorderWrap

#define NVCV_TEST_INST(DSTWRAPPER, SRCWRAPPER, DIMTYPE) \
    template void DeviceRunFillBorder(DSTWRAPPER &, SRCWRAPPER &, DIMTYPE, DIMTYPE, hipStream_t &)

#define T3D(VALUETYPE) cuda::Tensor3DWrap<VALUETYPE>
#define T4D(VALUETYPE) cuda::Tensor4DWrap<VALUETYPE>

#define B3D(VALUETYPE, BORDERTYPE) cuda::BorderWrapNHW<VALUETYPE, BORDERTYPE>
#define B4D(VALUETYPE, BORDERTYPE) cuda::BorderWrapNHWC<VALUETYPE, BORDERTYPE>

NVCV_TEST_INST(T3D(uchar4), B3D(const uchar4, NVCV_BORDER_CONSTANT), int3);
NVCV_TEST_INST(T3D(short2), B3D(const short2, NVCV_BORDER_CONSTANT), int3);
NVCV_TEST_INST(T4D(float1), B4D(const float1, NVCV_BORDER_CONSTANT), int4);
NVCV_TEST_INST(T3D(float4), B3D(const float4, NVCV_BORDER_REPLICATE), int3);
NVCV_TEST_INST(T4D(short1), B4D(const short1, NVCV_BORDER_REPLICATE), int4);
NVCV_TEST_INST(T3D(float3), B3D(const float3, NVCV_BORDER_WRAP), int3);
NVCV_TEST_INST(T4D(uchar1), B4D(const uchar1, NVCV_BORDER_WRAP), int4);
NVCV_TEST_INST(T3D(uchar3), B3D(const uchar3, NVCV_BORDER_REFLECT), int3);
NVCV_TEST_INST(T4D(uchar1), B4D(const uchar1, NVCV_BORDER_REFLECT), int4);
NVCV_TEST_INST(T3D(short1), B3D(const short1, NVCV_BORDER_REFLECT101), int3);
NVCV_TEST_INST(T4D(uchar1), B4D(const uchar1, NVCV_BORDER_REFLECT101), int4);

#undef T3D
#undef T4D

#undef NVCV_TEST_INST
