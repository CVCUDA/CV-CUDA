#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "DeviceMathWrappers.hpp" // to test in the device

#include <gtest/gtest.h>              // for EXPECT_EQ, etc.
#include <nvcv/cuda/MathWrappers.hpp> // the object of this test

namespace cuda = nvcv::cuda;

// Need to instantiate each test on TestMathWrappers, making sure not to use const types

// -------------------- To allow testing device-side round ---------------------

template<typename SourceType, typename TargetType>
__global__ void RunRound(TargetType *out, SourceType u)
{
    if constexpr (std::is_same_v<SourceType, TargetType>)
    {
        out[0] = cuda::round(u);
    }
    else
    {
        out[0] = cuda::round<cuda::BaseType<TargetType>>(u);
    }
}

template<typename TargetType, typename SourceType>
TargetType DeviceRunRoundDiffType(SourceType pix)
{
    TargetType *dTest;
    TargetType  hTest[1];

    EXPECT_EQ(hipSuccess, hipMalloc(&dTest, sizeof(TargetType)));

    RunRound<<<1, 1>>>(dTest, pix);

    EXPECT_EQ(hipSuccess, hipDeviceSynchronize());
    EXPECT_EQ(hipSuccess, hipMemcpy(hTest, dTest, sizeof(TargetType), hipMemcpyDeviceToHost));

    EXPECT_EQ(hipSuccess, hipFree(dTest));

    return hTest[0];
}

template<typename Type>
Type DeviceRunRoundSameType(Type pix)
{
    return DeviceRunRoundDiffType<Type, Type>(pix);
}

#define NVCV_TEST_INST_ROUND_SAME(TYPE) template TYPE DeviceRunRoundSameType(TYPE pix)

NVCV_TEST_INST_ROUND_SAME(unsigned char);
NVCV_TEST_INST_ROUND_SAME(int);
NVCV_TEST_INST_ROUND_SAME(float);
NVCV_TEST_INST_ROUND_SAME(double);

NVCV_TEST_INST_ROUND_SAME(char1);
NVCV_TEST_INST_ROUND_SAME(uint2);
NVCV_TEST_INST_ROUND_SAME(float3);
NVCV_TEST_INST_ROUND_SAME(double4);

#undef NVCV_TEST_INST_ROUND_SAME

#define NVCV_TEST_INST_ROUND_DIFF(SOURCE_TYPE, TARGET_TYPE) template TARGET_TYPE DeviceRunRoundDiffType(SOURCE_TYPE pix)

NVCV_TEST_INST_ROUND_DIFF(float, int);
NVCV_TEST_INST_ROUND_DIFF(double, unsigned int);
NVCV_TEST_INST_ROUND_DIFF(float3, int3);
NVCV_TEST_INST_ROUND_DIFF(double4, long4);

NVCV_TEST_INST_ROUND_DIFF(signed char, signed char);
NVCV_TEST_INST_ROUND_DIFF(float2, float2);
NVCV_TEST_INST_ROUND_DIFF(uint1, uint1);
NVCV_TEST_INST_ROUND_DIFF(double2, double2);

#undef NVCV_TEST_INST_ROUND_DIFF

// -------------------- To allow testing device-side min ----------------------

template<typename Type>
__global__ void RunMin(Type *out, Type a, Type b)
{
    out[0] = cuda::min(a, b);
}

template<typename Type>
Type DeviceRunMin(Type pix1, Type pix2)
{
    Type *dTest;
    Type  hTest[1];

    EXPECT_EQ(hipSuccess, hipMalloc(&dTest, sizeof(Type)));

    RunMin<<<1, 1>>>(dTest, pix1, pix2);

    EXPECT_EQ(hipSuccess, hipDeviceSynchronize());
    EXPECT_EQ(hipSuccess, hipMemcpy(hTest, dTest, sizeof(Type), hipMemcpyDeviceToHost));

    EXPECT_EQ(hipSuccess, hipFree(dTest));

    return hTest[0];
}

#define NVCV_TEST_INST_MIN(TYPE) template TYPE DeviceRunMin(TYPE pix1, TYPE pix2)

NVCV_TEST_INST_MIN(unsigned char);
NVCV_TEST_INST_MIN(int);
NVCV_TEST_INST_MIN(float);
NVCV_TEST_INST_MIN(double);

NVCV_TEST_INST_MIN(char1);
NVCV_TEST_INST_MIN(uint2);
NVCV_TEST_INST_MIN(float3);
NVCV_TEST_INST_MIN(double4);

NVCV_TEST_INST_MIN(short2);
NVCV_TEST_INST_MIN(char4);
NVCV_TEST_INST_MIN(ushort2);
NVCV_TEST_INST_MIN(uchar4);

#undef NVCV_TEST_INST_MIN

// -------------------- To allow testing device-side max ----------------------

template<typename Type>
__global__ void RunMax(Type *out, Type a, Type b)
{
    out[0] = cuda::max(a, b);
}

template<typename Type>
Type DeviceRunMax(Type pix1, Type pix2)
{
    Type *dTest;
    Type  hTest[1];

    EXPECT_EQ(hipSuccess, hipMalloc(&dTest, sizeof(Type)));

    RunMax<<<1, 1>>>(dTest, pix1, pix2);

    EXPECT_EQ(hipSuccess, hipDeviceSynchronize());
    EXPECT_EQ(hipSuccess, hipMemcpy(hTest, dTest, sizeof(Type), hipMemcpyDeviceToHost));

    EXPECT_EQ(hipSuccess, hipFree(dTest));

    return hTest[0];
}

#define NVCV_TEST_INST_MAX(TYPE) template TYPE DeviceRunMax(TYPE pix1, TYPE pix2)

NVCV_TEST_INST_MAX(unsigned char);
NVCV_TEST_INST_MAX(int);
NVCV_TEST_INST_MAX(float);
NVCV_TEST_INST_MAX(double);

NVCV_TEST_INST_MAX(char1);
NVCV_TEST_INST_MAX(uint2);
NVCV_TEST_INST_MAX(float3);
NVCV_TEST_INST_MAX(double4);

NVCV_TEST_INST_MAX(short2);
NVCV_TEST_INST_MAX(char4);
NVCV_TEST_INST_MAX(ushort2);
NVCV_TEST_INST_MAX(uchar4);

#undef NVCV_TEST_INST_MAX

// --------------------- To allow testing device-side exp ----------------------

template<typename Type>
__global__ void RunExp(Type *out, Type u)
{
    out[0] = cuda::exp(u);
}

template<typename Type>
Type DeviceRunExp(Type pix)
{
    Type *dTest;
    Type  hTest[1];

    EXPECT_EQ(hipSuccess, hipMalloc(&dTest, sizeof(Type)));

    RunExp<<<1, 1>>>(dTest, pix);

    EXPECT_EQ(hipSuccess, hipDeviceSynchronize());
    EXPECT_EQ(hipSuccess, hipMemcpy(hTest, dTest, sizeof(Type), hipMemcpyDeviceToHost));

    EXPECT_EQ(hipSuccess, hipFree(dTest));

    return hTest[0];
}

#define NVCV_TEST_INST_EXP(TYPE) template TYPE DeviceRunExp(TYPE pix)

NVCV_TEST_INST_EXP(unsigned char);
NVCV_TEST_INST_EXP(int);
NVCV_TEST_INST_EXP(float);
NVCV_TEST_INST_EXP(double);

NVCV_TEST_INST_EXP(char1);
NVCV_TEST_INST_EXP(uint2);
NVCV_TEST_INST_EXP(float3);
NVCV_TEST_INST_EXP(double4);

#undef NVCV_TEST_INST_EXP

// -------------------- To allow testing device-side sqrt ----------------------

template<typename Type>
__global__ void RunSqrt(Type *out, Type u)
{
    out[0] = cuda::sqrt(u);
}

template<typename Type>
Type DeviceRunSqrt(Type pix)
{
    Type *dTest;
    Type  hTest[1];

    EXPECT_EQ(hipSuccess, hipMalloc(&dTest, sizeof(Type)));

    RunSqrt<<<1, 1>>>(dTest, pix);

    EXPECT_EQ(hipSuccess, hipDeviceSynchronize());
    EXPECT_EQ(hipSuccess, hipMemcpy(hTest, dTest, sizeof(Type), hipMemcpyDeviceToHost));

    EXPECT_EQ(hipSuccess, hipFree(dTest));

    return hTest[0];
}

#define NVCV_TEST_INST_SQRT(TYPE) template TYPE DeviceRunSqrt(TYPE pix)

NVCV_TEST_INST_SQRT(unsigned char);
NVCV_TEST_INST_SQRT(int);
NVCV_TEST_INST_SQRT(float);
NVCV_TEST_INST_SQRT(double);

NVCV_TEST_INST_SQRT(char1);
NVCV_TEST_INST_SQRT(uint2);
NVCV_TEST_INST_SQRT(float3);
NVCV_TEST_INST_SQRT(double4);

#undef NVCV_TEST_INST_SQRT

// -------------------- To allow testing device-side abs ----------------------

template<typename Type>
__global__ void RunAbs(Type *out, Type u)
{
    out[0] = cuda::abs(u);
}

template<typename Type>
Type DeviceRunAbs(Type pix)
{
    Type *dTest;
    Type  hTest[1];

    EXPECT_EQ(hipSuccess, hipMalloc(&dTest, sizeof(Type)));

    RunAbs<<<1, 1>>>(dTest, pix);

    EXPECT_EQ(hipSuccess, hipDeviceSynchronize());
    EXPECT_EQ(hipSuccess, hipMemcpy(hTest, dTest, sizeof(Type), hipMemcpyDeviceToHost));

    EXPECT_EQ(hipSuccess, hipFree(dTest));

    return hTest[0];
}

#define NVCV_TEST_INST_ABS(TYPE) template TYPE DeviceRunAbs(TYPE pix)

NVCV_TEST_INST_ABS(unsigned char);
NVCV_TEST_INST_ABS(int);
NVCV_TEST_INST_ABS(float);
NVCV_TEST_INST_ABS(double);

NVCV_TEST_INST_ABS(char1);
NVCV_TEST_INST_ABS(uint2);
NVCV_TEST_INST_ABS(float3);
NVCV_TEST_INST_ABS(double4);

NVCV_TEST_INST_ABS(short2);
NVCV_TEST_INST_ABS(char4);

#undef NVCV_TEST_INST_ABS
